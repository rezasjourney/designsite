#include "hip/hip_runtime.h"


#include "model.cuh"
//#include "design.cuh"


#define NUM_POINTS 5
__device__ const float spread_check_loc_x[NUM_POINTS] = {0.0,0.0,1.0,1.0,0.5};
__device__ const float spread_check_loc_y[NUM_POINTS] = {0.0,1.0,0.0,1.0,0.5};
//#define NUM_POINTS 5
//__device__ const float spread_check_loc_x[NUM_POINTS] = {0.0,0.0,0.0,1.0,1.0,1.0,0.5,0.5,0.5};
//__device__ const float spread_check_loc_y[NUM_POINTS] = {0.0,0.5,1.0,0.0,0.5,1.0,0.0,1.0,0.5};

const char* feat_names[] = {
			"Text Size",
			"Graphic Size",
			"Min Text Size",
			"Min Graphic Size",
			"Whitespace",
			"Spread",
			"Text Out of Bounds",
			"Graphic Out of Bounds",
			"Text Overlap",
			"Graphic Overlap",
			"Graphic Text Overlap",
			"Text X Symmetry",
			"Text Y Symmetry",
			"Graphic X Symmetry",
			"Graphic Y Symmetry",
			"Align X-Left",
			"Align X-Center",
			"Align X-Right",
			"Align Y-Bottom",
			"Align Y-Center",
			"Align Y-Top",
			"Align Error X",
			"Align Error Y",
			"Align Group Sizes",
			"Text Importance Pearson",
			"Graphic Importance Pearson",
			"Text X Position",
			"Graphic X Position",
			"Text Y Position",
			"Graphic Y Position",
			"Text X Position - Reverse",
			"Graphic X Position - Reverse",
			"Text Y Position - Reverse",
			"Graphic Y Position - Reverse",
			"Pairwise Distances Avg",
			"Pairwise Distances Min",
			"Text Margins Avg",
			"Text Margins Min",
			"Graphic Margins Avg",
			"Graphic Margins Min",
			"Text Size Variance",
			"Text X Flow",
			"Text Y Flow",
			"Num Regions",
			"Text Region X Symmetry",
			"Graphic Region X Symmetry",
			"Text X Reverse Symmetry",
			"Text Y Reverse Symmetry",
			"Graphic X Reverse Symmetry",
			"Graphic Y Reverse Symmetry",
			"Text X Position Variance",
			"Graphic X Position Variance",
			"Text Y Position Variance",
			"Graphic Y Position Variance",
			"Text Distances Avg",
			"Element Region Difference",
			"Empty Region",
			"Region Overlap",
			"Line Length",
			"Element 0 Position Difference",
			"Element 1 Position Difference",
			"Element 2 Position Difference",
			"Element 3 Position Difference",
			"Element 4 Position Difference",
			"Element 5 Position Difference",
			"Element 6 Position Difference",
			"Element 7 Position Difference",
			"Element 8 Position Difference",
			"Element 9 Position Difference",
			"Element 10 Position Difference",
			"Element 11 Position Difference",
			"Element 12 Position Difference",
			"Element 13 Position Difference",
			"Element 14 Position Difference",
			"Element 0 Height Difference",
			"Element 1 Height Difference",
			"Element 2 Height Difference",
			"Element 3 Height Difference",
			"Element 4 Height Difference",
			"Element 5 Height Difference",
			"Element 6 Height Difference",
			"Element 7 Height Difference",
			"Element 8 Height Difference",
			"Element 9 Height Difference",
			"Element 10 Height Difference",
			"Element 11 Height Difference",
			"Element 12 Height Difference",
			"Element 13 Height Difference",
			"Element 14 Height Difference",
			"Graphic Size Variance",
			"Text Diag Flow",
			"Group Distance",
			"No Overlap Regions",
			"Text Size Reverse",
			"Graphic Size Reverse",
			"Whitespace Reverse",
			"Previous Layout",
			"Group Alignment",
			"Group Text Size Variance",
			"Group Graphic Size Variance",
			"Group Graphic X Align Variance",
			"Group Graphic Y Align Variance",
			"Hidden Element",
			"Alignment Lines",
			"Size Constraints",
			"Alignment Constraints",
			"Relative Difference"
};



__device__ float getAtan(Design *d,float x)
{
	int idx=abs(int(x*200.0));

	float a=0;
	if (idx>=20000)
		a= d->atan_fixed[20000-1]+x/20000.0;
	else
		a= d->atan_fixed[idx];

	if (x <0)
		return -a;
	else
		return a;
}



__global__ void evaluateLayoutKernel(Design *d, float *layout, float *params, float *atan_params, float *params_grad,int num_prev_layout,float *previous_layout,  float *eval)
{
	*eval= evaluateLayout(d, layout, params, atan_params, params_grad,num_prev_layout,previous_layout,false,false,true);
}

__global__ void evaluateLayoutKernelDebug(Design *d, float *layout, float *params, float *atan_params, float *params_grad,int num_prev_layout,float *previous_layout, float *eval)
{
	*eval= evaluateLayout(d, layout, params, atan_params, params_grad,num_prev_layout,previous_layout,false,true,true);
}

__device__ double evaluateLayout(Design *d, float *layout, float *params,  float *atan_params, float *params_grad,int num_prev_layout, float *previous_layout, bool multithread,bool debug,bool calc_gradient)
{


	//if (blockIdx.x%EVAL_SPLIT_NUM!=0)
	//	return 99999;

	int thread_id =threadIdx.x +  int(blockIdx.x/EVAL_SPLIT_NUM)  * blockDim.x;


	int eval_id=blockIdx.x%EVAL_SPLIT_NUM;
        
	//printf("thread %i, block %i of %i\n",threadIdx.x,blockIdx.x , blockDim.x);



	/****************************
	 *
	 * Calculate Features Inputs
	 *
	 ***************************/
	
	
	int ne=d->num_elements;
	float scale=max(d->width, d->height);
	float atan_param=0;
	float atan_2_param=0;
	float atan_xy=0;


	Box designBB;
	designBB.set(0,d->width,0,d->height);
	float design_area=designBB.area();

	float height,width,xp,yp;
	
	float visible[MAX_ELEMENTS];
	Box elem_bb[MAX_ELEMENTS];
	
	float aspect_ratio[MAX_ELEMENTS];
	int num_lines[MAX_ELEMENTS];
	int nv=ne;


	for (int i=0;i<ne;i++)
	{
		visible[i]=true;
		
		int alt=int(layout[NUM_VAR*i+6]);
		if (alt>-1)
		{
			//printf("alt %i\n",alt);
			if ((d->num_alt[i]>0))
				aspect_ratio[i]=d->alt_aspect_ratio[i*MAX_ALT+alt];
			else
				aspect_ratio[i]=d->aspect_ratio[i];
				
			if ((d->num_alt[i]>0))
				num_lines[i]=d->alt_num_lines[i*MAX_ALT+alt];
			else
				num_lines[i]=d->num_lines[i];		
				
			
			if (aspect_ratio[i]==0)
				return ASPECT_RATIO_ERROR;
				
			
			height=round(layout[NUM_VAR*i+2]*d->height);
			width=round(height/aspect_ratio[i]);
			xp=round(layout[NUM_VAR*i]*d->width);
			yp=round(layout[NUM_VAR*i+1]*d->height);
	
			elem_bb[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
	
			if (height<=0)
			{
				//printf("Invalid negative height: %f\n",layout[NUM_VAR*i+2]);
				return INVALID_ELEMENT_ERROR;	
			}
	
			if ((debug) and (thread_id==0))
			{
				printf("%i: %f %f %f %f, fix %f (w/h) %f %f\n",i, layout[NUM_VAR*i],layout[NUM_VAR*i+1],layout[NUM_VAR*i+2],layout[NUM_VAR*i+3],layout[NUM_VAR*i+4],width,height);
				printf("%i: l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f, ar %.3f, alt %i\n",i,elem_bb[i].l,elem_bb[i].r,elem_bb[i].b,elem_bb[i].t,aspect_ratio[i],alt);
			}
	
			if ((elem_bb[i].l>=elem_bb[i].r) || (elem_bb[i].b>=elem_bb[i].t))
			{
				printf("l>r or b>t for element %i: l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f\n",i,elem_bb[i].l,elem_bb[i].r,elem_bb[i].b,elem_bb[i].t);
				printf("\t %f %f %f %f, w/h: %f %f, ar: %f\n",layout[NUM_VAR*i],layout[NUM_VAR*i+1],layout[NUM_VAR*i+2],layout[NUM_VAR*i+3],width,height,aspect_ratio[i]);
				return INVALID_ELEMENT_ERROR;
			}
	
			if ((elem_bb[i].l<-5*d->width) || (elem_bb[i].t<-5*d->height)|| (elem_bb[i].r>=5*d->width)|| (elem_bb[i].t>=5*d->height))
			//if ((elem_bb[i].r<=0) || (elem_bb[i].t<=0)|| (elem_bb[i].l>=d->width)|| (elem_bb[i].b>=d->height))
			{
				printf("oob %i: %f %f %f %f, w/h: %f %f, ar: %f\n",i, layout[NUM_VAR*i],layout[NUM_VAR*i+1],layout[NUM_VAR*i+2],layout[NUM_VAR*i+3],width,height,aspect_ratio[i]);
				printf("\t l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f\n",elem_bb[i].l,elem_bb[i].r,elem_bb[i].b,elem_bb[i].t);
			
				return INVALID_ELEMENT_ERROR;
			}

		}
		else
		{
			nv--;
			visible[i]=false;
			elem_bb[i].set(-1001,-1000, -1001,-1000);
		}
	}



	float bb_distance[2][MAX_ELEMENTS][MAX_ELEMENTS];

	float dist_scale=sqrt(d->height*d->height + d->width*d->width);

	for (int i=0;i<ne;i++)
	for (int j=i+1;j<ne;j++)
	{
        float locXDiff=-1*min((elem_bb[i].r-elem_bb[j].l),(elem_bb[j].r-elem_bb[i].l));
        bb_distance[0][i][j]=locXDiff;
        bb_distance[0][j][i]=locXDiff;

        float locYDiff=-1*min((elem_bb[i].t-elem_bb[j].b),(elem_bb[j].t-elem_bb[i].b));
        bb_distance[1][i][j]=locYDiff;
        bb_distance[1][j][i]=locYDiff;
	}



	//element types
	bool text_elements[MAX_ELEMENTS];
	bool graphic_elements[MAX_ELEMENTS];

	int num_text=0;
	int num_graphic=0;

	for (int i=0;i<ne;i++)
	{
		if (visible[i])
		{
			if (d->type[i]==1)
			{
				num_text++;
				text_elements[i]=1;
			}
			else
			{
				num_graphic++;
				text_elements[i]=0;
			}
	
			graphic_elements[i]=1-text_elements[i];
		}
	}
	
	if ((debug)&&(num_text==0))
	{
		printf("ERROR. No text elements\n");
		
		for (int i=0;i<ne;i++)
			printf("checking %i type %i visible %i\n",i,int(d->type[i]),int(visible[i]));
	}


	
	//calculate element sizes

	float sizes[MAX_ELEMENTS];
	
	for (int i=0;i<ne;i++)
	{
		if (!visible[i])
			continue;
			
		if (text_elements[i])
			sizes[i]=10.0*(((elem_bb[i].t-elem_bb[i].b)/num_lines[i])/(400));
		else
		{
			Box intersect=getBoxIntersection(designBB,elem_bb[i]);
			sizes[i]=(intersect.t-intersect.b)/ (400);
		}
		if ((thread_id==0) and (debug))
			printf("Size of element %d, f=%f\n", i,sizes[i]);
	}
	

	float2 center_pos[MAX_ELEMENTS];

	
	for (int i=0;i<ne;i++)
	{
		if (!visible[i])
			continue;
		center_pos[i].x=((elem_bb[i].l+elem_bb[i].r)/2.0)/float(d->width);
		center_pos[i].y=((elem_bb[i].t+elem_bb[i].b)/2.0)/float(d->height);
	}



	
	int internal_alignment[MAX_ELEMENTS];

	for (int i=0;i<ne;i++)
	{

		if (layout[NUM_VAR*i+3]==-1.0)
			internal_alignment[i]=(int) d->alignment[i];
		else
			internal_alignment[i]=(int) layout[NUM_VAR*i+3];

		if ((thread_id==0) and (debug))
			printf("Internal Alignment. Element %i, original %i, current %f, %i\n", i, d->alignment[i], layout[NUM_VAR*i+3],internal_alignment[i]);
	}

	/*
	int num_regions=0;
	int empty_regions=0;
	Box regions[MAX_ELEMENTS*2];
	Box regions_flipped_x[MAX_ELEMENTS*2];
	int region_type[MAX_ELEMENTS*2];
	int region_id[MAX_ELEMENTS];
	int offset=ne*NUM_VAR;
		
	if (layout[offset]>-1)
	{

		
		for (int i=0;i<ne;i++)
		{
			if (layout[offset+i*NUM_RVAR]>-1)
			{
				region_type[i]=layout[offset+i*NUM_RVAR];
				regions[i].set(layout[offset+i*NUM_RVAR+1],  layout[offset+i*NUM_RVAR+1]+layout[offset+i*NUM_RVAR+3],layout[offset+i*NUM_RVAR+2], layout[offset+i*NUM_RVAR+2]+layout[offset+i*NUM_RVAR+4]);
				
				regions_flipped_x[i].set(round(d->width-regions[i].r), round(d->width-regions[i].l),round(regions[i].b), round(regions[i].t));
				
				
				num_regions++;
			}
			else
				break;
		}
				
		for(int j=0;j<num_regions;j++)
		{
			bool empty_region=true;
			for(int i=0;i<ne;i++)
			{
				if (layout[i*NUM_VAR+5]==j)
				{
					region_id[i]=j;
					empty_region=false;	
				}		
			}
			if (empty_region)
				empty_regions++;
		}
		
		
	}
	
	
	if ((debug) and (thread_id==0))
	{
		for (int i=0;i < num_regions;i++)
			printf("R%i. Type %i: %.2f %.2f %.2f %.2f\n",i,region_type[i],regions[i].l,regions[i].r,regions[i].b,regions[i].t);

		for (int i=0;i < ne;i++)
			printf("Item %i in region R%i\n",i,region_id[i]);
			
			
		for (int i=0;i < ne;i++)
		{
			if (layout[ne*NUM_VAR + i*NUM_RVAR]>-1)
			{
				printf("Layout- R%i. Type %i: %.2f %.2f %.2f %.2f\n",i,layout[ne*NUM_VAR + i*NUM_RVAR + 1],layout[ne*NUM_VAR + i*NUM_RVAR + 2],layout[ne*NUM_VAR + i*NUM_RVAR + 3],layout[ne*NUM_VAR + i*NUM_RVAR + 4]);
			}
		}
			
	}
	*/
	/*

	float *region_pos=&layout[ne*NUM_VAR];
	memset(region_pos,-1,sizeof(float)*MAX_ELEMENTS*NUM_RVAR);

	for (int i=0;i < num_regions;i++)
	{

		layout[ne*NUM_VAR + i*NUM_RVAR]=region_type[i];
		layout[ne*NUM_VAR + i*NUM_RVAR + 1]=regions[i].l;
		layout[ne*NUM_VAR + i*NUM_RVAR + 2]=regions[i].r;
		layout[ne*NUM_VAR + i*NUM_RVAR + 3]=regions[i].b;
		layout[ne*NUM_VAR + i*NUM_RVAR + 4]=regions[i].t;

		if ((debug) and (thread_id==0))
			printf("Pre- R%i. Type %i: %.2f %.2f %.2f %.2f\n",i,region_type[i],regions[i].l,regions[i].r,regions[i].b,regions[i].t);
	}
	
	for (int i=num_regions;i < MAX_ELEMENTS;i++)
		layout[ne*NUM_VAR + i*NUM_RVAR]=-1;
	*/


	
	


	/****************************
	 *
	 * Calculate Features
	 *
	 ***************************/



	float *weights=params;

	double features[NUM_FEATURES];
	double nio_grads[NUM_FEATURES];
	//double nio_grads2[NUM_FEATURES];
	int features_ids[NUM_FEATURES];
	
	for (int i=0;i< NUM_FEATURES;i++)
	{
		features_ids[i]=-1;
		features[i]=0;
		params_grad[i]=0;
		params_grad[i + 2*NUM_FEATURES]=0;
		nio_grads[i]=NIO_DEFAULT;
		//nio_grads2[i]=NIO_DEFAULT;
	}



	int f_cnt=0;

	if ((eval_id==0)|| (!multithread))
	{	
		
		

	
		/*
		Box layout_regions[MAX_ELEMENTS*2];
	
		int num_layout_regions=0;
		for (int i=0;i<ne;i++)
		{
			if (layout[ne*NUM_VAR+i*NUM_RVAR]>-1)
			{
				num_layout_regions++;
				layout_regions[i].set(layout[offset+i*NUM_RVAR+1],  layout[offset+i*NUM_RVAR+1]+layout[offset+i*NUM_RVAR+3],layout[offset+i*NUM_RVAR+2], layout[offset+i*NUM_RVAR+2]+layout[offset+i*NUM_RVAR+4]);
			}
		}
	
	
		float region_overlap=0;
		for (int i=0;i < num_layout_regions;i++)
		{
			float region_area=0;
			for(int j=0;j<num_layout_regions;j++)
			{
				if (i!=j)
					region_area+=getBoxIntersection(layout_regions[j],layout_regions[i]).area();				
			}
			
			region_overlap+=region_area/layout_regions[i].area();
		}
	
		features[f_cnt]=region_overlap/(num_regions+0.0001);
		features_ids[f_cnt++]=REGION_OVERLAP_FEAT;
	
	
	
		//number of regions feature
		features[f_cnt]=float(num_layout_regions)/nv;
		features_ids[f_cnt++]=NUM_REGIONS_FEAT;
		*/
		
	
		/***************************
		 * 
		 * Scale Features
		 * 
		 **************************/
	
		float text_size_sum=0;
		float graphic_size_sum=0;
		float text_cons_sum=0;
		float graphic_cons_sum=0;
		
	
		for (int i=0;i<ne;i++)
		{
			if (!visible[i])
				continue;
				
			if (text_elements[i])
			{
				text_size_sum+=sizes[i];
				
				if (visible[i])
					text_cons_sum+=max(0.0,MIN_TEXT_SIZE-sizes[i])+ (int)(sizes[i]<MIN_TEXT_SIZE);

			}
			else
			{
				graphic_size_sum+=sizes[i];
				if (visible[i])
					graphic_cons_sum+=max(0.0,MIN_GRAPHIC_SIZE-sizes[i])+ (int)(sizes[i]<MIN_GRAPHIC_SIZE);
			}

		}
	
		float text_size_mean=text_size_sum/num_text;
	
	
		features[f_cnt]=text_cons_sum/num_text;
		features_ids[f_cnt++]=MIN_TEXT_SIZE_FEAT;
	

	
		atan_param=params[NUM_FEATURES+TEXT_SIZE_FEAT];
		atan_2_param=atan_params[NUM_FEATURES+TEXT_SIZE_FEAT];
	
		float text_size_feat=0;
		float text_size_grad1=0;
		
	
		float text_line_length_feat=0;
		//float text_line_length_grad1=0;
		
		//float line_atan_param=params[NUM_FEATURES+LINE_LENGTH_FEAT];
		//float line_atan_2_param=atan_params[NUM_FEATURES+LINE_LENGTH_FEAT];
		
		
		for (int i=0;i<ne;i++)
		{
			if (visible[i] && text_elements[i])
			{
				//atan_xy=getAtan(d,sizes[i]*atan_param);
				atan_xy=atan(sizes[i]*atan_param);
				text_size_feat+=atan_xy/atan_2_param;
				text_size_grad1+=atan_deriv(sizes[i],atan_param,atan_2_param,atan_xy);
				
	
				//float line_feat=(aspect_ratio[i]);
				text_line_length_feat+=aspect_ratio[i]*aspect_ratio[i];
				//float line_atan_xy=atan(line_feat*line_atan_param);
				
				//text_line_length_feat+=line_atan_xy/line_atan_2_param;
				//text_line_length_grad1+=atan_deriv(line_feat,line_atan_param,line_atan_2_param,line_atan_xy);
			}
		}
		
		
		
	
		features[f_cnt]=-1*text_size_feat/num_text;
		features_ids[f_cnt]=TEXT_SIZE_FEAT;
		nio_grads[f_cnt++]=text_size_grad1/num_text;
	
		features[f_cnt]=text_size_feat/num_text;
		features_ids[f_cnt]=TEXT_SIZE_REVERSE_FEAT;
		nio_grads[f_cnt++]=-1*text_size_grad1/num_text;
		
		features[f_cnt]=text_line_length_feat/num_text;
		features_ids[f_cnt++]=LINE_LENGTH_FEAT;
	
	
		float graphic_size_mean=0;
	
		if (num_graphic>0)
		{
			graphic_size_mean=graphic_size_sum/num_graphic;
	
			features[f_cnt]=graphic_cons_sum/num_graphic;
			features[f_cnt+1]=-1*graphic_size_mean;
			features[f_cnt+2]=graphic_size_mean;
		}
		else
		{
			features[f_cnt]=0;
			features[f_cnt+1]=0;
			features[f_cnt+2]=0;
		}
	
		features_ids[f_cnt]=MIN_GRAPHIC_SIZE_FEAT;
		features_ids[f_cnt+1]=GRAPHIC_SIZE_FEAT;
		features_ids[f_cnt+2]=GRAPHIC_SIZE_REVERSE_FEAT;
		f_cnt+=3;
	
	
	
	
	
	
	
	
		/**************************
		 *
		 * Importance features
		 *
		 **************************/
	
	
		 float text_imp_sum=0;
		 float graphic_imp_sum=0;
	
		 for (int i=0;i<ne;i++)
		 {
		 	if (!visible[i])
		 		continue;
		 		
			 if (text_elements[i])
				 text_imp_sum+=float(d->importance[i]);
			 else
				 graphic_imp_sum+=float(d->importance[i]);
		 }
		 float text_imp_mean=(text_imp_sum)/float(num_text);
		 float graphic_imp_mean=(graphic_imp_sum)/float(num_graphic+0.00001);
	
		 float text_size_var=0, graphic_size_var=0;
		 float text_imp_var=0,graphic_imp_var=0;
		 float text_both=0,graphic_both=0;
		 float text_xx=0, text_xy=0;
		 float graphic_xx=0, graphic_xy=0;
	
		 for (int i=0;i<ne;i++)
		 {
		 	 if (!visible[i])
		 	 	continue;
		 	 	
			 if (text_elements[i])
			 {
				 text_both+=(float(d->importance[i])-text_imp_mean)*(sizes[i]-text_size_mean);
				 text_imp_var+=((float(d->importance[i])-text_imp_mean)*((float(d->importance[i])-text_imp_mean)));
				 text_size_var +=(sizes[i]-text_size_mean)*(sizes[i]-text_size_mean);
	
				 text_xx+=(float(d->importance[i])/10.0)*(float(d->importance[i])/10.0);
				 text_xy+=(float(d->importance[i])/10.0)*sizes[i];
				 //if (thread_id==0)
				//	 printf("%i size %f, imp %f\n",i,sizes[i],float(d->importance[i]));
			 }
			 else
			 {
				 graphic_both+=(float(d->importance[i])-graphic_imp_mean)*(sizes[i]-graphic_size_mean);
				 
				 graphic_imp_var+=((float(d->importance[i])-graphic_imp_mean)*(float(d->importance[i])-graphic_imp_mean));
				 graphic_size_var+=((sizes[i]-graphic_size_mean)*(sizes[i]-graphic_size_mean));
	
	
				 graphic_xx+=sizes[i]*sizes[i];
				 graphic_xy+=float(d->importance[i])*sizes[i];
			 }
			 
		 }
	

	
		//printf("text size variance %f\n",text_size_var/num_text);
		if (num_text>1)
			features[f_cnt]=-text_size_var/num_text;
		else
			features[f_cnt]=0;
		features_ids[f_cnt++]=TEXT_SIZE_VAR_FEAT;
	
		if (num_graphic>1)
			features[f_cnt]=1000*graphic_size_var/num_graphic;
		else
			features[f_cnt]=0;
		features_ids[f_cnt++]=GRAPHIC_SIZE_VAR_FEAT;
	
		 float graphic_imp=0, text_imp=0;
		 if ((num_text>1) and (text_size_var!=0) and (text_imp_var!=0))
		 	 text_imp=text_both/(sqrt(text_size_var)*sqrt(text_imp_var));
		 else
			 text_imp=0;
	
		features[f_cnt]=-1*text_imp;
		features_ids[f_cnt++]=TEXT_IMPORTANCE_PEARSON_FEAT;
	
		if ((num_graphic>1) and (graphic_size_var!=0) and (graphic_imp_var!=0))
			graphic_imp=graphic_both/(sqrt(graphic_size_var)*sqrt(graphic_imp_var));
		else
			graphic_imp=0;
		features[f_cnt]=-1*graphic_imp;
		features_ids[f_cnt++]=GRAPHIC_IMPORTANCE_PEARSON_FEAT;
	
	
		/*
		if (num_text>1)
			features[f_cnt]=-1*(num_text/text_xx)*(text_xy/num_text);
		else
			features[f_cnt]=0;
		//if (thread_id==0)
		//	printf("text_xx %f text_xy %f b%f\n",text_xx,text_xy,(num_text/text_xx)*(text_xy/num_text));
	
		features_ids[f_cnt++]=TEXT_IMPORTANCE_REGRESSION_FEAT;
	
		if (num_graphic>1)
			features[f_cnt]=-1*(num_graphic/graphic_xx)*(graphic_xy/num_graphic);
		else
			features[f_cnt]=0;
	
		features_ids[f_cnt++]=GRAPHIC_IMPORTANCE_REGRESSION_FEAT;
		*/
	
		/**************************
		 *
		 * Positioning features
		 *
		 **************************/
		
		
		//only if 
		if (weights[TEXT_XPOS_FEAT]>=1)
		{
			//printf("calculating position features\n");
			
			float text_xpos_sum=0, graphic_xpos_sum=0;
			float text_ypos_sum=0,graphic_ypos_sum=0;

			
			for (int i=0;i<ne;i++)
			{
				if (!visible[i])
					continue;
	
				if (text_elements[i])
				{
					text_xpos_sum+=center_pos[i].x;
					text_ypos_sum+=center_pos[i].y;
				}
				else
				{
					graphic_xpos_sum+=center_pos[i].x;
					graphic_ypos_sum+=center_pos[i].y;
				}
				
		
			}
		
			float text_xpos_mean=text_xpos_sum/float(num_text+0.001);
			float text_ypos_mean=text_ypos_sum/float(num_text+0.001);
		
			float graphic_xpos_mean=graphic_xpos_sum/float(num_graphic+0.001);
			float graphic_ypos_mean=graphic_ypos_sum/float(num_graphic+0.001);
		
		
			float text_xpos_var=0, graphic_xpos_var=0;
			float text_ypos_var=0,graphic_ypos_var=0;
		
			for (int i=0;i<ne;i++)
			{
				if (!visible[i])
					continue;
					
				if (text_elements[i])
				{
					text_xpos_var+=(text_xpos_mean-center_pos[i].x)*(text_xpos_mean-center_pos[i].x);
					text_ypos_var+=(text_ypos_mean-center_pos[i].y)*(text_ypos_mean-center_pos[i].y);
				}
				else
				{
					graphic_xpos_var+=(graphic_xpos_mean-center_pos[i].x)*(graphic_xpos_mean-center_pos[i].x);
					graphic_ypos_var+=(graphic_ypos_mean-center_pos[i].y)*(graphic_ypos_mean-center_pos[i].y);
				}
				
			}
		
			text_xpos_var=10*text_xpos_var/(num_text+0.001);
			text_ypos_var=10*text_ypos_var/(num_text+0.001);
		
			graphic_xpos_var=10*graphic_xpos_var/(num_graphic+0.001);
			graphic_ypos_var=10*graphic_ypos_var/(num_graphic+0.001);
		
			//if (thread_id==0)
			//	printf("%f %f %f %f\n",text_xpos_var, text_ypos_var,graphic_xpos_var,graphic_ypos_var);
		
			features[f_cnt]=-1*text_xpos_mean;
			features_ids[f_cnt++]=TEXT_XPOS_FEAT;
			features[f_cnt]=-1*graphic_xpos_mean;
			features_ids[f_cnt++]=GRAPHIC_XPOS_FEAT;
			features[f_cnt]=-1*text_ypos_mean;
			features_ids[f_cnt++]=TEXT_YPOS_FEAT;
			features[f_cnt]=-1*graphic_ypos_mean;
			features_ids[f_cnt++]=GRAPHIC_YPOS_FEAT;
		
			features[f_cnt]=text_xpos_mean-1;
			features_ids[f_cnt++]=TEXT_XPOS_REVERSE_FEAT;
			features[f_cnt]=graphic_xpos_mean-1;
			features_ids[f_cnt++]=GRAPHIC_XPOS_REVERSE_FEAT;
			features[f_cnt]=text_ypos_mean-1;
			features_ids[f_cnt++]=TEXT_YPOS_REVERSE_FEAT;
			features[f_cnt]=graphic_ypos_mean-1;
			features_ids[f_cnt++]=GRAPHIC_YPOS_REVERSE_FEAT;
		
		
			features[f_cnt]=text_xpos_var;
			features_ids[f_cnt++]=TEXT_XPOS_VAR_FEAT;
			features[f_cnt]=graphic_xpos_var;
			features_ids[f_cnt++]=GRAPHIC_XPOS_VAR_FEAT;
			features[f_cnt]=text_ypos_var;
			features_ids[f_cnt++]=TEXT_YPOS_VAR_FEAT;
			features[f_cnt]=graphic_ypos_var;
			features_ids[f_cnt++]=GRAPHIC_YPOS_VAR_FEAT;	
		}
		

	
		/***********************
		 * 
		 * Overlap Features
		 * 
		 ***********************/
	
	
		Box overlap_regions[MAX_ELEMENTS];
		Box box;
		for (int i=0;i<d->num_overlap_regions;i++)
		{
	
			if (d->overlap_region_elem[i]>-1)
				box=elem_bb[d->overlap_region_elem[i]];
			else
				box=designBB;
	
			overlap_regions[i].l=box.l+ (box.r-box.l)*d->overlap_regions[i].l;
			overlap_regions[i].r=box.l+ (box.r-box.l)*d->overlap_regions[i].r;
			overlap_regions[i].b=box.b+ (box.t-box.b)*d->overlap_regions[i].b;
			overlap_regions[i].t=box.b+ (box.t-box.b)*d->overlap_regions[i].t;
		}

	
	
		float no_overlap_sum=0;
		
		for (int i=0;i<d->num_overlap_regions;i++)
		{
	
			if ((thread_id==0) and (debug))
				printf("region %i, id %i, %f %f %f %f\n", i,d->overlap_region_elem[i],overlap_regions[i].l,overlap_regions[i].r,overlap_regions[i].b,overlap_regions[i].t);
	
			for (int j=0;j<ne;j++)
			{
				if (d->overlap_region_elem[i]==j)
					continue;
	
				Box intersect=getBoxIntersection(elem_bb[j],overlap_regions[i]);
				if ((thread_id==0) and (debug))
					printf("%i %i, %f %f %f %f, intersect.area: %.2f\n",i,j,elem_bb[j].l, elem_bb[j].r, elem_bb[j].b, elem_bb[j].t, intersect.area());
	
				no_overlap_sum+=intersect.area()/(elem_bb[j].area());
			}
	
		}
	
	
	
		features[f_cnt]=no_overlap_sum/ne;
		features_ids[f_cnt++]=NO_OVERLAP_FEAT;
	
	
	
		float tt_overlap_sum=0;
		float gt_overlap_sum=0;
		float gg_overlap_sum=0;
	
		float overlap_area=0;
	
		for (int i=0;i<ne;i++)
		for (int j=i+1;j<ne;j++)
		{
			Box intersect=getBoxIntersection(elem_bb[i],elem_bb[j]);
			Box design_int=getBoxIntersection(designBB,intersect);
	
			float elem_area1=elem_bb[i].area();
			float elem_area2=elem_bb[j].area();
			float elem_area=min(elem_area1,elem_area2);
			float intersect_area=design_int.area();
			
			
			if (elem_area==0)
			{
				printf("wtf. element area %f. %f %f\n",elem_area,elem_area1,elem_area2);
				return ZERO_AREA_ERROR;
			}
			
			if (elem_area<=1)
				continue;
				
			overlap_area+=intersect_area;
			
			if (text_elements[i] && text_elements[j])
				tt_overlap_sum+=intersect_area/(elem_area);
			else if ((text_elements[i] && graphic_elements[j]) || (graphic_elements[i] && text_elements[j]))
				gt_overlap_sum+=intersect_area/(elem_area);
			else
				gg_overlap_sum+=intersect_area/(elem_area);
		}
		
		
	
		features[f_cnt]=gt_overlap_sum/(float(ne*ne)*0.5);
		features_ids[f_cnt++]=GRAPHIC_TEXT_OVERLAP_FEAT;
	
		features[f_cnt]=tt_overlap_sum/(float(ne*ne)*0.5);
		features_ids[f_cnt++]=TEXT_OVERLAP_FEAT;
	
		features[f_cnt]=gg_overlap_sum/(float(ne*ne)*0.5);
		features_ids[f_cnt++]=GRAPHIC_OVERLAP_FEAT;
	
		//Out of bounds features
		float text_oob_sum=0;
		float graphic_oob_sum=0;
		float inside_area=0;
	
		for (int i=0;i<ne;i++)
		{
			if (!visible[i])
				continue;
				
			Box intersect=getBoxIntersection(designBB,elem_bb[i]);
			float elem_area=elem_bb[i].area();
			
			if (elem_area==0)
			{
				printf("wtf. element area %f\n",elem_area);
				return ZERO_AREA_ERROR;
			}
			
			float indesign_area=intersect.area();
			inside_area+=indesign_area;
		
			if (text_elements[i])
				text_oob_sum+=1-indesign_area/elem_area;
			else
				graphic_oob_sum+=1-indesign_area/elem_area;
			
		}
	
	
		features[f_cnt]=graphic_oob_sum/float(num_graphic+0.01);
		features_ids[f_cnt++]=GRAPHIC_OUT_OF_BOUNDS_FEAT;
	
		features[f_cnt]=text_oob_sum/float(num_text+0.01);
		features_ids[f_cnt++]=TEXT_OUT_OF_BOUNDS_FEAT;
	
	
	
		/**************************
		 *
		 * White space features
		 *
		 **************************/
	
	
		//features[f_cnt]=-1*(1- (inside_area - overlap_area)/designBB.area());
		//features_ids[f_cnt++]=WHITESPACE_FEAT;
	
		//features[f_cnt]=(1- (inside_area - overlap_area)/designBB.area());
		//features_ids[f_cnt++]=WHITESPACE_REVERSE_FEAT;
	
	
	
		float spread_dist=0;
		float check_x=0;
		float check_y=0;
		int min_point=0;
		int min_element=0;
		for (int c=0;c < NUM_POINTS;c++)
		{		
			
			check_x=(spread_check_loc_x[c])*d->width;
			check_y=(spread_check_loc_y[c])*d->height;
			
			float min_dist=9999;
			int curr_min_element=-1;
			
			for (int i=0;i<ne;i++)
			{
				float d1=sqrt((check_x-elem_bb[i].l)*(check_x-elem_bb[i].l) + (check_y-elem_bb[i].b)*(check_y-elem_bb[i].b));
				float d2=sqrt((check_x-elem_bb[i].r)*(check_x-elem_bb[i].r) + (check_y-elem_bb[i].b)*(check_y-elem_bb[i].b));
				float d3=sqrt((check_x-elem_bb[i].l)*(check_x-elem_bb[i].l) + (check_y-elem_bb[i].t)*(check_y-elem_bb[i].t));
				float d4=sqrt((check_x-elem_bb[i].r)*(check_x-elem_bb[i].r) + (check_y-elem_bb[i].t)*(check_y-elem_bb[i].t));
				
			
				
				float mid_x=elem_bb[i].mid_x();
				float mid_y=elem_bb[i].mid_y();
				float d5=sqrt((check_x-mid_x)*(check_x-mid_x) + (check_y-mid_y)*(check_y-mid_y));
				
				
				float curr_min=	min(min(min(min(min(min_dist,d1),d2),d3),d4),d5);	
				
				if (curr_min<min_dist)		
				{
					min_dist=curr_min;
					curr_min_element=i;
					
				}
			}
			if ((thread_id==0) and (debug))
				printf("min distance to point %i is %f, curr_min element %i\n",c, min_dist,curr_min_element);

			spread_dist+=(min_dist/scale)*(min_dist/scale);
			/*
			if (min_dist>max_spread_dist)
			{
				max_spread_dist=min_dist;
				min_point=c;
				min_element=curr_min_element;
			}
			//max_spread_dist=max(max_spread_dist,min_dist);
			*/
		}	
		
		//if ((thread_id==0) and (debug))
		//	printf("min distance %f was element %i to point %i (%f %f)\n",max_spread_dist, min_element, min_point,spread_check_loc_x[min_point], spread_check_loc_y[min_point]);

			
		float spread_feat=((spread_dist/NUM_POINTS));
	
		features[f_cnt]=spread_feat;
		features_ids[f_cnt++]=SPREAD_FEAT;
	
	
	
		
		
		
	
		/**************************
		 *
		 * Pairwise Distance and Margins
		 *
		 **************************/
	
		float border_margins[MAX_ELEMENTS][2];
		float nearest_border_margin[MAX_ELEMENTS];
		float nearest_element_distance[MAX_ELEMENTS];
		//float nearest_text_distance[MAX_ELEMENTS];
	
	
		for (int i=0;i<ne;i++)
		{
	
			float min_dist=999;
			float min_text_dist=999;
	
			for (int j=0;j<ne;j++)
			{
				if (i!=j)
				{
					float elem_dist=max(max(bb_distance[0][i][j], bb_distance[1][i][j]),0.0);
					min_dist=min(min_dist, elem_dist);
	
					if (text_elements[j])
						min_text_dist=min(min_text_dist, elem_dist);
				}
			}
	
			nearest_element_distance[i]=min_dist/scale;
			//nearest_text_distance[i]=min_text_dist/scale;
	
			border_margins[i][0]=min(elem_bb[i].l,d->width-elem_bb[i].r);
			border_margins[i][1]=min(elem_bb[i].b,d->height-elem_bb[i].t);
	
			nearest_border_margin[i]=min(border_margins[i][0],border_margins[i][1])/scale;
	
		}
	
		atan_param=params[NUM_FEATURES+PAIRWISE_DIST_AVG_FEAT];
		atan_2_param=atan_params[NUM_FEATURES+PAIRWISE_DIST_AVG_FEAT];
	
	
	
		float pairwise_dist_min=999;
		float pairwise_dist_sum=0;
		float pairwise_dist_grad=0;
		float text_margin_min=999;
		//float text_margin_sum=0;
		float graphic_margin_min=999;
		//float graphic_margin_sum=0;
		//float text_dist_sum=0;
	
	
		for (int i=0;i<ne;i++)
		{
			if (!visible[i])
				continue;
	
			pairwise_dist_min=min(pairwise_dist_min,nearest_element_distance[i]);

			atan_xy=atan(nearest_element_distance[i]*atan_param);
			pairwise_dist_sum+=1-atan_xy/atan_2_param;
	
			//pairwise_dist_grad+=-1*atan_deriv(nearest_element_distance[i],atan_param,atan_2_param,atan_xy);
	
			if (text_elements[i])
			{
				text_margin_min=min(text_margin_min,nearest_border_margin[i]);
				//text_margin_sum+=nearest_border_margin[i];
				//text_dist_sum+=nearest_text_distance[i];
			}
			else
			{
				graphic_margin_min=min(graphic_margin_min,nearest_border_margin[i]);
				//graphic_margin_sum+=nearest_border_margin[i];
			}
		}
	
	
		features[f_cnt]=pairwise_dist_sum/nv;
		nio_grads[f_cnt]=pairwise_dist_grad/nv;
		features_ids[f_cnt++]=PAIRWISE_DIST_AVG_FEAT;
	
		//features[f_cnt]=0;
		//features_ids[f_cnt++]=PAIRWISE_DIST_MIN_FEAT;
	
		setFeature(PAIRWISE_DIST_MIN_FEAT,f_cnt,pairwise_dist_min,1,params,atan_params,features,features_ids,nio_grads)
	
	
	
		//float text_margin_avg=text_margin_sum/float(num_text);
		//float graphic_margin_avg=graphic_margin_sum/float(num_graphic);
		//features[f_cnt]=0;
		//features_ids[f_cnt++]=TEXT_MARGIN_DIST_AVG_FEAT;
		//features[f_cnt]=0;
		//features_ids[f_cnt++]=GRAPHIC_MARGIN_DIST_AVG_FEAT;

	
		setFeature(TEXT_MARGIN_DIST_MIN_FEAT,f_cnt,text_margin_min,1,params,atan_params,features,features_ids,nio_grads)
		setFeature(GRAPHIC_MARGIN_DIST_MIN_FEAT,f_cnt,graphic_margin_min,1,params,atan_params,features,features_ids,nio_grads);
		//setFeature(TEXT_DIST_FEAT,f_cnt,text_dist_sum/num_text,1,params,atan_params,features,features_ids,nio_grads);
	
	
		/*
		atan_param=params[NUM_FEATURES+GRAPHIC_MARGIN_DIST_MIN_FEAT];
		atan_2_param=atan_params[NUM_FEATURES+GRAPHIC_MARGIN_DIST_MIN_FEAT];
		atan_xy=atan(graphic_margin_min*atan_param);
		features[f_cnt]=1-atan_xy/atan_2_param;
		features_ids[f_cnt]=GRAPHIC_MARGIN_DIST_MIN_FEAT;
		nio_grads[f_cnt++]=-1*atan_deriv(features[f_cnt],atan_param,atan_2_param,atan_xy);
		*/
	
		//if (thread_id==0)
		//	printf("test %.2f %.2f %.2f %i %.2f\n",graphic_margin_min,atan_param,features[f_cnt-1],features_ids[f_cnt-1],weights[features_ids[f_cnt-1]]*atan_param*nio_grads[f_cnt-1]);
	
	
	
		
		/**************************
		 *
		 * Orig layout features
		 *
		 **************************/
	
	
		
		if ((d->check_layout_exists) and (!(d->fixed_regions)))
		{
			
			//need to set the distances from the selected element
			if (d->check_layout_distances[0]==-1)
			{
				
				
				int selected=-1;
				for (int i=0;i < ne;i++)
				{
					d->check_layout_distances[i]=0;
					if ((abs(d->check_layout[i*NUM_VAR+4]-SELECTED_FIX)<0.01) || (abs(d->check_layout[i*NUM_VAR+4]-SELECTED_NOFIX)<0.01) )
						selected=i;
					
				}
				
				if (selected>-1)
				{
					for (int i=0;i < ne;i++)
					{
						float x_dist=max(bb_distance[0][selected][i],0.0);
						float y_dist=max(bb_distance[1][selected][i],0.0);
						
						float selected_dist=(min(x_dist,y_dist))/dist_scale;						
						
						d->check_layout_distances[i]=selected_dist;
						//printf("distance from element %i to selected element %i: %.3f\n",i,selected,selected_dist);
					}
				}
				
			}
			
			
			if ((thread_id==0) and (debug))
			{
				for (int i=0;i < ne;i++)
					printf("check layout distance from %i %.3f\n",i,d->check_layout_distances[i]);
			}
			
			
			
			
			float relative_diff=0;
			/*	
			for (int i=0;i<ne;i++)
			{
				if((d->check_layout[i*NUM_VAR+4]==1) && ((layout[i*NUM_VAR]!=d->check_layout[i*NUM_VAR])||(layout[i*NUM_VAR+1]!=d->check_layout[i*NUM_VAR+1])||(layout[i*NUM_VAR+2]!=d->check_layout[i*NUM_VAR+2])))
				{
					printf("Possible error. Layout  vs check layout (%f,%f,%f) (%f,%f,%f)\n ",i,layout[i*NUM_VAR],layout[i*NUM_VAR+1],layout[i*NUM_VAR+2],d->check_layout[i*NUM_VAR],d->check_layout[i*NUM_VAR+1],d->check_layout[i*NUM_VAR+2]);
				
					//return CHECK_LAYOUT_ERROR;
				}
			}
			*/
			
			Box check_bbs[MAX_ELEMENTS];
			float2 check_center_pos[MAX_ELEMENTS];

			for (int i=0;i < ne;i++)
			{
				
				if (visible[i])
				{
					height=round(d->check_layout[i*NUM_VAR+2]*d->height);
					width=round(height/aspect_ratio[i]);
					xp=round(d->check_layout[i*NUM_VAR]*d->width);
					yp=round(d->check_layout[i*NUM_VAR+1]*d->height);
		
					check_bbs[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
				
					check_center_pos[i].x=((check_bbs[i].l+check_bbs[i].r)/2.0)/float(d->width);
					check_center_pos[i].y=((check_bbs[i].t+check_bbs[i].b)/2.0)/float(d->height);
				
				}
			}
			
				
			
			for (int i=0;i < ne;i++)
			{
				
				if ((d->check_layout[i*NUM_VAR+4]<1) && (visible[i]))
				{
					float h_diff1=0,h_diff=0;
					float pos_diff=0;
					
					
					float diff_x1=0,diff_x2=0,diff_y1=0,diff_y2=0;
					for (int j=0;j < ne;j++)
					{
						diff_x1=max(check_center_pos[i].x-check_center_pos[j].x-0.03,0.0);
						diff_x2=max(center_pos[i].x-center_pos[j].x-0.03,0.0);
						
						diff_y1=max(check_center_pos[i].y-check_center_pos[j].y-0.03,0.0);
						diff_y2=max(center_pos[i].y-center_pos[j].y-0.03,0.0);
						
						
						//if ((thread_id==0) && (debug) && (i>=1)&&(j>=1))
						//	printf("i %i j %i, center check x (%.3f - %.3f=%.3f) , center curr x  (%.3f - %.3f=%.3f)\n",i,j,check_center_pos[i].x,check_center_pos[j].x,diff_x1,center_pos[i].x,center_pos[j].x,diff_x2);
							
						
						
						if (((abs(diff_x1-diff_x2)>0.001) && (diff_x1*diff_x2<=0)) || ((abs(diff_y1-diff_y2)>0.001) && (diff_y1*diff_y2<=0)))
						{
							relative_diff++;
							
							if ((thread_id==0) && (debug) && (i>=1)&&(j>=1))
								printf("relative diff err");
							
						}	
						
					}
	
					Box check_bb=check_bbs[i];
					
					int nl=0;
					int alt=int(layout[NUM_VAR*i+6]); 

					if ((d->num_alt[i]>0))
						nl=d->alt_num_lines[i*MAX_ALT+alt];
					else
						nl=d->num_lines[i];	
					
					float check_size=0;
					if (text_elements[i])
						check_size=10.0*(((check_bb.t-check_bb.b)/nl)/(400));
					else
					{
						Box intersect=getBoxIntersection(designBB,check_bb);
						check_size=(intersect.t-intersect.b)/ (400);
					}
					
					
					
					float2 check_pos;
					
					check_pos.x=((check_bb.l+check_bb.r)/2.0)/float(d->width);
					check_pos.y=((check_bb.t+check_bb.b)/2.0)/float(d->height);
					
					
					h_diff1=abs(check_size-sizes[i]) ;					
					//h_diff2=abs(((elem_bb[i].t-elem_bb[i].b) -  (check_bb.t-check_bb.b))/d->height) ;
					h_diff=h_diff1;
		
					pos_diff= sqrt((center_pos[i].x-check_pos.x)*(center_pos[i].x-check_pos.x) + (center_pos[i].y-check_pos.y)*(center_pos[i].y-check_pos.y));

				
					float pos_weight=0,height_weight=0;
					if ((d->check_layout[i*NUM_VAR+4]>0))
					{
						pos_weight=d->check_layout[i*NUM_VAR+4]*2;
						height_weight=d->check_layout[i*NUM_VAR+4]*2;	
					}
					
					if (d->check_layout[i*NUM_VAR+4]==SELECTED_NOFIX)
					{
						h_diff=0;
						pos_diff=0;
					}

					
						
					float alt_diff=0;
					
					
					if ((d->num_alt[i]>0) &&  (layout[NUM_VAR*i+6]>-1) && (d->check_layout[NUM_VAR*i+6]>-1))
						alt_diff=abs(layout[NUM_VAR*i+6]-d->check_layout[i*NUM_VAR+6]);
							
					if ((thread_id==0)&&(debug))
						printf("pos diff element %i, (bb pos %.1f %.1f) layout %f, checklayout %f, alt_diff %f, nl %i, layout align %.1f, check layout align %.1f \n",i,elem_bb[i].l, elem_bb[i].b,layout[NUM_VAR*i+6],d->check_layout[i*NUM_VAR+6],alt_diff,nl,layout[NUM_VAR*i+3],d->check_layout[NUM_VAR*i+3]);
		
					
					
					if (nl>0)
					{
						int check_align=0;
						if (d->check_layout[NUM_VAR*i+3]==-1.0)
							check_align=(int) d->alignment[i];
						else
							check_align=(int) d->check_layout[NUM_VAR*i+3];
					
						alt_diff+=abs(internal_alignment[i]-check_align);
						
						if ((thread_id==0)&&(debug))
							printf("alt diff element %i: %.1f, check align %i, internal align %i, layout %.1f, check_layout %.1f \n",i,alt_diff,check_align,internal_alignment[i],layout[NUM_VAR*i+3],d->check_layout[NUM_VAR*i+3] );
				
					}				
					
					
					//float fix_pos_weight=max(0.0,d->check_layout_distances[i])*2;
					//float fix_height_weight=max(0.0,d->check_layout_distances[i])*2;
					
					float fix_pos_weight=0;
					float fix_height_weight=0;	
						
					features[f_cnt]= pos_weight*max(pos_diff-0.00,0.0) + fix_pos_weight*pos_diff;
					features_ids[f_cnt++]=ELEMENT_POSITION_DIFF_FEATS+i;
		
					features[f_cnt]=height_weight*(max(h_diff-0.0,0.0)+(alt_diff/5)) + fix_height_weight*h_diff;
					features_ids[f_cnt++]=ELEMENT_HEIGHT_DIFF_FEATS+i;
					
					if ((thread_id==0) and (debug))
						printf("elem %i, pos diff %f h diff %f, fixed_amount %f, feat %f, loc %i\n",i,pos_diff,h_diff,layout[i*NUM_VAR+4],features[f_cnt-2],f_cnt-2 );
					
				}
				/*
				if (d->fixed_regions)
				{
					float region_area=0;
					for(int j=0;j<num_regions;j++)
					{
						if (d->type[i]==layout[ne*NUM_VAR+j*NUM_RVAR])
						{
							region_area+=getBoxIntersection(regions[j],elem_bb[i]).area();
						}				
					}
					
					outside_regions+=max(1-region_area/elem_bb[i].area(),0.0);
				}
				*/
			}
			

			
			features[f_cnt]=relative_diff/(nv*nv);
			
			if ((thread_id==0) && (debug))
				printf("relative diff sum %f %f\n",relative_diff,features[f_cnt]);
				
			features_ids[f_cnt++]=RELATIVE_DIFF_FEAT;
			
		}
		
		
		/*
		else
		{
			
			if (d->fixed_regions)
			{
				
				for (int i=0;i < ne;i++)
				{
					if (layout[i*NUM_VAR+4]>FIX_LAYOUT_THRESH)
						continue;
					
					float region_area=0;
					for(int j=0;j<num_regions;j++)
					{
						if (d->type[i]==layout[ne*NUM_VAR+j*NUM_RVAR])
						{
							region_area+=getBoxIntersection(regions[j],elem_bb[i]).area();		
						}
					}
					
					outside_regions+=max(1-region_area/elem_bb[i].area(),0.0);
				}
		
				features[f_cnt]=outside_regions/nv;
				features_ids[f_cnt++]=ELEMENT_REGION_DIFF_FEAT;
				
				
			}
			else
			{
				features[f_cnt]=0;
				features_ids[f_cnt++]=ELEMENT_REGION_DIFF_FEAT;		
		
			}
			
		
			
			for (int i=0;i < MAX_ELEMENTS;i++)
			{
				features[f_cnt]=0;
				features_ids[f_cnt++]=ELEMENT_POSITION_DIFF_FEATS+i;
				features[f_cnt]=0;
				features_ids[f_cnt++]=ELEMENT_HEIGHT_DIFF_FEATS+i;
			}
			
			
		}
	

		features[f_cnt]=empty_regions;
		features_ids[f_cnt++]=EMPTY_REGION_FEAT;
	
		/**************************
		 * Previous Layout features
		 **************************/
		
	
		float prev_diff_feat=0;
	
		if (num_prev_layout>0)
		{
			float alpha=0.8;
			float alpha_sum=0;
			
			float max_diff=0;
			for (int n=0;n < num_prev_layout;n++)
			{
			
				float prev_diff=0;
				for (int i=0;i<ne;i++)
				{
					prev_diff+=abs(previous_layout[n*d->layout_size+i*NUM_VAR] - layout[i*NUM_VAR]);
					prev_diff+=abs(previous_layout[n*d->layout_size+i*NUM_VAR+1] - layout[i*NUM_VAR+1]);
					prev_diff+=2.0*abs(previous_layout[n*d->layout_size+i*NUM_VAR+2] - layout[i*NUM_VAR+2]);
				}
				float layout_diff=prev_diff/(ne);
				max_diff=max(max_diff,exp(-(layout_diff*layout_diff)*5.0));
				//prev_diff_feat=prev_diff_feat*alpha+exp(-(layout_diff*layout_diff)*5.0);
				//alpha_sum=alpha_sum*0.9+1;
			}	
			
			//prev_diff_feat=prev_diff_feat/alpha_sum;
			
			prev_diff_feat=max_diff;
			
			//printf("prev_diff_feat %f\n",prev_diff_feat);
		}
	
	
		features[f_cnt]=prev_diff_feat;
		features_ids[f_cnt++]=PREVIOUS_LAYOUT_FEAT;
	



		/**************************
		 * Hidden element features
		 **************************/
		features[f_cnt]=ne-nv;
		features_ids[f_cnt++]=HIDDEN_ELEM_FEAT;
	
		/**************************
		 *
		 * Symmetry features
		 *
		 **************************/
		float text_xsymm_sum=0;
		float graphic_xsymm_sum=0;
		float text_ysymm_sum=0;
		float graphic_ysymm_sum=0;
	
		float text_area_sum=0.001;
		float graphic_area_sum=0.001;
	

		Box intersect_x,intersect_x2;
		Box intersect_y,intersect_y2;
	
	
		Box flipped_x[MAX_ELEMENTS];
		Box flipped_y[MAX_ELEMENTS];	

		for (int i=0;i<ne;i++)
		{
			flipped_x[i].set(round(d->width-elem_bb[i].r), round(d->width-elem_bb[i].l),round(elem_bb[i].b), round(elem_bb[i].t));
			flipped_y[i].set(round(elem_bb[i].l), round(elem_bb[i].r),round(d->height-elem_bb[i].t), round(d->height-elem_bb[i].b));
		}
	
	
	
		for (int i=0;i<ne;i++)
		{
			//curr_region=regions[region_id[i]];
	
			//flipped.set(curr_region.l+ (curr_region.r-elem_bb[i].r), curr_region.l+ (curr_region.r-elem_bb[i].l),elem_bb[i].b, elem_bb[i].t);
			//intersect_x=getBoxIntersection(flipped,elem_bb[i]);
			if (!visible[i])
				continue;
	
			if (text_elements[i])
				text_area_sum+=elem_bb[i].area();
			else
				graphic_area_sum+=elem_bb[i].area();
	
	
			for (int j=i;j<ne;j++)
			{
				
				if ((visible[j])&&(text_elements[i]==text_elements[j]))
				{
	
					intersect_x=getBoxIntersection(flipped_x[j],elem_bb[i]);
					intersect_x2=getBoxIntersection(designBB,intersect_x);
	
					intersect_y=getBoxIntersection(flipped_y[j],elem_bb[i]);
					intersect_y2=getBoxIntersection(designBB,intersect_y);
	
					float scale=1;
					if (j!=i)
						scale=2.0;
					if (text_elements[i])
					{
						text_xsymm_sum+=scale*intersect_x2.area();
						text_ysymm_sum+=scale*intersect_y2.area();
					}
					else
					{
	
						graphic_xsymm_sum+=scale*intersect_x2.area();
						graphic_ysymm_sum+=scale*intersect_y2.area();
					}
				}
			}
		}
	
	
		float symm_feat1=graphic_xsymm_sum/graphic_area_sum-1;
		float symm_feat2=graphic_ysymm_sum/graphic_area_sum-1;
		float symm_feat3=text_xsymm_sum/text_area_sum-1;
		float symm_feat4=text_ysymm_sum/text_area_sum-1;
	
	
		if ((thread_id==0) && (debug))
		{
	
			atan_param=params[NUM_FEATURES+GRAPHIC_XSYMMETRY_FEAT];
			atan_2_param=atan_params[NUM_FEATURES+GRAPHIC_XSYMMETRY_FEAT];
			atan_xy=atan(symm_feat1*atan_param);
			printf("symm %f , feat %f, %f\n", graphic_xsymm_sum/graphic_area_sum, symm_feat1, atan_xy/atan_2_param+1);
		}
	
	
		setFeature(GRAPHIC_XSYMMETRY_FEAT,f_cnt,symm_feat1,2,params,atan_params,features,features_ids,nio_grads)
		setFeature(GRAPHIC_YSYMMETRY_FEAT,f_cnt,symm_feat2,2,params,atan_params,features,features_ids,nio_grads)
		setFeature(TEXT_XSYMMETRY_FEAT,f_cnt,symm_feat3,2,params,atan_params,features,features_ids,nio_grads)
		setFeature(TEXT_YSYMMETRY_FEAT,f_cnt,symm_feat4,2,params,atan_params,features,features_ids,nio_grads)
	
		//features[f_cnt]=-1*graphic_xsymm_sum/graphic_area_sum;
		//features_ids[f_cnt++]=GRAPHIC_XSYMMETRY_FEAT;
		//features[f_cnt]=-1*graphic_ysymm_sum/graphic_area_sum;
		//features_ids[f_cnt++]=GRAPHIC_YSYMMETRY_FEAT;
		//features[f_cnt]=-1*text_xsymm_sum/text_area_sum;
		//features_ids[f_cnt++]=TEXT_XSYMMETRY_FEAT;
		//features[f_cnt]=-1*text_ysymm_sum/text_area_sum;
		//features_ids[f_cnt++]=TEXT_YSYMMETRY_FEAT;
	
	
		if (weights[GRAPHIC_XSYMMETRY_REVERSE_FEAT]+weights[GRAPHIC_YSYMMETRY_REVERSE_FEAT]+weights[TEXT_XSYMMETRY_REVERSE_FEAT]+weights[TEXT_YSYMMETRY_REVERSE_FEAT]!=0)
		{
			features[f_cnt]=graphic_xsymm_sum/graphic_area_sum -1;
			features_ids[f_cnt++]=GRAPHIC_XSYMMETRY_REVERSE_FEAT;
			features[f_cnt]=graphic_ysymm_sum/graphic_area_sum -1;
			features_ids[f_cnt++]=GRAPHIC_YSYMMETRY_REVERSE_FEAT;
		
			features[f_cnt]=text_xsymm_sum/text_area_sum -1;
			features_ids[f_cnt++]=TEXT_XSYMMETRY_REVERSE_FEAT;
			features[f_cnt]=text_ysymm_sum/text_area_sum -1;
			features_ids[f_cnt++]=TEXT_YSYMMETRY_REVERSE_FEAT;
		}
		
	
		/*
		float symm_feat5=graphic_region_xsymm_sum/graphic_area_sum-1;
		float symm_feat6=text_region_xsymm_sum/text_area_sum-1;
	
		setFeature(GRAPHIC_REGION_XSYMMETRY_FEAT,f_cnt,symm_feat5,2,params,atan_params,features,features_ids,nio_grads)
		setFeature(TEXT_REGION_XSYMMETRY_FEAT,f_cnt,symm_feat6,2,params,atan_params,features,features_ids,nio_grads)
		*/
	
		/*
		float region_area_sum=0, region_xsymm_sum=0;
	
		for (int i=0;i<num_regions;i++)
		{
	
			region_area_sum+=regions[i].area();
			
			for (int j=i;j<num_regions;j++)
			{
				
					intersect_x=getBoxIntersection(regions_flipped_x[j],regions[i]);
					intersect_x2=getBoxIntersection(designBB,intersect_x);
	
					float scale=0;
					if (j!=i)
						scale=2.0;
					
					region_xsymm_sum+=scale*intersect_x2.area();
			
				
			}
		}
		
	
		features[f_cnt]=0;
		features_ids[f_cnt++]=GRAPHIC_REGION_XSYMMETRY_FEAT;
	
		features[f_cnt]=0;
		features_ids[f_cnt++]=TEXT_REGION_XSYMMETRY_FEAT;
		*/

		float graphic_region_symm_sum=0;
		float text_region_symm_sum=0;
		for (int i=0;i<ne;i++)
		{
			
			float min_left=elem_bb[i].l, min_right=d->width-elem_bb[i].r;
			
			//check if we overlap on y-axis, then update min amounts
			for (int j=0;j<ne;j++)
			{
				
				if (i==j)
					continue;
				float y_overlap=min((elem_bb[i].t-elem_bb[j].b),(elem_bb[j].t-elem_bb[i].b));
				
				if (y_overlap>0)
				{
					float left=elem_bb[i].l-elem_bb[j].r;
					float right=elem_bb[j].l-elem_bb[i].r;
					
					if (left>0)
						min_left=min(min_left,left);
						
					if (right>0)
						min_right=min(min_right,right);		
				}
				
			}
			
			float reg_width=min_left+min_right;
			
			if (text_elements[i])
				text_region_symm_sum+=abs(min_left-min_right)/reg_width;
			else
				graphic_region_symm_sum+=abs(min_left-min_right)/reg_width;
			
	
	
		}
		
	
		features[f_cnt]=graphic_region_symm_sum/(num_graphic+0.001)-1;
		features_ids[f_cnt++]=GRAPHIC_REGION_XSYMMETRY_FEAT;
	
		features[f_cnt]=text_region_symm_sum/(num_text+0.001)-1;
		features_ids[f_cnt++]=TEXT_REGION_XSYMMETRY_FEAT;
		


	
		/**************************
		 *
		 * Flow features
		 *
		 **************************/
	
		float text_xflow_sum=0;
		float text_yflow_sum=0;
		float text_diag_flow_sum=0;
		float text_xflow_cnt=0;
		float text_yflow_cnt=0;
		float text_diag_flow_cnt=0;
	
		float x_diff, y_diff;
		float x_imp_dist, y_imp_dist;
	
		float em1_cent;
		for (int i=0;i<ne;i++)
		{
	
			float imp_i=d->importance[i];
			//if (d->group_id[i]>-1)
			//	imp_i=group_imp[d->group_id[i]];
	
			if ((visible[i]) && (text_elements[i]))
			{
				em1_cent=center_pos[i].x;
	
				for (int j=i+1;j<ne;j++)
				{
					if ((!text_elements[j]) || (!visible[j]))
						continue;
	
					float imp_j=d->importance[j];
					//if (d->group_id[j]>-1)
					//	imp_j=group_imp[d->group_id[j]];
	
					float left_diff=(elem_bb[i].l - elem_bb[j].l)/d->width;
					float cent_diff=em1_cent - center_pos[j].x;
				
					if (abs(cent_diff) < abs(left_diff))
						x_diff=cent_diff;
					else
						x_diff=left_diff;
	
					y_diff= (elem_bb[i].b - elem_bb[j].b)/d->height;
	
					if ( x_diff < -0.01)
						x_imp_dist=max(imp_j-imp_i,0.0);
					else if ( x_diff > 0.01)
						x_imp_dist=max(imp_i-imp_j,0.0);
					else
						x_imp_dist=0;
	
	
					if (y_diff <  -0.01)
						y_imp_dist=max(imp_j-imp_i,0.0);
					else if ( y_diff > 0.01)
						y_imp_dist=max(imp_i-imp_j,0.0);
					else
						y_imp_dist=0;
	
					float pix_dist=(max(bb_distance[0][i][j]/d->width,bb_distance[1][i][j]/d->height))+0.1;
	
	
					if (x_imp_dist*pix_dist>0)
					{
	
						text_xflow_sum+=x_imp_dist;
						text_xflow_cnt++;
	
						if (((y_diff <=  0.01) and (imp_j > imp_i)) or ((y_diff >=  -0.01) and (imp_j< imp_i)))
						{
							text_diag_flow_sum+=x_imp_dist;
							//text_diag_flow_sum+=x_imp_dist*pix_dist;
							text_diag_flow_cnt+=1;
						}
	
					}
	
					if (y_imp_dist*pix_dist>0)
					{
						if ((debug) and (thread_id==0))
							printf("Flow Y: %i %i, y_diff %f,y_imp_dist %f, pix_dist %f, imps %f %f, group_id %i\n",i,j,y_diff,y_imp_dist,pix_dist, imp_i,imp_j,int(d->group_id[j]));
						
						text_yflow_sum+=y_imp_dist;
						//text_yflow_sum+=y_imp_dist*pix_dist;
						text_yflow_cnt+=1;
					}
				}
			}
		}
	
	
		features[f_cnt]=text_xflow_sum/(text_xflow_cnt+0.001);
		features_ids[f_cnt++]=TEXT_XFLOW_FEAT;
		features[f_cnt]=text_yflow_sum/(text_yflow_cnt+0.001);
		features_ids[f_cnt++]=TEXT_YFLOW_FEAT;
		features[f_cnt]=text_diag_flow_sum/(text_diag_flow_cnt+0.001);
		features_ids[f_cnt++]=TEXT_DIAG_FLOW_FEAT;
	



	}
	
	
	
	if ((eval_id==1) || (!multithread))
	{
		
		

		int group_count[MAX_ELEMENTS];
		int group_imp[MAX_ELEMENTS];
		for (int i=0;i<MAX_ELEMENTS;i++)
		{
			group_imp[i]=0;
			group_count[i]=0;
		}
	
		for (int i=0;i<ne;i++)
		{
			if (d->group_id[i]>-1)
			{
				group_count[d->group_id[i]]++;
				group_imp[d->group_id[i]]=max(group_imp[d->group_id[i]],d->importance[i]);
				
				if ((debug) && (thread_id==0))
					printf("group id %i has imp %i\n",d->group_id[i],group_imp[d->group_id[i]]);
			}
		}
			
		int num_groups=0;
		for (int i=0;i<ne;i++)
		{
			if (group_count[i]>1)
				num_groups++;
		}



		/**************************
		 *
		 * Alignment features
		 *
		 **************************/
		
		float align_dist[6][MAX_ELEMENTS][MAX_ELEMENTS];
		int aligned[6][MAX_ELEMENTS][MAX_ELEMENTS];
		

			
		
		

	
		float locations[6][MAX_ELEMENTS];
	
		bool fixed_alignment=true;
	
	
		for (int k=0;k<6;k++)
		{
			bool x_align=k<3;
	
	
			for (int i=0;i<ne;i++)
			{
				if (k==0)
					locations[k][i]=elem_bb[i].l/scale;
				else if (k==1)
					locations[k][i]=((elem_bb[i].l+elem_bb[i].r)/2.0)/scale;
				else if (k==2)
					locations[k][i]=elem_bb[i].r/scale;
				else if (k==3)
					locations[k][i]=elem_bb[i].b/scale;
				else if (k==4)
					locations[k][i]=((elem_bb[i].t+elem_bb[i].b)/2.0)/scale;
				else
					locations[k][i]=elem_bb[i].t/scale;
			}
	
			for (int i=0;i<ne;i++)
			for (int j=i+1;j<ne;j++)
			{
				aligned[k][i][j]=0;


				if ((x_align and (bb_distance[0][i][j]<0)) or (!x_align and (bb_distance[1][i][j]<0)))
				{

					float loc_diff=min(abs(locations[k][i]-locations[k][j]),0.99);
	
					
					
					
					fixed_alignment=true;
					if ((num_lines[i]>1)and (k<3) and (internal_alignment[i]!=k))
							fixed_alignment=false;

					if ((num_lines[j]>1)and (k<3) and (internal_alignment[j]!=k))
							fixed_alignment=false;

					if (!fixed_alignment)
						align_dist[k][i][j]=200;
					else
						align_dist[k][i][j]=loc_diff;
						
						
					//if ((thread_id==0) and (debug))
					//	printf("align dist k %i: %i %i. dist %.3f. internal %i %i\n",k,i,j,align_dist[k][i][j],internal_alignment[i],internal_alignment[j]);

				}
				else
					align_dist[k][i][j]=100;
			

			}
			
		}
	
	

	
		for (int i=0;i<ne;i++)
		{
			if (!visible[i])
				continue;
			
			for (int k=0;k<6;k++)
				aligned[k][i][i]=1;
	
	
			for (int j=i+1;j<ne;j++)
			{
				if (!visible[j])
					continue;
				
				float d0=align_dist[0][i][j];
				float d1=align_dist[1][i][j];
				float d2=align_dist[2][i][j];
				float d3=align_dist[3][i][j];
				float d4=align_dist[4][i][j];
				float d5=align_dist[5][i][j];
	
	
				if (min(min(d0,d1),d2)<ALIGN_THRESH)
				{
					if ((d0<d1) and (d0<d2))
						aligned[0][i][j]=1;
					else if ((d1<d0) and (d1<d2))
						aligned[1][i][j]=1;
					else
						aligned[2][i][j]=1;
		
				}
	
				if (min(min(d3 ,d4),d5)<ALIGN_THRESH)
				{
					if ((d3<d4) and (d3<d5))
						aligned[3][i][j]=1;
					else if ((d4<d3) and (d3<d5))
						aligned[4][i][j]=1;
					else
						aligned[5][i][j]=1;
				}
			}
		}
	
	
	
	
		int num_multi_line=0;
		for (int i=0;i<ne;i++)
			if ((num_lines[i]>1) &&(visible[i]))
				num_multi_line++;
		
	
		
		float align_xerr=0, align_yerr=0;
		int	err_idx;
		float denom=float(nv*nv -nv + num_multi_line)+0.0001;
		
		for (int k=0;k<6;k++)
		{
	
			float align_sum=0;
			for (int i=0;i<ne;i++)
			{
				if (!visible[i])
					continue;
					
				if ((num_lines[i]>1) and (internal_alignment[i]==k))
					align_sum+=1.0;
				
				for (int j=i+1;j<ne;j++)
				{
					if (aligned[k][i][j]>0)
					{
						align_sum+=2.0;
						
						err_idx=min(int(align_dist[k][i][j]*1000.0),999);
						if (k<3)
							align_xerr+=d->align_err[err_idx];
						else
							align_yerr+=d->align_err[err_idx];
							
						if ((thread_id==0) and (debug))
							printf("alignment k %i: %i %i. dist %.3f. idx %i, err %.3f \n",k,i,j,align_dist[k][i][j],err_idx,d->align_err[err_idx]);
					}
				}	
			}
			
			features[f_cnt]=-1*align_sum/denom;
			features_ids[f_cnt++]=ALIGN_XLEFT_FEAT+k;
			
		}
		
		if ((thread_id==0) and (debug))
			printf("align err x %.3f, y %.3f \n",align_xerr, align_yerr );	
		
		
		features[f_cnt]=align_xerr/(nv*nv*3);
		features_ids[f_cnt++]=ALIGN_XERROR_FEAT;
	
		features[f_cnt]=align_yerr/(nv*nv*3);
		features_ids[f_cnt++]=ALIGN_YERROR_FEAT;

	
		if ((debug)	&& (thread_id==0))
			printf("refining %i. num constraints %i\n",d->refine,d->num_constraints);
		
		
		//if (d->refine)
		//{
		float size_constraints_err=0;
		float alignment_constraints_err=0;
		float align_lines_err=0;
		
		for (int i=0;i<d->num_constraints;i++)
		{
			int elem=d->constraints[i*NUM_AVAR];
			int k=d->constraints[i*NUM_AVAR+1];
			int num_other=d->constraints[i*NUM_AVAR+2];
			
			//do alignment constraints later
			if (k<10)
			{
					
				if ((k<2)&& (num_lines[elem]>1) &&(d->fixed_alignment[elem]==0)  && (internal_alignment[elem]!=k))
				{
					align_lines_err+=1;	
					if ((debug)	&& (thread_id==0))
						printf("internal alignment error\n");				
				}
					
				
				for (int j=0;j<num_other;j++)
				{
					int other_elem=d->constraints[i*NUM_AVAR+3+j];
					float dist=abs(locations[k][elem]-locations[k][other_elem]);
					//if (dist>ALIGN_THRESH)
					align_lines_err+=dist;
						
						
					if ((debug)	&& (thread_id==0))
					{
						printf("alignment line dist %f, elem %i, k %i, other elem %i, internal align %i\n",dist, elem,k,other_elem,internal_alignment[other_elem]);
					}
						
						
					if ((k<2) &&(num_lines[other_elem]>1)&& (d->fixed_alignment[other_elem]==0) && (internal_alignment[other_elem]!=k))
					{
						align_lines_err+=1;	
						if ((debug)	&& (thread_id==0))
							printf("internal alignment error\n");
					}
					
				}
			}
			if (k==SIZE_CONSTRAINT)
			{
				for (int j=0;j<num_other;j++)
				{
					int other_elem=d->constraints[i*NUM_AVAR+3+j];
					
					float size_diff=abs(sizes[elem]-sizes[other_elem])/sizes[elem];
					float weight=d->check_layout[elem*NUM_VAR+4]*d->check_layout[elem*NUM_VAR+4];
					size_constraints_err+=weight*size_diff;
					
					if ((debug)	&& (thread_id==0) && (size_diff>0.001))
					{
						printf("constraint size error %i %i %.3f\n", elem, other_elem, size_diff);
					}
				}
			
			}
			if (k==ALIGN_CONSTRAINT)
			{
				
				float min_align_dist=999;
				for (int a=0;a<6;a++)
				{
					if ((a==2)|| (a==5))
						continue;
					bool is_aligned=true;
				
					float curr_align_dist=0;
					
					for (int j=0;j<num_other;j++)
					{
						int other_elem=d->constraints[i*NUM_AVAR+3+j];
						float a_dist=abs(locations[a][elem]-locations[a][other_elem]);
						curr_align_dist+= a_dist;
						if (a_dist>ALIGN_THRESH);
							is_aligned=false;
						
					}
					
					if ((debug)	&& (thread_id==0))
						printf("align constraint %i, type %i, %i,%.3f \n",i, a,is_aligned,curr_align_dist);
					
					
					if (is_aligned)
					{
					
						min_align_dist=0;
						break;
					}
					else
						min_align_dist=min(min_align_dist,curr_align_dist);		
				}
				
				alignment_constraints_err+=min_align_dist;
				
			}
		}
		
		features[f_cnt]=size_constraints_err;
		features_ids[f_cnt++]=SIZE_CONSTRAINTS_FEAT;
		
		features[f_cnt]=alignment_constraints_err;
		features_ids[f_cnt++]=ALIGN_CONSTRAINTS_FEAT;
		
		features[f_cnt]=align_lines_err;
		features_ids[f_cnt++]=ALIGN_LINES_FEAT;
	
		//}
	
		/**************************
		 *
		 * Group features
		 *
		 **************************/	
		if (num_groups>0)
		{
			
	
			float group_dist_sum=0;
			float group_align_sum=0;
			float group_align_x[MAX_ELEMENTS*MAX_ELEMENTS];
			float group_align_y[MAX_ELEMENTS*MAX_ELEMENTS];
			
			int group_align_x_cnt=0;
			int group_align_y_cnt=0;
		
		
			int group_members=0;
			
			for (int i=0;i<ne;i++)
			{
		
				if ((d->group_id[i]>-1) and (group_count[d->group_id[i]]>1))
				{
					group_members++;
					float group_alignment=0;
					
					
					int nearest=-1;
					float min_dist=999;
					for (int j=0;j<ne;j++)
					{
						if ((i!=j) and (d->group_id[i]==d->group_id[j]))
						{				
							for (int k=0;k<6;k++)
							{
								//  and (align_dist[k][i][j]<ALIGN_THRESH/3.0)
								if (aligned[k][min(i,j)][max(i,j)]>0)
								{
									group_alignment=1;
									
									if (k<3)
									{
										group_align_x[group_align_x_cnt]=k;
										group_align_x_cnt++;	
									}
									else
									{
										group_align_y[group_align_y_cnt]=k;
										group_align_y_cnt++;	
									}
								}
								
							}
							
							float elem_dist=max(max(bb_distance[0][i][j], bb_distance[1][i][j]),0.0)/dist_scale;
							
							if (elem_dist <min_dist)
							{
								min_dist=elem_dist;
								nearest=j;
							}
						}
					}
					group_align_sum+=group_alignment;
					
					
					if ((thread_id==0)and (debug))
						printf("Group Nearest Distance %i: %.3f %i\n",i,min_dist,nearest);
					
					
					if (min_dist==999)
					{
						printf("error. couldn't find a group member. element %i, group id %i\n", i,d->group_id[i]);
		
						for (int j=0;j<ne;j++)
						{
							printf("\t element %i, group id %i\n", j,d->group_id[j]);
		
							if ((i!=j) and (d->group_id[i]==d->group_id[j]))
							{
								float elem_dist=max(max(bb_distance[0][i][j], bb_distance[1][i][j]),0.0);
								printf("\t dist %.2f\n",elem_dist);
							}
						}
		
					}
		
					group_dist_sum+=min_dist;
				}
		
			}
		
			features[f_cnt]=group_dist_sum/(num_groups+0.0001);
			features_ids[f_cnt++]=GROUP_DIST_FEAT;
		
			features[f_cnt]=-1*group_align_sum/(group_members+0.0001);
			features_ids[f_cnt++]=GROUP_ALIGN_FEAT;
			
			
			//float group_align_var_sum=0;
			float group_text_mean_sum=0;
			float group_graphic_mean_sum=0;
			
			float group_text=0.001;
			float group_graphic=0.001;
			for (int i=0;i<ne;i++)
			{
				if (group_count[i]>1)
				{	
					if (text_elements[i])
					{
						group_text_mean_sum+=sizes[i];
						group_text++;	
					}
					else
					{
						group_graphic_mean_sum+=sizes[i];
						group_graphic++;
					}
				}
			}
			
			float group_text_mean=group_text_mean_sum/group_text;
			float group_graphic_mean=group_graphic_mean_sum/group_text;
		
			float group_text_var=0;
			float group_graphic_var=0;
			
			for (int i=0;i<ne;i++)
			{
				if (group_count[i]>1)
				{	
					if (text_elements[i])
						group_text_var+=(sizes[i]-group_text_mean)*(sizes[i]-group_text_mean);
					else
						group_graphic_var+=(sizes[i]-group_graphic_mean)*(sizes[i]-group_graphic_mean);
				}
			}
			
			
			features[f_cnt]=group_text_var/group_text;
			features_ids[f_cnt++]=GROUP_TEXT_SIZE_VAR_FEAT;
			
			features[f_cnt]=group_graphic_var/group_graphic;
			features_ids[f_cnt++]=GROUP_GRAPHIC_SIZE_VAR_FEAT;
		
		
		
			float mean_x_align=0,mean_y_align=0;
			
			for (int i=0;i< group_align_x_cnt;i++)
				mean_x_align+=group_align_x[i];
			mean_x_align=mean_x_align/float(group_align_x_cnt+0.001);
			
			for (int i=0;i< group_align_y_cnt;i++)
				mean_y_align+=group_align_y[i];
			mean_y_align=mean_y_align/float(group_align_y_cnt+0.001);
			
			float var_x_align=0,var_y_align=0;
			for (int i=0;i< group_align_x_cnt;i++)
				var_x_align+=(mean_x_align-group_align_x[i])*(mean_x_align-group_align_x[i]);
				
			for (int i=0;i< group_align_y_cnt;i++)
				var_y_align+=(mean_y_align-group_align_y[i])*(mean_y_align-group_align_y[i]);
			
			features[f_cnt]=var_x_align/float(group_align_x_cnt+0.001);
			features_ids[f_cnt++]=GROUP_ALIGN_X_VAR_FEAT;
		
			features[f_cnt]=var_y_align/float(group_align_y_cnt+0.001);
			features_ids[f_cnt++]=GROUP_ALIGN_Y_VAR_FEAT;
			
		}
		
		
		
		
	}

	
	
	
	if ((thread_id==0)and (debug))
		printf("Evaluating model");
	/**************************
	 * Evaluate model
	 **************************/
	
	
	double feat=0;
	double eval=0;
	
	if ((eval_id==0) || (!multithread))
		eval=-500;
	
	for (int i=0;i<f_cnt;i++)
	{
		
		//params_grad[features_ids[i]]=0;
		
		//if (features_ids[i]==-1)
		//{
		//	printf("ERROR\n, undefined for feature %i\n",i);
		//	continue;	
		//}

		if (features[i]==0)
		{
			//if (debug)
			//	printf(" %i feat test %i %f  %f %f\n",i,features_ids[i],0,weights[features_ids[i]],0);
			params_grad[features_ids[i]]=0;
			params_grad[features_ids[i]+NUM_FEATURES]=0;
			continue;
		}

		atan_param=params[NUM_FEATURES+features_ids[i]];

		if (nio_grads[i]==NIO_DEFAULT)
		{


			atan_2_param=atan_params[NUM_FEATURES+features_ids[i]];
			//atan_xy=getAtan(d,features[i]*atan_param);
			atan_xy=atan(features[i]*atan_param);
			feat=atan_xy/atan_2_param;

			if (calc_gradient)
				params_grad[features_ids[i]+NUM_FEATURES]=weights[features_ids[i]]*atan_deriv(features[i],atan_param,atan_2_param,atan_xy);

			
		}
		else
		{
			feat=features[i];
			
			if (calc_gradient)
			{
				params_grad[features_ids[i]+NUM_FEATURES]=weights[features_ids[i]]*nio_grads[i];
				//params_grad[features_ids[i]+2*NUM_FEATURES]=weights[features_ids[i]]*nio_grads2[i];
			}
			
		}
		
		//params_grad[features_ids[i]+2*NUM_FEATURES]=0;

		/*
		if ( (debug))
		{
			printf(" %i feat test %i %f %f %f\n",i,features_ids[i],feat,weights[features_ids[i]],feat*weights[features_ids[i]]);
			//for (int j=0;j<ne;j++)
			//{
			//	printf("element %i: l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f\n",j,elem_bb[j].l,elem_bb[j].r,elem_bb[j].b,elem_bb[j].t);
			//	printf("\t %f %f %f %f, alt: %f\n",layout[NUM_VAR*j],layout[NUM_VAR*j+1],layout[NUM_VAR*j+2],layout[NUM_VAR*j+3],layout[NUM_VAR*j+6]);
			//	return 99996;
			//}		
		}
		*/
		

		eval+=feat*weights[features_ids[i]];
		params_grad[features_ids[i]]=feat;


	}
	
	//printf("eval id %i, fcnt %i, eval %.3f\n",eval_id,f_cnt,eval);

	/*
	if ((thread_id==0) and (debug))
	{
		float test_eval=-500;
		printf("Model Feature Breakdown: %f\n",eval);
		for (int k=0;k<NUM_FEATURES;k++)
		{
			printf("%i\t%5.1f \t %4.2f\t %4.2f \t nl: %4.2f \n", k,params[k],params_grad[k],params[k]*params_grad[k],params[k+NUM_FEATURES]);
			test_eval+=params[k]*params_grad[k];
		}
		
		
		if (abs(test_eval-eval)>0.1)
			printf("error. test eval %f, eval %f\n",test_eval,eval);
	}
	*/

	/**************************
	 * Cleanup
	 **************************/


	if ((thread_id==0)and (debug))
		printf("finished debug with eval %f\n",eval*0.25);
	

	return eval*0.25;





}



void addParameterOffsets(char *filename, int num_params, float *h_params)
{
    FILE *fp = NULL;
	fp = fopen(filename, "r");

	if (!fp)
		return;


	char param_name[100];
	char val_str[100];
	float value;

	fgets(param_name,sizeof(param_name),fp);
	fgets(val_str,sizeof(val_str),fp);

	value = atof(val_str);
	printf("param_name: %s has value %f\n",param_name,value);

	if (!strcmp(param_name,"whitespace\n"))
	{
		if (value>=50)
			h_params[WHITESPACE_FEAT]+=(value-50)*5;
		else
			h_params[WHITESPACE_REVERSE_FEAT]+=(50-value)*5;


	}
	if (!strcmp(param_name,"text_size\n"))
	{
		if (value>=50)
			h_params[TEXT_SIZE_FEAT]+=(value-50)*5;
		else
			h_params[TEXT_SIZE_REVERSE_FEAT]+=(50-value)*5;

	}
	if (!strcmp(param_name,"graphic_size\n"))
	{
		if (value>=50)
			h_params[GRAPHIC_SIZE_FEAT]+=(value-50)*5;
		else
			h_params[GRAPHIC_SIZE_REVERSE_FEAT]+=(50-value)*5;
	}
	if (!strcmp(param_name,"symmetry\n"))
	{


	}

}


void saveParametersToFile(char *filename,float *params,int num_params)
{
	//printf("Saving parameters to file %s\n",filename);
	
	
	FILE *fp = fopen(filename, "w");
	
	fprintf(fp,"Weights,,,\n");
	
	for (int i=0;i< NUM_FEATURES;i++)
	{
		fprintf(fp,"%s,",feat_names[i]);
		for (int j=0;j<3;j++)
		{
			float fl=params[i+j*NUM_FEATURES];
			if (ceilf(fl) == fl)
				fprintf(fp,"%i",int(fl));
			else
				fprintf(fp,"%f",fl);
			if (j <2)
				fprintf(fp,",");
		}
		
		fprintf(fp,"\n",feat_names[i]);		
	}
	fclose(fp);
	
}


float *loadParametersFromFile(char * default_param_file, char *filename,int num_params)
{
	float *default_params=readParameters(default_param_file,num_params);
	float *params=readParameters(filename,num_params);
	
	for (int i=0;i<num_params;i++)
	{
		if (default_params[i]==-1)
		{
			printf("Error, default parameter %i is uninitialized \n",i);
			default_params[i]=0;
		}
	
		if (params[i]==-1)
			params[i]=default_params[i];
			
	}
	
	free(default_params);
	return params;
}

float *readParameters(char *filename,int num_params)
{
	printf("Loading parameters from file %s\n",filename);
	float *params = (float*)malloc( sizeof(float)*num_params);

	for(int i=0;i<num_params;i++)
		params[i]=-1;

	FILE *fp = fopen(filename, "r");

	if (!fp)
		return 0;

	char str[200];
	char param_name[100];

	while(fgets(str,sizeof(str),fp) != NULL)
    {
	   // strip trailing '\n' if it exists
	   int len = strlen(str)-1;
	   if(str[len] == '\n')
		  str[len] = 0;

	   char *comma = strstr(str, ",");


	   memset(param_name,0,100);
	   strncpy(param_name,str,comma-str);

	   comma=comma+sizeof(char);
	   float val=atof(comma);

	   char *next_val = strstr(comma, ",");
	   next_val=next_val+sizeof(char);
	   float val2=atof(next_val);

	   next_val = strstr(next_val, ",");
	   next_val=next_val+sizeof(char);
	   float val3=atof(next_val);

	   //printf("%s %.2f %.2f %.2f\n", param_name,val,val2,val3);

	   for (int i=0;i < NUM_FEATURES;i++)
	   {
		   if (!strcmp(param_name,feat_names[i]))
		   {
			   	params[i]=val;
			   	params[i+NUM_FEATURES]=val2;
			   	params[i+2*NUM_FEATURES]=val3;
			   /*
		   		if (val3>1)
		   		{
		   			
		   			float r=float(rand())/RAND_MAX;
		   			//printf ("%i, %f, %i, %f\n",rand(),float(rand()),RAND_MAX,r);
		   			params[i]=max(params[i]+(r-0.5)*val3,0.0);
		   			printf("setting param %s to %.3f\n",feat_names[i],params[i]);
		   		}
		   		*/
		   }
	   }
	}
	fclose (fp);

	printf("Finished loading parameters\n");


	return params;
}



	/*

	//create alignment groups
	for (int k=0;k<6;k++)
	{
		bool x_align=k<3;

		bool changed=true;

		while (changed)
		{
			changed=false;

			for (int i=0;i<ne;i++)
			for (int j=i+1;j<ne;j++)
			{
				if (aligned[k][i][j]==1)
				{

					for (int q=0;q<ne;q++)
					{
						if ((q==i) or (q==j))
							continue;

						if ((aligned[k][i][q] or aligned[k][q][i]) and (aligned[k][q][j]==0))
						{
							if ((x_align and bb_distance[1][j][q]>0) or ((not x_align) and bb_distance[0][j][q]>0) )
							{
								changed=true;
								aligned[k][q][j]=1;
								aligned[k][j][q]=1;
							}
						}
						if ((aligned[k][j][q] or aligned[k][q][j]) and (aligned[k][q][i]==0))
						{
							if ((x_align and bb_distance[1][i][q]>0) or ((not x_align) and bb_distance[0][i][q]>0) )
							{
								changed=true;
								aligned[k][q][i]=1;
								aligned[k][i][q]=1;
							}
						}
					}
				}
			}
		}
	}
	
	
	
	
	
	*/


	/*
	//take this code out eventually. only here for comparing with element 
	else
	{		
		num_regions=ne+d->num_overlap_regions;
		for (int i=0;i<ne;i++)
		{
			if (text_elements[i])
				region_type[i]=1;
			else
				region_type[i]=2;
				
			regions[i]=elem_bb[i];
	
		}
		for (int i=0;i<d->num_overlap_regions;i++)
		{
			region_type[i+ne]=2;
			regions[i+ne]=overlap_regions[i];
		}
	
	
		bool merge_complete=false;
		Box proposed;
	
		while (not merge_complete)
		{
	
			merge_complete=true;
	
			for (int i=0;i<num_regions;i++)
			for (int j=i+1;j<num_regions;j++)
			{
				if (region_type[i]!=region_type[j])
					continue;
	
				proposed.set(min(regions[i].l,regions[j].l), max(regions[i].r,regions[j].r),min(regions[i].b,regions[j].b), max(regions[i].t,regions[j].t));
	
				//try to merge i and j
				bool merge=true;
	
				for (int k=0;k<num_regions;k++)
				{
					if ((i==k) or (j==k))
						continue;
	
					if (not ((regions[k].l>=proposed.r) or (regions[k].r<=proposed.l) or \
							(regions[k].t<=proposed.b) or (regions[k].b>=proposed.t)))
					{
						merge=false;
						k=num_regions;
						break;
					}
				}
	
				//merge i and j
				if (merge)
				{
					//if ((debug) and (thread_id==0))
					//	printf("Merging %i and %i\n",i,j);
	
	
					regions[j]=regions[num_regions-1];
					region_type[j]=region_type[num_regions-1];
					regions[i]=proposed;
	
					merge_complete=false;
					num_regions--;
					i=MAX_ELEMENTS;
					j=MAX_ELEMENTS;
					break;
				}
			}
	
		}
	
	
	
	
	
	
		//bool left_region=false,bottom_region=false;
		float dist;
		bool overlap1, overlap2;
	
		//expand the boundaries of the regions to the edge of the designs
		for (int i=0;i<num_regions;i++)
		{
			float nearest_dist[4];
			int nearest_idx[4];
	
			for (int dir=0;dir<4;dir++)
			{
				nearest_dist[dir]=9999;
				nearest_idx[dir]=-1;
			}
	
			for (int dir=0;dir<4;dir++)
			{
	
				for (int j=0;j<num_regions;j++)
				{
					if (i==j)
						continue;
	
					if 	(dir==0)
						dist=regions[i].l-regions[j].r;
					if (dir==1)
						dist=regions[j].l-regions[i].r;
					if (dir==2)
						dist=regions[i].b-regions[j].t;
					if (dir==3)
						dist=regions[j].b-regions[i].t;
	
					if ((dist>=0) and (dist<nearest_dist[dir]))
					{
						nearest_dist[dir]=dist;
						nearest_idx[dir]=j;
					}
				}
	
				//expand to the boundary if there's nothing else
				if ((nearest_idx[dir]==-1) and (nearest_dist[dir]>0))
				{
					if ((debug) and (thread_id==0))
						printf("shifting region %i, boundary %i\n", i,dir);
	
					if 	(dir==0)
						regions[i].l=0;
					if (dir==1)
						regions[i].r=d->width;
					if (dir==2)
						regions[i].b=0;
					if (dir==3)
						regions[i].t=d->height;
				}
				//push boundaries to halfway point between nearest
				else
	
				{
	
	
					float cut=0;
					if 	(dir==0)
					{
	
						regions[i].l=regions[i].l-nearest_dist[dir]/2.0;
						regions[nearest_idx[dir]].r=regions[i].l;
						cut=regions[i].l;
					}
					else if (dir==1)
					{
						regions[i].r=regions[i].r+nearest_dist[dir]/2.0;
						regions[nearest_idx[dir]].l=regions[i].r;
						cut=regions[i].r;
					}
					else if (dir==2)
					{
						regions[i].b=regions[i].b-nearest_dist[dir]/2.0;
						regions[nearest_idx[dir]].t=regions[i].b;
						cut=regions[i].b;
					}
					else if (dir==3)
					{
						regions[i].t=regions[i].t+nearest_dist[dir]/2.0;
						regions[nearest_idx[dir]].b=regions[i].t;
						cut=regions[i].t;
					}
	
					if ((debug) and (thread_id==0))
						printf("merging boundary region %i and %i, boundary %i, %f \n", i,nearest_idx[dir],dir,cut);
	
				}
			}
	
		}
	
		Box proposed1;
		Box proposed2;
		//bool corner=false;
	
		//expand filled regions into remaining empty space
		for (int i=0;i<num_regions;i++)
		{
			for (int k=0;k<num_regions;k++)
			{
				if (k==i)
					continue;
	
				///if (((regions[i].l==regions[k].r) and (regions[i].t==regions[k].b)) or ((regions[k].l==regions[i].r) and (regions[k].t==regions[i].b)))
				//if (((regions[i].l==regions[k].r) or (regions[i].r==regions[k].l)) and ((regions[i].t==regions[k].b) or (regions[i].b==regions[k].t)))
	
				//check corners
				//corner=false;
				if (((regions[i].l==regions[k].r) or (regions[i].r==regions[k].l)) and ((regions[i].t==regions[k].b) or (regions[i].b==regions[k].t)))
				{
					proposed1.set(regions[k].l,regions[k].r, regions[i].b,regions[i].t);
					proposed2.set(regions[i].l,regions[i].r, regions[k].b,regions[k].t);
	
					overlap1=false;
					overlap2=false;
					for (int j=0;j < num_regions;j++)
					{
						if ((j!=i)and (j!=k))
						{
							overlap1= overlap1 or anyBoxIntersection(regions[j], proposed1);
							overlap2= overlap2 or anyBoxIntersection(regions[j], proposed2);
						}
					}
					if (not overlap1)
					{
						
						//if (regions[i].l < regions[k].l)
						//	regions[k].l=regions[i].l;
						//else
						//	regions[i].l=regions[k].l;
						
						if (regions[i].b < regions[k].b)
							regions[k].b=regions[i].b;
						else
							regions[i].b=regions[k].b;
					}
	
					if (not overlap2)
					{
	
						if (regions[i].b < regions[k].b)
							regions[k].t=regions[i].t;
						else
							regions[i].t=regions[k].t;
						
						//if (regions[i].l < regions[k].l)
						//	regions[i].r=regions[k].r;
						//else
						//	regions[k].r=regions[i].r;
						
					}
				}
			}
		}
	

		for (int i=0;i<ne;i++)
		{
			float overlap_area=0;
	
			for (int j=0;j<num_regions;j++)
			{
				if (anyBoxIntersection(regions[j], elem_bb[i]))
				{
					float area=getBoxIntersection(regions[j], elem_bb[i]).area();
					if (area > overlap_area)
					{
						overlap_area=area;
						region_id[i]=j;
					}
				}
			}
		}
		
	}
	*/


