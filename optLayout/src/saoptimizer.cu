#include "hip/hip_runtime.h"
#include "saoptimizer.cuh"



#define CURR_LOCATION_SHIFT 0
#define GLOBAL_LOCATION_SHIFT 1
#define CURR_HEIGHT_SHIFT 2
#define GLOBAL_ALIGN 3
#define ALIGN_ELEMENT 4
#define SWITCH_ALIGNMENT 5
#define SWITCH_ALTERNATE 6
#define FLIP_TWO_ELEMENTS 7
#define SHIFT_ALIGNED_ELEMENTS 8
#define SCALE_TYPE 9

#define NUM_PROPOSALS 16
//__device__ const int prop_choice[NUM_PROPOSALS] = {0};
__device__ const int prop_choice[NUM_PROPOSALS] = {0,0,1,1,2,2,3,4,4,5,6,6,7,8,8,9};

#define NUM_REFINE_PROPOSALS 12
//__device__ const int refine_prop_choice[NUM_REFINE_PROPOSALS] = {0};
__device__ const int refine_prop_choice[NUM_REFINE_PROPOSALS] = {0,0,1,2,2,4,4,5,6,8,8,9};
//__device__ const int refine_prop_choice[NUM_PROPOSALS] = {0,0,1,1,2,2,3,4,4,5,6,7,8,9};

#define REG_ELEMENT_SWITCH 0
#define REG_ELEMENT_HEIGHT 1
#define REG_ALIGN 2
#define REG_ORIENTATION 3
#define REG_CURR_LOCATION_SHIFT 4
#define REG_GLOBAL_LOCATION_SHIFT 6
#define REG_BB_SHIFT 7
#define REG_ELEMENT_ORDER 8
#define REG_SWAP 9
#define REG_MERGE 10
#define REG_SPLIT 11
#define REG_ALIGN_TWO 12
#define REG_ELEMENT_SPLIT 13
#define REG_SWITCH_ALTERNATE 14


#define NUM_FIXED_REGION_PROPOSALS 9
#define NUM_REGION_PROPOSALS 18

//__device__ const int fix_reg_prop_choice[NUM_FIXED_REGION_PROPOSALS] = {1,1,2,4,4,6,6,12,12};
//__device__ const int reg_prop_choice[NUM_REGION_PROPOSALS] = {1,1,2,4,4,6,6,12,12, 0,0,3,7,7,10,11,13,14};


#define MAX_PROPOSAL_TRIES 5



#include "rand.cuh"

/*

//, float *opt_layout
__global__ void simulatedAnnealing(Design *d, int num_iter,float start_temp, float end_temp, float *init_layouts,float *proposals,  float *params,  float *params_grads, float *eval, int *return_status, float *opt_layouts)
{

	*return_status=-1;

	//Todo: set these with some random numbers from the GPU
	z1=200*blockIdx.x ;
	z2=200*blockIdx.y ;
	z3=200*threadIdx.x;
	z4=200*threadIdx.y;

	int thread_id=blockIdx.x;

	//access the memory for this thread
	float *init_layout=init_layouts;
	float *proposal=proposals;
	float *opt_layout=opt_layouts;
	float *params_grad=params_grads;


	float fx=evaluateLayout(d,init_layout,params,params_grad);
	float fmin=fx;
	memcpy(opt_layout,init_layout, d->layout_size*sizeof(float));

	float *curr_layout=init_layout;

	float prop, fprop;

	int prop_hist[NUM_PROPOSALS];
	memset(prop_hist,0,NUM_PROPOSALS*(sizeof(int)));


	//main annealing loop
	for (int i=0;i<num_iter;i++)
	{

		float temp=((end_temp-start_temp)* (float(i)/float(num_iter)))+start_temp;

		//generate proposal
		int prop_type=getProposal(d,curr_layout,proposal);
		prop_hist[prop_type]+=1;

		fprop=evaluateLayout(d,proposal,params,params_grad);


		if (fprop<fx)
			prop=1;
		else
		{
			prop=exp(-1*temp*fprop)/(exp(-1*temp*fx)+0.000001);

			if (isnan(prop))
				prop=0;
			else
				prop=min(1.0,prop);
		}

		printf("Annealing Iteration %i, fx %.3f,fprop %.3f, prop %.2f\n\n", i, fx, fprop,prop);

        if (prop > randu())
        {
        	memcpy(curr_layout, proposal, d->layout_size*sizeof(float));

            fx=fprop;
        }

        if (fx < fmin)
        {
        	memcpy(opt_layout, proposal, d->layout_size*sizeof(float));
        	fmin=fx;
        }
	}


	//assert(evaluateLayout(d,opt_layout,params,params_grad)==fmin);

	*eval=evaluateLayout(d,opt_layout,params,params_grad);

	float *xp=opt_layout;
	float *yp=&opt_layout[d->num_elements];
	float *hs=&opt_layout[2*d->num_elements];
	float *alts=&opt_layout[3*d->num_elements];


	printf("Best Layout\n");
	for (int i=0;i<d->num_elements;i++)
	{
		printf("\t %i x %i \t y %i \t h %i \t a %i\n",i,int(xp*d->width), int(yp*d->height), int(hs[i]*d->height), int(alts[i]));
	}
	printf("Best eval %.2f", *eval);


	*return_status=0;


	for (int i=0;i<NUM_PROPOSALS;i++)
	{
		printf("Prop %i, cnt %i", i, prop_hist[i]);
	}
}


*/


//, float *opt_layout
__global__ void parallelTempering(Design *d, int debug_mode, int iter_count, float start_temp, float end_temp, int num_iter,int num_params,float *params,float *atan_params,  float *temperatures, int *temp_ids,float *init_layouts,float *proposals,   float *params_grads, float *opt_layouts,float *evals, int num_previous_layout, float *previous_layout,float *random_seed,int *barrier, float *eval_sum)
{

	//if (blockIdx.x%EVAL_SPLIT_NUM!=0)
	//	return ;


	int thread_id= threadIdx.x + int(blockIdx.x/EVAL_SPLIT_NUM) * blockDim.x;
	
	int eval_id= blockIdx.x%EVAL_SPLIT_NUM;
	
	//Todo: set these with some random numbers from the CPU
	z3=random_seed[thread_id*4];
	z4=random_seed[thread_id*4+1];
	z1=random_seed[thread_id*4+2];
	z2=random_seed[thread_id*4+3];
	
	//access the memory for this thread
	float *init_layout=&init_layouts[thread_id*d->layout_size];
	float *proposal=&proposals[thread_id*d->layout_size];
	float *opt_layout=&opt_layouts[thread_id*d->layout_size];
	float *params_grad=&params_grads[thread_id*num_params];
	float temp=temperatures[thread_id];

	int ne=d->num_elements;
	//int offset=ne*NUM_VAR;

	
	//int prop_hist[NUM_PROPOSALS];
	//int accept_hist[NUM_PROPOSALS];
	//memset(prop_hist,0,NUM_PROPOSALS*(sizeof(int)));
	//memset(accept_hist,0,NUM_PROPOSALS*(sizeof(int)));

	float fx=evaluateLayout(d,init_layout,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);
	
	
	
	for (int j=0;j<d->layout_size;j++)
		opt_layout[j]=init_layout[j];

	float *curr_layout=init_layout;

	float prop, fprop;

	
	for(int i=0;i< d->num_elements;i++)
	{			
		if (curr_layout[i*NUM_VAR+2]<0)	
			printf("Error in element %i, %.3f %.3f %.3f\n",i, curr_layout[i*NUM_VAR], curr_layout[i*NUM_VAR+1], curr_layout[i*NUM_VAR+2]);
		
	}
	
	
	//main annealing loop
	for (int i=0;i<num_iter;i++)
	{
		

		//temp=((end_temp-start_temp)* (float(i)/float(num_iter)))+start_temp;
		if (end_temp!=-1)
			temp=((end_temp-start_temp)* (float(i)/float(num_iter)))+start_temp;

	
		//bool screwed_up=false;
		for (int j=0;j <ne;j++)
		{
			
			
			if((d->check_layout_exists) &&(d->check_layout[j*NUM_VAR+4]>FIX_LAYOUT_THRESH) && ((curr_layout[j*NUM_VAR]!=d->check_layout[j*NUM_VAR])||(curr_layout[j*NUM_VAR+1]!=d->check_layout[j*NUM_VAR+1])||(curr_layout[j*NUM_VAR+2]!=d->check_layout[j*NUM_VAR+2])))
			{	printf("PT error. Iteration %i. Layout vs check layout element %i (%f,%f,%f) (%f,%f,%f)\n ",i,j,curr_layout[j*NUM_VAR],curr_layout[j*NUM_VAR+1],curr_layout[j*NUM_VAR+2],d->check_layout[j*NUM_VAR],d->check_layout[j*NUM_VAR+1],d->check_layout[j*NUM_VAR+2]);
				return;
			}
			
			
			if ((curr_layout[j*NUM_VAR+2]==0) && (curr_layout[j*NUM_VAR+3]==0))
			{
				printf("element %i has an error\n",j);
				return	;
			}
		}
	

		//generate proposal
		int prop_type=-1;

		if (eval_id==0)
		{
			prop_type=getProposal(d,curr_layout,proposal,d->refine,i==0);
			
			if (prop_type<0)
				return;
		}
		
		/*
		//sync blocks
		
	    atomicSub( &(barrier[thread_id]) , 1 );
    	while ( atomicCAS( &(barrier[thread_id]) , 0 , 0 ) != 0 );  
    	
    	//reset the block counter
    	if (eval_id==0)
    	{
			barrier[thread_id]= EVAL_SPLIT_NUM;
			eval_sum[thread_id]=0;
		}
		while ( atomicCAS( &(barrier[thread_id]) , EVAL_SPLIT_NUM , EVAL_SPLIT_NUM ) != EVAL_SPLIT_NUM );  
	
		
		fprop_partial=evaluateLayout(d,proposal,params,atan_params,params_grad,num_previous_layout,previous_layout,true,false);
		
		atomicAdd( &(eval_sum[thread_id]) , fprop_partial );
		atomicSub( &(barrier[thread_id]) , 1 );
		while ( atomicCAS( &(barrier[thread_id]) , 0 , 0 ) != 0 );  
		
    	if (eval_id==0)
    		barrier[thread_id]= EVAL_SPLIT_NUM;
		while ( atomicCAS( &(barrier[thread_id]) , EVAL_SPLIT_NUM , EVAL_SPLIT_NUM ) != EVAL_SPLIT_NUM );  
		*/
		
		if (eval_id==0)
		{
			//fprop=eval_sum[thread_id];
			//fprop=randu();	
			
			fprop=evaluateLayout(d,proposal,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);
			
			/*
			if (abs(fprop-eval_sum[thread_id])>0.01)
				printf("fprop %.2f doesnt match fprop test %.2f\n",fprop,eval_sum[thread_id]);
			*/
			
			
			if (!isfinite(fprop))
			{
				if (thread_id==0)
				{
					
					printf("ERROR: fprop isnt finite. %.3f\n",fprop);
					
					for(int j=0;j< d->num_elements;j++)
						printf("Element %i, %.3f %.3f %.3f %.3f %.3f\n",j, curr_layout[j*NUM_VAR], curr_layout[j*NUM_VAR+1], curr_layout[j*NUM_VAR+2], curr_layout[j*NUM_VAR+3], curr_layout[j*NUM_VAR+4]);
						
					
					fprop=evaluateLayout(d,proposal,params,atan_params,params_grad,num_previous_layout,previous_layout,false,true,false);
					
					printf("Feature Breakdown:\n");
					for (int k=0;k<NUM_FEATURES;k++)
						printf("%i\t%5.1f \t %4.2f\t %4.1f \tnl:%3.1f\n", k,params[k],params_grad[k],params[k]*params_grad[k],params[k+NUM_FEATURES]);
			
					
				}
				
				return; 
				
			}
			
			if (fprop==99997)
			{
				printf("fucked up with prop type: %d\n",prop_type);
				return;
			}
			
			
			if (fprop==CHECK_LAYOUT_ERROR)
			{
				printf("check layout error: %d\n",prop_type);
				return;
			}
			
			if (fprop==ASPECT_RATIO_ERROR)
			{
				printf("aspect ratio error: %d\n",prop_type);
				return;
			}
				
			
			
			if (fprop<fx)
				prop=1;
			if (fprop>99990)
				prop=0;
			else
			{
				
				prop=exp(-1*(fprop-fx)/temp);
				//prop=fprop;
	
				if (!isfinite(prop))
					prop=0;
				else
					prop=min(1.0,prop);
					
			}
	
			
	        if (prop > randu())
	        {
	        	for (int j=0;j<d->layout_size;j++)
	        		curr_layout[j]=proposal[j];
	            fx=fprop;
	            
	            //accept_hist[prop_type]+=1;
	        }
        	
       }

	}
	
	
	if (eval_id==0)
	{
		//memcpy(opt_layout,curr_layout, d->layout_size*sizeof(float));
		//memcpy(init_layout, curr_layout, d->layout_size*sizeof(float));
		for (int  i=0;i<d->layout_size;i++)
		{
			opt_layout[i]=curr_layout[i];
			init_layout[i]=curr_layout[i];
		}
		
		evals[thread_id]=fx;

	}
	
	

	//free(param_var);
	
}



__device__ float getAlignmentDisplacement(int num_elements,int elem,Box *bb,int axis,int align_type)
{

	int other_elem=(num_elements)*randu();

	int cnt=0;
	while ((cnt <100)&&(other_elem==elem))
	{
		other_elem=(num_elements)*randu();
		cnt++;	
	}
	if (cnt==100)
	{
		printf("ERROR in getAlignmentDisplacement\n");
		return 0;
	}

	//x axis alignment
	if (axis==0)
	{
		if (align_type==0)
			return bb[other_elem].l-bb[elem].l;
		else if (align_type==1)
			return (bb[other_elem].l+bb[other_elem].r)/2.0-(bb[elem].l+bb[elem].r)/2.0;
		else
			return bb[other_elem].r-bb[elem].r;
	}
	//y axis alignment
	else
	{
		if (align_type==0)
			return bb[other_elem].b-bb[elem].b;
		else if (align_type==1)
			return (bb[other_elem].b+bb[other_elem].t)/2.0-(bb[elem].b+bb[elem].t)/2.0;
		else
			return bb[other_elem].t-bb[elem].t;
	}

}

	



__device__ int getProposal(Design *d, float *curr_layout, float *proposal,bool refine,bool debug)
{



	float aspect_ratio[MAX_ELEMENTS];
	int num_lines[MAX_ELEMENTS];

	float elem_select_strength[MAX_ELEMENTS];

	int ne=d->num_elements;

	Box bb[MAX_ELEMENTS];
	Box new_bb[MAX_ELEMENTS];
	
	
	float height,width,xp,yp;
	
	
	float cum_sum=0;
	for (int i=0;i<ne;i++)
	{
		float curr_strength=1.0;
		if (d->check_layout_exists)
		{
			curr_strength=max(0.0,1-d->check_layout[NUM_VAR*i+4]);
			//printf("fix %f strength %f\n",d->check_layout[NUM_VAR*i+4],curr_strength);
		}

		if (d->check_layout[NUM_VAR*i+4]>=1)
			curr_strength=0;
		cum_sum+=curr_strength;
		elem_select_strength[i]=cum_sum;
		
		int alt=int(curr_layout[NUM_VAR*i+6]);
		
		if (alt<0)
		{
			//visible[i]=false;
			bb[i].set(-1001,-1000,-1001,-1000);
		}
		else
		{
			if (d->num_alt[i]>0)
				aspect_ratio[i]=d->alt_aspect_ratio[i*MAX_ALT+alt];
			else
				aspect_ratio[i]=d->aspect_ratio[i];
				
			if (d->num_alt[i]>0)
				num_lines[i]=d->alt_num_lines[i*MAX_ALT+alt];
			else
				num_lines[i]=d->num_lines[i];	
			
			height=round(curr_layout[NUM_VAR*i+2]*d->height);
			
			if (height<=0)
			{
				printf("Error in curr_layout element %i has height %.2f\n", i, height);
				return -1;
			}
			width=round(height/aspect_ratio[i]);
			xp=round(curr_layout[NUM_VAR*i]*d->width);
			yp=round(curr_layout[NUM_VAR*i+1]*d->height);
	
			bb[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
		}

		/*
		if ((!isfinite(curr_layout[i*NUM_VAR]))|| (!isfinite(curr_layout[i*NUM_VAR+1]))|| (!isfinite(curr_layout[i*NUM_VAR+2])))
		{
			printf("Error in curr_layout element %i has prop %f %f %f, alt %f\n",i,curr_layout[i*NUM_VAR],curr_layout[i*NUM_VAR+1],curr_layout[i*NUM_VAR+2],curr_layout[i*NUM_VAR+6]);
			return -1;
		}
		*/
		

		//if (debug)
		//	printf("%i: %.2f %.2f %.2f %.2f\n",i,bb[i].l,bb[i].r,bb[i].b,bb[i].t);
		/*
		if ((bb[i].l<-2*d->width) || (bb[i].t<-2*d->height)|| (bb[i].r>=2*d->width)|| (bb[i].t>=2*d->height))
		{
				printf("screwed up %i: %f %f %f %f, w/h: %f %f, ar: %f\n \t l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f\n",i, curr_layout[NUM_VAR*i],curr_layout[NUM_VAR*i+1],curr_layout[NUM_VAR*i+2],curr_layout[NUM_VAR*i+3],width,height,aspect_ratio[i],bb[i].l,bb[i].r,bb[i].b,bb[i].t);
				return -1;
		}
		*/

		//if ((bb[i].r<=0) || (bb[i].t<=0)|| (bb[i].l>=d->width)|| (bb[i].b>=d->height))
		//	screwed_up=true;
		//if ((curr_layout[i*NUM_VAR]>=0.99) ||  (curr_layout[i*NUM_VAR+1]>=0.99))


	}
	
	
	int elems[MAX_ELEMENTS];

	

	float disp_x=0;
	float disp_y=0;
	float scale_mod=1;

	int m=0;
	int prop_select=0;
	Box new_box;
	for (m=0;m<MAX_PROPOSAL_TRIES;m++)
	{

		
		//memcpy(proposal, curr_layout, d->layout_size*sizeof(float));
		for (int j=0;j<d->layout_size;j++)
			proposal[j]=curr_layout[j];



		float rand=randu();

		
		float elem_select=cum_sum*rand;
		int elem=-1;
		for (int i=0;i<ne;i++)
		{
			if 	(elem_select_strength[i]>elem_select)
			{
				elem=i;
				break;
			}
		}
		
		if (elem==-1)
		{
			//printf("elem neg %i %.3f cum_sum %.3f\n",elem,elem_select,cum_sum );
			
			//for (int i=0;i<ne;i++)
			//	printf("\t i%i %.3f %.3f\n",i,elem_select_strength[i],d->check_layout[NUM_VAR*i+4]);
			continue;
		}
		
		
		
		/*
		int elem=((float)ne*rand);

		int cnt=0;
		while ((cnt<20)&&((proposal[NUM_VAR*elem+4]>=1) || ((proposal[NUM_VAR*elem+4]==SELECTED_NOFIX) && (randu()<0.95))))
		{
			elem =((float)ne*randu());
			cnt++;
		}
		if (cnt==20)
			return -1;
		*/
		
		
		
		int select;
		
		//if (!refine)
		//{
			select=(((float)NUM_PROPOSALS)*randu());
			prop_select=prop_choice[select];
		
		// }
		//else
		//{
		//	select=(((float)NUM_REFINE_PROPOSALS)*randu());
		//	prop_select=refine_prop_choice[select];	
		//}
		
		if (abs(proposal[NUM_VAR*elem+4]-SELECTED_NOFIX)<0.001)
			prop_select=SWITCH_ALIGNMENT;
		
		
		
		
		//bool already_overlapped=checkBoundingBoxOverlap(new_box,bb,ne,elem);
		bool already_overlapped=false;
		
		//bool skipped=false;

		int num_mod_elements=1;

		disp_x=0;
		disp_y=0;
		scale_mod=1;
		int new_align=-1;

		if (prop_select==CURR_LOCATION_SHIFT)
		{

			float location_variance=0.1;
			float2 rands=randn();
			//printf("random n %.5f,%.5f \n",rands.x, rands.y);


			if (rands.x>0)
				disp_x=rands.y*location_variance*d->width;
			else
				disp_y=rands.y*location_variance*d->height;

			//printf("Current location shift on element %i, dim %i, added %.3f\n", elem, ((int) rands.x>0),rands.y*location_variance);

		}
		else if ((prop_select==CURR_HEIGHT_SHIFT))
		{


			float height_variance=0.05;
			if (d->type[elem]!=1)
				height_variance=height_variance*2;
				
				
			float new_height=0;
			while (new_height<0.01)
			{
				float2 rands=randn();
				new_height=proposal[elem*NUM_VAR+2]+rands.y*height_variance;
			}

			scale_mod=new_height/proposal[elem*NUM_VAR+2];
			
			

			//printf("Height shift on element %i, added %.3f\n", elem,new_height);

			float elem_width=bb[elem].r-bb[elem].l;
			float elem_height=bb[elem].t-bb[elem].b;
			if ((elem_width*scale_mod < 2) or (elem_height*scale_mod < 2))
				continue;

			/*
            disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width)/2.0);
            disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height)/2.0);
            
			float scale=max(d->width, d->height);
			
			for (int i=0;i<ne;i++)
			{
				if (abs(bb[elem].r/scale- (bb[i].r/scale))<ALIGN_THRESH)
				{
	                disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width));
	               // disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height));
				}
				else if (abs(bb[elem].mid_x()/scale - (bb[i].mid_x()/scale))<ALIGN_THRESH)
				{
	                disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width)/2.0);
	              //  disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height)/2.0);
				}
			
				if (abs(bb[elem].t/scale- (bb[i].t/scale))<ALIGN_THRESH)
				{
	                disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height));
	               // disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height));
				}
				else if (abs(bb[elem].mid_y()/scale - (bb[i].mid_y()/scale))<ALIGN_THRESH)
				{
					disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height)/2.0);
				}
			}
			*/

			float r=randu();
			if (r<0.33)
                disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width)/2.0);
			else if (r<0.66)
                disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width));
			
			
			r=randu();
			
			if (r<0.33)
			{
               disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height)/2.0);
                //printf("scale_mod %.3f, disp_x %.3f disp_y %.3f\n",scale_mod,disp_x,disp_y);
			}
			else if (r<0.66)
				disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height));
			
			

			
			/*
			else if  (r<0.66)
			{
	            disp_x=-1.0*round((round(elem_width*scale_mod)-elem_width));
                disp_y=-1.0*round((round(elem_height*scale_mod)-elem_height));	
			}
			*/
			
		}




		else if (prop_select==GLOBAL_LOCATION_SHIFT)
		{

			Box shift_box;
			shift_box.set(bb[elem].l,bb[elem].r,bb[elem].b,bb[elem].t);


			int count=1;
			bool overlap=true;
			while (overlap and (count < 20))
			{
				float new_x=((d->width-(bb[elem].r-bb[elem].l))*randu());
				float new_y=((d->height-(bb[elem].t-bb[elem].b))*randu());

				shift_box.set_pos(new_x,new_y);

				overlap=checkBoundingBoxOverlap(shift_box,bb,ne,elem);
				count++;
			}

			disp_x=shift_box.l-bb[elem].l;
			disp_y=shift_box.b-bb[elem].b;

		}
		else if (prop_select==ALIGN_ELEMENT)
		{

			float rand_select=randu();

			if (rand_select<0.2)
				disp_x=getAlignmentDisplacement(d->num_elements,elem,bb,0,-1);
			else if (rand_select<0.4)
				disp_x=getAlignmentDisplacement(d->num_elements,elem,bb,0,0);
			else if (rand_select<0.6)
				disp_x=getAlignmentDisplacement(d->num_elements,elem,bb,0,1);
			else if (rand_select<0.7)
				disp_y=getAlignmentDisplacement(d->num_elements,elem,bb,1,-1);
			else if (rand_select<0.8)
				disp_y=getAlignmentDisplacement(d->num_elements,elem,bb,1,0);
			else
				disp_y=getAlignmentDisplacement(d->num_elements,elem,bb,1,1);
		}
		else if (prop_select==SWITCH_ALIGNMENT)
		{
			float rand_select=randu();
			
			if (d->fixed_alignment[elem]==0)
			{
				if (rand_select<0.5)
					new_align=0;
				//else if (rand_select<1.1)
				else
					new_align=1;
				//else
				//	new_align=2;
			}
			else
				continue;
		}
		else if (prop_select==GLOBAL_ALIGN)
		{

			float pos=int(round(randu()*10.0)/10.0);

			float axis_select=randu();

			if (axis_select <0.5)
			{
				float center=(bb[elem].l+bb[elem].r)/2.0;
				disp_x=pos*d->width - center;
			}
			else
			{
				float center=(bb[elem].b+bb[elem].t)/2.0;
				disp_y=pos*d->height - center;
			}
		}
		else if (prop_select==SWITCH_ALTERNATE)
		{

			int old_alt=proposal[elem*NUM_VAR+6];	
			
	
			if ((d->optional[elem]) && (old_alt>-1) && (randu()<0.5))
				proposal[elem*NUM_VAR+6]=-1;
			
			else if ((old_alt<0) || (d->num_alt[elem]>0))
			{
				float elem_aspect_ratio=d->aspect_ratio[elem];
				float elem_num_lines=d->num_lines[elem];

				 if (d->num_alt[elem]>0)
				 {

					int alt=(d->num_alt[elem]*randu());
					
					if ((d->alt_num_lines[elem*MAX_ALT+alt]>2)&&(d->alt_aspect_ratio[elem*MAX_ALT+alt]>0.5))
						continue;
					
					float scale=d->alt_num_lines[elem*MAX_ALT+alt]/d->alt_num_lines[elem*MAX_ALT+old_alt];
					
					elem_aspect_ratio=d->alt_aspect_ratio[elem*MAX_ALT+alt];
					elem_num_lines=d->alt_num_lines[elem*MAX_ALT+alt];
					
					proposal[NUM_VAR*elem+2]=proposal[NUM_VAR*elem+2]*scale;
					
					float scaled_height=(bb[elem].t-bb[elem].b)*scale;
					float scaled_width=scaled_height/elem_aspect_ratio;
					
					if ((scaled_width < 3) or (scaled_height < 3))
						continue;
						
					proposal[elem*NUM_VAR+6]=alt;
					
				}
				else
				{
					proposal[elem*NUM_VAR+6]=0;
				}
				
	
				height=(proposal[NUM_VAR*elem+2]*d->height);
				width=(height/elem_aspect_ratio);
				xp=(proposal[NUM_VAR*elem]*d->width);
				yp=(proposal[NUM_VAR*elem+1]*d->height);
		
				bb[elem].set(round(xp+(width*d->bb_left[elem])), round(xp+(width*d->bb_right[elem])), round(yp + (height*d->bb_bottom[elem])), round(yp + (height*d->bb_top[elem])));
				
				//if ((bb[elem].l>=bb[elem].r) || (bb[elem].b>=bb[elem].t))
				//	continue;
				
				//printf("changed alt from %i to %i. num alt %i \n",int(proposal[elem*NUM_VAR+6]),alt,d->num_alt[elem]);
				
				
			 }
			else
				continue;
			
		
		}
		
		else if ((prop_select==FLIP_TWO_ELEMENTS) && (d->num_elements>1))
		{
			
		
			int other_elem=(d->num_elements)*randu();
			
		
			int cnt=0;
			while ((cnt <50)&&((other_elem==elem)||(proposal[other_elem*NUM_VAR+4]>FIX_LAYOUT_THRESH)))
			{
				other_elem=(d->num_elements)*randu();
				cnt++;	
			}
			
			
			elems[0]=elem;
			elems[1]=other_elem;
			
			num_mod_elements=2;		
			//for (int i=0;i<ne;i++)
			//	new_bb[i].set(bb[i].l, bb[i].r,bb[i].b,bb[i].t);
			
			float rand_select=randu();
			bool flip_x=false;
			bool flip_y=false;
			if (rand_select<0.33)
				flip_y=true;
			else if (rand_select<0.66)
				flip_x=true;
			else if (rand_select<1)
			{
				flip_x=true;
				flip_y=true;

			}
			
			float tmp;
			if (flip_x)
			{
				tmp=proposal[elem*NUM_VAR];
				proposal[elem*NUM_VAR]=proposal[other_elem*NUM_VAR];
				proposal[other_elem*NUM_VAR]=tmp;		
			}
			if  (flip_y)
			{
				tmp=proposal[elem*NUM_VAR+1];
				proposal[elem*NUM_VAR+1]=proposal[other_elem*NUM_VAR+1];
				proposal[other_elem*NUM_VAR+1]=tmp;						
			}
				
		}
		else if ((prop_select==SHIFT_ALIGNED_ELEMENTS) && (d->num_elements>1))
		{
			
			float scale=max(d->width, d->height);
			
			
			float location_variance=0.1;
			float2 rands=randn();

			if (rands.x>0)
				disp_x=rands.y*location_variance*d->width;
			else
				disp_y=rands.y*location_variance*d->height;
			
			float elem_l=bb[elem].l/scale;
			float elem_m=bb[elem].mid_x()/scale;
			float elem_r=bb[elem].r/scale;
				
			num_mod_elements=0;
			for (int i=0;i<ne;i++)
			{
				if (proposal[i*NUM_VAR+4]<FIX_LAYOUT_THRESH)
					if  ((abs(elem_l- (bb[i].l/scale))<ALIGN_THRESH) || (abs(elem_m- (bb[i].mid_x()/scale))<ALIGN_THRESH) || (abs(elem_r- (bb[i].r/scale))<ALIGN_THRESH))
					{
						proposal[i*NUM_VAR]+=disp_x/d->width;
						proposal[i*NUM_VAR+1]+=disp_y/d->height;
						elems[num_mod_elements]=i;
						num_mod_elements++;
					}
				
			}
			
			if (num_mod_elements==0)
				continue;
		}
		
		
		
		else if ((prop_select==SCALE_TYPE))
		{
		
			float2 rands=randn();
			int type=0;
			if (rands.x>0)
				type=1;
		

			float height_variance=0.1;

				
			float scale_factor= 1.0+rands.y*height_variance;
			
			//printf("type %i, scale factor %.2f\n",type,scale_factor);

			num_mod_elements=0;
			for (int i=0;i<ne;i++)
			{
				if ((d->type[i]==type)  && (proposal[i*NUM_VAR+4]<FIX_LAYOUT_THRESH))
				{
					proposal[i*NUM_VAR+2]=proposal[i*NUM_VAR+2]*scale_factor;
					elems[num_mod_elements]=i;
					num_mod_elements++;
				}
			}
		}

		
		
		
		
		
		/*
		if (num_mod_elements>1)
		{
			for (int i=0;i<ne;i++)
				new_bb[i].set(bb[i].l, bb[i].r,bb[i].b,bb[i].t);
				
			for (int n=0;n<num_mod_elements;n++)
			{
	
				int i=elems[n];
				height=round(proposal[NUM_VAR*i+2]*d->height);
				width=round(height/aspect_ratio[i]);
				xp=round(proposal[NUM_VAR*i]*d->width);
				yp=round(proposal[NUM_VAR*i+1]*d->height);
				new_bb[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
				
				
				
				if ((new_box.l<-2*d->width) || (new_box.b<-2*d->height)|| (new_box.r>2*d->width)|| (new_box.t>2*d->height))
				{
					
					printf("ERROR setting new box, prop_select %i for elem %i (of %i), fix amt %.3f, prop (%.3f %.3f %.3f ) vs curr(%.3f %.3f %.3f ) \n box l/r %.3f %.3f, b/t %.3f %.3f\n",prop_select,i,num_mod_elements,proposal[NUM_VAR*i+4],proposal[i*NUM_VAR],proposal[i*NUM_VAR+1],proposal[i*NUM_VAR+2],curr_layout[i*NUM_VAR],curr_layout[i*NUM_VAR+1],curr_layout[i*NUM_VAR+2],new_bb[i].l,new_bb[i].r,new_bb[i].b,new_bb[i].t);
					return -1;
				}
				
			}
		}
		*/

		
		
		
		if ((new_align>-1)&&(num_mod_elements==1))
		{
			proposal[elem*NUM_VAR+3]=(float)new_align;
			
			
			//if ((proposal[NUM_VAR*elem+4]>FIX_LAYOUT_THRESH))
			//	printf("switching alignment proposal %i for elem %i , fix amt %.3f, prop (%.3f) vs curr(%.3f) \n",prop_select,elem,proposal[NUM_VAR*elem+4],proposal[NUM_VAR*elem+3],curr_layout[elem*NUM_VAR+3]);
			
			
			break;
		}
		
	
		bool passed_checks=true;
		for (int n=0;n<num_mod_elements;n++)
		{
			
			if (num_mod_elements==1)
			{
				
				proposal[elem*NUM_VAR]+=disp_x/d->width;
				proposal[elem*NUM_VAR+1]+=disp_y/d->height;
		
				new_box.set(bb[elem].l+disp_x,bb[elem].r+disp_x,bb[elem].b+disp_y,bb[elem].t+disp_y);
		
				if (scale_mod!=1)
				{
					proposal[elem*NUM_VAR+2]=proposal[elem*NUM_VAR+2]*scale_mod;
					new_box.scale(scale_mod);
				}	
			}
			else
			{
				elem=elems[n];
				
				height=round(proposal[NUM_VAR*elem+2]*d->height);
				width=round(height/aspect_ratio[elem]);
				xp=round(proposal[NUM_VAR*elem]*d->width);
				yp=round(proposal[NUM_VAR*elem+1]*d->height);
				new_box.set(round(xp+(width*d->bb_left[elem])), round(xp+(width*d->bb_right[elem])), round(yp + (height*d->bb_bottom[elem])), round(yp + (height*d->bb_top[elem])));
				
				/*
				if ((new_box.l<-2*d->width) || (new_box.b<-2*d->height)|| (new_box.r>2*d->width)|| (new_box.t>2*d->height))
				{
					printf("ERROR setting new box, m %i, prop_select %i for elem %i (of %i), fix amt %.3f, ar %f,width %.3f height %.3f, prop (%.3f %.3f %.3f )  \n box old l/r %.3f %.3f, b/t %.3f %.3f, new l/r %.3f %.3f, b/t %.3f %.3f\n",m,prop_select,elem,num_mod_elements,proposal[NUM_VAR*elem+4],aspect_ratio[elem],width,height,proposal[elem*NUM_VAR],proposal[elem*NUM_VAR+1],proposal[elem*NUM_VAR+2],bb[elem].l,bb[elem].r,bb[elem].b,bb[elem].t,new_box.l,new_box.r,new_box.b,new_box.t);
					return -1;
				}
				*/
				
			}
			
			
			
			if ((!isfinite(proposal[elem*NUM_VAR]))|| (!isfinite(proposal[elem*NUM_VAR+1]))|| (!isfinite(proposal[elem*NUM_VAR+2])))
			{
				printf("Error in proposal type %i, element %i has prop %f %f %f\n",prop_select,elem,proposal[elem*NUM_VAR],proposal[elem*NUM_VAR+1],proposal[elem*NUM_VAR+2]);
				passed_checks=false;
				break;
			}
		
		

		
			//skip these checks if we're switching alternates to allow hiding/showing objects
			if ((prop_select==SWITCH_ALTERNATE))
				continue;
			
			
			float elem_size=10.0*(((new_box.t-new_box.b)/num_lines[elem])/max(d->width, d->height));
			
			if (elem_size<MIN_TEXT_SIZE)
			{
				passed_checks=false;
				break;
			}
			
			
			if ((round(new_box.r)-round(new_box.l)<3) || (round(new_box.t)-round(new_box.b)<3))
			{
				passed_checks=false;
				break;
			}
			
			/*
			if ((new_box.l<-2*d->width) || (new_box.b<-2*d->height)|| (new_box.r>2*d->width)|| (new_box.t>2*d->height))
			{
				
				printf("ERROR in proposal %i for elem %i (of %i), fix amt %.3f, prop (%.3f %.3f %.3f ) vs curr(%.3f %.3f %.3f ) \n box l/r %.3f %.3f, b/t %.3f %.3f\n",prop_select,elem,num_mod_elements,proposal[NUM_VAR*elem+4],proposal[elem*NUM_VAR],proposal[elem*NUM_VAR+1],proposal[elem*NUM_VAR+2],curr_layout[elem*NUM_VAR],curr_layout[elem*NUM_VAR+1],curr_layout[elem*NUM_VAR+2],new_box.l,new_box.r,new_box.b,new_box.t);
				passed_checks=false;
				break;
			}
			*/
				
				
			if ((new_box.r>d->width) || (new_box.t>d->height)|| (new_box.l<0)|| (new_box.b<0))
			{
				passed_checks=false;
				break;
			}
	
			/*
			if ((!already_overlapped) && (checkBoundingBoxOverlap(new_box,bb,ne,elem)))
			{
				passed_checks=false;
				break;
			}
			*/
		
			if ((proposal[NUM_VAR*elem+4]>1.0) && ((proposal[elem*NUM_VAR+2]!=curr_layout[elem*NUM_VAR+2]) || (proposal[elem*NUM_VAR+1]!=curr_layout[elem*NUM_VAR+1]) || (proposal[elem*NUM_VAR]!=curr_layout[elem*NUM_VAR])))
			{
				printf("ERROR in proposal %i for elem %i , fix amt %.3f, prop (%.3f %.3f %.3f ) vs curr(%.3f %.3f %.3f ) \n",prop_select,elem,proposal[NUM_VAR*elem+4],proposal[elem*NUM_VAR],proposal[elem*NUM_VAR+1],proposal[elem*NUM_VAR+2],curr_layout[elem*NUM_VAR],curr_layout[elem*NUM_VAR+1],curr_layout[elem*NUM_VAR+2]);
				passed_checks=false;
				break;
			} 

		
		}
		
		
		
		
		if (passed_checks)
			break;
		
	}
	

	/*
	for (int i=0;i<ne;i++)
	{
		height=round(proposal[NUM_VAR*i+2]*d->height);
		width=round(height/aspect_ratio[i]);
		xp=round(proposal[NUM_VAR*i]*d->width);
		yp=round(proposal[NUM_VAR*i+1]*d->height);

		Box elem_bb;
		elem_bb.set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));


		//if ((elem_bb.r<=0) || (elem_bb.t<=0)|| (elem_bb.l>=d->width)|| (elem_bb.b>=d->height))
		if ((elem_bb.r==elem_bb.l)||(elem_bb.b==elem_bb.t))
		{
			printf("wtf1? %i prop select %i d %.2f %.2f, s %.2f, m %i  \n\t %1.2f %1.2f %1.2f %1.2f\n\t %1.2f %1.2f %1.2f %1.2f \n\t %1.2f %1.2f %1.2f %1.2f\n \t prop: %1.2f %1.2f %1.2f %1.2f (w/h) %3.1f %3.1f\n",i,prop_select, disp_x, disp_y, scale_mod,m,bb[i].l, bb[i].r,bb[i].b,bb[i].t,elem_bb.l,elem_bb.r,elem_bb.b,elem_bb.t,new_box.l,new_box.r,new_box.b,new_box.t,proposal[NUM_VAR*i],proposal[NUM_VAR*i+1],proposal[NUM_VAR*i+2],proposal[NUM_VAR*i+3],width,height);
		}
	}
	*/
	
	//failed to get a good proposal so give us and return the current layout
	if (m==MAX_PROPOSAL_TRIES)
	{
		for (int j=0;j<d->layout_size;j++)
			proposal[j]=curr_layout[j];
	}
	


	/*

	if (not screwed_up)
	{
		for (int i=0;i<ne;i++)
		{
			height=round(proposal[NUM_VAR*i+2]*d->height);
			width=round(height/aspect_ratio[i]);
			xp=round(proposal[NUM_VAR*i]*d->width);
			yp=round(proposal[NUM_VAR*i+1]*d->height);

			Box elem_bb;
			elem_bb.set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));


			if ((elem_bb.r<=0) || (elem_bb.t<=0)|| (elem_bb.l>=d->width)|| (elem_bb.b>=d->height))
			{
				printf("wtf1? %i d %.2f %.2f, s %.2f, m %i , screwed %i \n\t %1.2f %1.2f %1.2f %1.2f\n\t %1.2f %1.2f %1.2f %1.2f \n\t %1.2f %1.2f %1.2f %1.2f\n \t prop: %1.2f %1.2f %1.2f %1.2f (w/h) %3.1f %3.1f\n",i,disp_x, disp_y, scale_mod,m,screwed_up,bb[i].l, bb[i].r,bb[i].b,bb[i].t,elem_bb.l,elem_bb.r,elem_bb.b,elem_bb.t,new_box.l,new_box.r,new_box.b,new_box.t,proposal[NUM_VAR*i],proposal[NUM_VAR*i+1],proposal[NUM_VAR*i+2],proposal[NUM_VAR*i+3],width,height);
			}
		}
	}
	*/


	//free(bb);
	//if (screwed_up)
	//	return -1;

	return prop_select;
}



__global__ void finiteDiffLayoutGrad(Design *d,int num_dir,float *directions,float *dir_map,int num_params, float *params, float *atan_params,float *params_grads, float *layout,float *layouts, float *layout_grad,float *line_search,int num_previous_layout, float *previous_layout)
{
	
	//if (blockIdx.x%EVAL_SPLIT_NUM==0)
	//	return;
	
	

	int thread_id= threadIdx.x + blockIdx.x * blockDim.x;
	
	if (thread_id>=num_dir)
	{
		//printf("thread_id %i > num_dir %i\n",thread_id,num_dir);
		return;	
	}
	

	//int num_var=d->num_elements*3;
	
	
	//float *dir=&(directions[thread_id*num_var]);
	float *dir=&(dir_map[thread_id*d->layout_size]);
	float *layout_copy=&(layouts[thread_id*d->layout_size]);
	float *params_grad=&(params_grads[thread_id*num_params]);
	
	
	
	double y=evaluateLayout(d,layout,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);
	
	
	
	double delta=FD_DELTA;
	
	for(int j=0;j<d->layout_size;j++)
		layout_copy[j]=layout[j]+delta*dir[j];	
	
	

	/*
	for(int j=0;j<num_var;j++)
	{
		if (abs(dir[j])>0.01)
		{
			int elem=(j%d->num_elements);
			int elem_var=(j/d->num_elements);
			int idx=elem*NUM_VAR+elem_var;
			
			if (layout[elem*NUM_VAR+4]!=1.0)
				layout_copy[idx]=layout[idx]+delta*dir[j];
			
		}
	}*/


		

	double y2=evaluateLayout(d,layout_copy,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);
	
	for(int j=0;j<d->layout_size;j++)
		layout_copy[j]=layout[j]-delta*dir[j];	


	double y1=evaluateLayout(d,layout_copy,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);

	layout_grad[thread_id]=(float)((y2-y1)/(2.0*delta));
	
	
	if ((y1==INVALID_ELEMENT_ERROR) || (y2==INVALID_ELEMENT_ERROR))
	{
		printf("INVALID_ELEMENT_ERROR called in gradient check\n direction:\n");
		/*
		for(int j=0;j<num_var;j++)
		{
			if (abs(dir[j])>0.01)
			{
				int elem=(j%d->num_elements);
				int elem_var=(j/d->num_elements);
				int idx=elem*NUM_VAR+elem_var;
				printf("elem %i, var %i, delta*dir %f, delta %f,dir %f\n",elem,elem_var,delta*dir[j],delta,dir[j]);
				
			}
		}
		*/
	}
	
	//if ((y==y1) ||(y==y2))
	//	printf("delta too small. y %f, y1 %f, y2 %f\n",y,y1,y2);
	
	//if ((y<y1) && (y<y2))
	//	printf("delta too big. y %f, y1 %f, y2 %f\n",y,y1,y2);
	
	
	//do line search in this direction
	
	float *ls=&(line_search[thread_id*NUM_LINE_STEPS]);
	
	if (y2>y1)
		delta=-1*delta;
		
	ls[0]=y;
	
	for(int i=1;i< NUM_LINE_STEPS;i++)
	{

		/*
		for(int j=0;j<num_var;j++)
		{
			if (abs(dir[j])>0.01)
			{
				int elem=(j%d->num_elements);
				int elem_var=(j/d->num_elements);
				int idx=elem*NUM_VAR+elem_var;
				if (layout[elem*NUM_VAR+4]!=1.0)
				{
					layout_copy[idx]=layout[idx]+delta*dir[j];
					if (elem_var==2) 
						layout_copy[idx]=max(layout_copy[idx],0.005);
				}
			}
		}
		*/
		for(int j=0;j<d->layout_size;j++)
			layout_copy[j]=layout[j]+delta*dir[j];	
		
		double y_step=evaluateLayout(d,layout_copy,params,atan_params,params_grad,num_previous_layout,previous_layout,false,false,false);
		
		
		ls[i]=y_step;
		
		delta=delta*2;
	}
	

	
	
	if ((!isfinite(layout_grad[thread_id])))
	{
		printf("error in gradient calc %f %f\n",y1,y2);
		
		if ((!isfinite(y2)))
			y2=evaluateLayout(d,layout_copy,params,atan_params,params_grad,num_previous_layout,previous_layout,false,true,false);
	}

		
	
}


/*


__device__ int getRegionProposal(Design *d, float *curr_layout, float *proposal,bool debug)
{
	

		
	memcpy(proposal, curr_layout, d->layout_size*sizeof(float));
	
	int num_text_regions=0,num_graphic_regions=0,num_regions;
	int ne=d->num_elements;
	int offset=ne*NUM_VAR;
	int num_text=0, num_graphic=0;
	
	num_text_regions=0;
	num_graphic_regions=0;
	for(int i=0;i< ne;i++)
	{
		if (int(curr_layout[offset+i*NUM_RVAR])==1)
			num_text_regions++;
		else if (int(curr_layout[offset+i*NUM_RVAR])==2)
			num_graphic_regions++;
			
		if (d->type[i]==1)
			num_text++;
		else
			num_graphic++;
	}
	
	
	//if (debug)
	if (randu()<0.001)
		printf("getRegionProposal");
	
	
	float aspect_ratio[MAX_ELEMENTS];
	//int num_lines[MAX_ELEMENTS];



	
	int flip_cnt=0;

	float display=false;

	int prop_select=0;
			
	Box merge_r1;
	Box merge_r2;
	//bool fucked;
	//bool regions_fucked;
	int elem;
	int r;
	
	Box region_bb[MAX_ELEMENTS];
	Box bb[MAX_ELEMENTS];
	int num_elements_reg[MAX_ELEMENTS];
	int reg_change=0;
	
	
	int m;
	for (m=0;m<MAX_PROPOSAL_TRIES;m++)
	{

		memcpy(proposal, curr_layout, d->layout_size*sizeof(float));
		
		
		for (int i=0;i<ne;i++)
		{
			int alt=int(curr_layout[NUM_VAR*i+6]);
			if (alt>-1)
				aspect_ratio[i]=d->alt_aspect_ratio[i*MAX_ALT+alt];
			else
				aspect_ratio[i]=d->aspect_ratio[i];
		}

		
		num_regions=num_text_regions+num_graphic_regions;
		
		//if (((num_text>0) and (num_text_regions==0)) or ((num_graphic>0) and (num_graphic_regions==0)))
		//{	
		//	printf("error text %i %i, graphic %i %i\n",num_text,num_text_regions,num_graphic,num_graphic_regions);
		//	return -1;
		//}
		
		//bool contains_fixed[MAX_ELEMENTS];
		//for (int i=0;i<ne;i++)
		//	contains_fixed[i]=false;
		
		
		
		for (int i=0;i<ne;i++)
			num_elements_reg[i]=0;
			
		reg_change=0;

		
		
		//if element has no region, do random initial assignment
		for (int i=0;i<ne;i++)
		{
			int r=proposal[i*NUM_VAR+5];
			
			if (r>-1)
			{
				num_elements_reg[r]++;
				
				if (proposal[offset+r*NUM_RVAR]==-1)
				{
					printf("ERROR in initialization. region %i doesn't exist\n",r);
					return -1;
				}
			}
			
			//
			if ((r==-1) and (proposal[i*NUM_VAR+4]<FIX_LAYOUT_THRESH))
			{
				//printf("error, element %i has no region\n",i);
				//assignElementRegion(d,proposal,curr_layout,i,num_text_regions,num_graphic_regions);
				float height=round(proposal[NUM_VAR*i+2]*d->height);
				float width=round(height/aspect_ratio[i]);
				float xp=round(proposal[NUM_VAR*i]*d->width);
				float yp=round(proposal[NUM_VAR*i+1]*d->height);
				
				proposal[offset+num_regions*NUM_RVAR]=d->type[i];
				proposal[offset+num_regions*NUM_RVAR+1]=round(xp+(width*d->bb_left[i])); 
				proposal[offset+num_regions*NUM_RVAR+2]=round(yp+(height*d->bb_bottom[i]));
				proposal[offset+num_regions*NUM_RVAR+3]=width;
				proposal[offset+num_regions*NUM_RVAR+4]=height;
				proposal[offset+num_regions*NUM_RVAR+5]=1.0;	
				proposal[offset+num_regions*NUM_RVAR+6]=0.0;
				
				proposal[i*NUM_VAR+5]=num_regions;
				num_elements_reg[num_regions]=1;
				reg_change=1;
				num_regions++;	
			}
		}
		
		if (num_regions==0)
		{
			printf("ERROR in initialization. No regions at all?");
		
			for (int i=0;i<ne;i++)
				printf("element %i has assigned region %i\n",i,int(curr_layout[i*NUM_VAR+5]));
			
			for (int i=0;i<ne;i++)
				printf("region %i has type %i\n",i,int(curr_layout[offset+i*NUM_RVAR]))	;	

			return -1;
			
		}
		
		
		for (int reg=0;reg<num_regions;reg++)
		{
			if (num_elements_reg[reg]<1)
			{
				printf("ERROR in initialization. m %i, Region %i (type %i): %.2f %.2f %.2f %.2f has no elements. num regions %i (t %i g %i) \n",m,reg,int(proposal[offset+r*NUM_RVAR]), proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4],num_regions,num_text_regions,num_graphic_regions);
				
				
				//for (int reg2=0;reg2<num_regions;reg2++)
				//	printf("region %i has num elements %i\n",reg2,num_elements_reg[reg2]);
				
				//for (int i=0;i<ne;i++)
				//	printf("element %i has assigned region %i\n",i,int(curr_layout[i*NUM_VAR+5]));
				
				//for (int i=0;i<ne;i++)
				//	printf("region %i has type %i\n",i,int(curr_layout[offset+i*NUM_RVAR]))	;	

				
				return -1;
				
			}
		}
		
			
	
		//if (d->fixed_regions)
		//{	
		//	int select=(((float)NUM_FIXED_REGION_PROPOSALS)*randu());
		//	prop_select=fix_reg_prop_choice[select];
		//}
		//else
		//{
			int select=(((float)NUM_REGION_PROPOSALS)*randu());
			prop_select=reg_prop_choice[select];
		//}
		
		


		
		
		elem=((float)ne*randu());		
		
		int cnt=0;
		while ((proposal[elem*NUM_VAR+4]>FIX_LAYOUT_THRESH) && (cnt<20))
		{
			elem=((float)ne*randu());
			cnt++;	
		}
		if (cnt==20)	
		{
			printf("ERROR in selecting element");
			return -1;
		}
		
		r=((float)num_regions*randu());
		
		
		
	
		for (int i=0;i<num_regions;i++)
		{
			region_bb[i].set(proposal[offset+i*NUM_RVAR+1], proposal[offset+i*NUM_RVAR+1]+ proposal[offset+i*NUM_RVAR+3], proposal[offset+i*NUM_RVAR+2],proposal[offset+i*NUM_RVAR+2]+proposal[offset+i*NUM_RVAR+4]);

		}
	
		
		float height,width,xp,yp;
	
		for (int i=0;i<ne;i++)
		{
			height=(proposal[NUM_VAR*i+2]*d->height);
			width=(height/aspect_ratio[i]);
			xp=(proposal[NUM_VAR*i]*d->width);
			yp=(proposal[NUM_VAR*i+1]*d->height);
	
			bb[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
			//bb[i].set(xp, xp+(width), yp, yp + (height));
			
			if ((bb[i].l>=bb[i].r)|| (bb[i].b>=bb[i].t))
			{
				printf("ERROR. input bounding box for element %i fucked l/r: %f %f b/t: %f %f\n",i,bb[i].l,bb[i].r,bb[i].b,bb[i].t);
				return -1;
			}	
		}
			
			
		
			
		float location_variance=0.1;

		if (prop_select==REG_SWAP)
		{
			if (num_regions>1)
			{
				int r2=((float)num_regions*randu());
				int cnt=0;
				while ((cnt<100)&&(r2==r))
				{
					r2=((float)num_regions*randu());
					cnt++;	
				}
				if (cnt==100)
				{
					printf("ERROR in selection %i\n",num_regions);
					return -1;
				}
					
				float temp;
				for (int i=1;i<3;i++)
				{
					temp=proposal[offset+r*NUM_RVAR+i];
					proposal[offset+r*NUM_RVAR+i]=proposal[offset+r2*NUM_RVAR+i];
					proposal[offset+r2*NUM_RVAR+i]=temp;
				}
			}
			else
				continue;
		}	
		
		else if (prop_select==REG_GLOBAL_LOCATION_SHIFT)
		{
	
			
			Box shift_box;
			shift_box.set(region_bb[r].l,region_bb[r].r,region_bb[r].b,region_bb[r].t);
	
			int cnt=1;
			bool overlap=true;
			while (overlap and (cnt < 20))
			{
				float new_x=((d->width-(region_bb[r].r-region_bb[r].l))*randu());
				float new_y=((d->height-(region_bb[r].t-region_bb[r].b))*randu());
	
				shift_box.set_pos(new_x,new_y);
	
				overlap=checkBoundingBoxOverlap(shift_box,region_bb,num_regions,r);
				cnt++;
			}
			
			
			proposal[offset+r*NUM_RVAR+1]+=shift_box.l-region_bb[r].l;
			proposal[offset+r*NUM_RVAR+2]+=shift_box.b-region_bb[r].b;
			//proposal[offset+r*NUM_RVAR+3]+=disp_y;
			//proposal[offset+r*NUM_RVAR+4]+=disp_y;
		}
		
		else if(prop_select==REG_CURR_LOCATION_SHIFT)
		{
			float2 rands=randn();
			
			if (randu()>0.5)
			{
				float x_offset=rands.y*location_variance*d->width;
				proposal[offset+r*NUM_RVAR+1]=min(max(proposal[offset+r*NUM_RVAR+1]+x_offset,0.0),d->width-1);	
				
			}
			else
			{
				float y_offset=rands.y*location_variance*d->height;
				proposal[offset+r*NUM_RVAR+2]=min(max(proposal[offset+r*NUM_RVAR+2]+y_offset,0.0),d->height-1);		
			}
		}
		
		
		else if(prop_select==REG_BB_SHIFT)
		{
			float2 rands=randn();
			
			int orientation=int(proposal[offset+r*NUM_RVAR+6]);
				
			if (orientation==1)	
				proposal[offset+r*NUM_RVAR+3]=min(max(proposal[offset+r*NUM_RVAR+3]+rands.y*location_variance*0.5*d->width,5.0),d->width-1);
			else
				proposal[offset+r*NUM_RVAR+4]=min(max(proposal[offset+r*NUM_RVAR+4]+rands.y*location_variance*0.5*d->height,5.0),d->height-1);
		}
	
		else if (prop_select==REG_ALIGN_TWO)
		{
	
			float rand_select=randu();
			if (num_regions>1)
			{
				if (rand_select<0.2)
					proposal[offset+r*NUM_RVAR+1]+=getAlignmentDisplacement(num_regions,r,region_bb,0,-1);
				else if (rand_select<0.4)
					proposal[offset+r*NUM_RVAR+1]+=getAlignmentDisplacement(num_regions,r,region_bb,0,0);
				else if (rand_select<0.6)
					proposal[offset+r*NUM_RVAR+1]+=getAlignmentDisplacement(num_regions,r,region_bb,0,1);
				else if (rand_select<0.7)
					proposal[offset+r*NUM_RVAR+2]+=getAlignmentDisplacement(num_regions,r,region_bb,1,-1);
				else if (rand_select<0.8)
					proposal[offset+r*NUM_RVAR+2]+=getAlignmentDisplacement(num_regions,r,region_bb,1,0);
				else
					proposal[offset+r*NUM_RVAR+2]+=getAlignmentDisplacement(num_regions,r,region_bb,1,1);
					
			}
			else
				continue;
			
		}	
			
		else if (prop_select==REG_ELEMENT_SWITCH)
		{
			//assignElementRegion(d,proposal,curr_layout,elem,num_text_regions,num_graphic_regions);
			
			
			int curr_region=proposal[elem*NUM_VAR+5];
			
			if (num_regions>2)
			{
				int r2=curr_region;
				
				int cnt=0;
				while ((cnt<100)&& (r2==curr_region))
				{
					r2=((float)num_regions*randu());
					cnt++;	
				}
				if(cnt==100)
				{
					printf("ERROR in selection %i\n",num_regions);
					return -1;
				}
					
				//this is the only element, so remove curr_region
				if (num_elements_reg[curr_region]==1)
				{
	
					if (num_regions-1 != curr_region)
					{
					
						for (int i=0;i < ne;i++)
						{
							if ((int(proposal[i*NUM_VAR+5])==num_regions-1)) 
								proposal[i*NUM_VAR+5]=curr_region;
						}	
					
						for (int i=0;i < NUM_RVAR;i++)
							proposal[offset+curr_region*NUM_RVAR+i]=proposal[offset+(num_regions-1)*NUM_RVAR+i];
	
					}
					
					proposal[offset+(num_regions-1)*NUM_RVAR]=-1;
					num_regions--;	
					reg_change=-1;			
				}
		
				if (r2==num_regions)
					proposal[elem*NUM_VAR+5]=curr_region;
				else
					proposal[elem*NUM_VAR+5]=r2;
					
			}
			else
				continue;
		}
		
	
		else if ((prop_select==REG_ELEMENT_SPLIT) and (num_regions<MAX_ELEMENTS))
		{
			
			int cnt=0;
			for (int i=0;i < ne;i++)
				if (proposal[i*NUM_VAR+5]==proposal[elem*NUM_VAR+5])
					cnt++;
					
			if (cnt>1)
			{
				
				Box shift_box;
				shift_box.set(bb[elem].l,bb[elem].r,bb[elem].b,bb[elem].t);
				
				int count=1;
				bool overlap=true;
				while (overlap and (count < 20))
				{
					float new_x=((d->width-(bb[elem].r-bb[elem].l))*randu());
					float new_y=((d->height-(bb[elem].t-bb[elem].b))*randu());
		
					shift_box.set_pos(new_x,new_y);
		
					overlap=checkBoundingBoxOverlap(shift_box,bb,d->num_elements,elem);
					count++;
				}
				
				
				proposal[offset+num_regions*NUM_RVAR]=d->type[elem];
				proposal[offset+num_regions*NUM_RVAR+1]=shift_box.l;
				proposal[offset+num_regions*NUM_RVAR+2]=shift_box.b;
				proposal[offset+num_regions*NUM_RVAR+3]=bb[elem].width();
				proposal[offset+num_regions*NUM_RVAR+4]=bb[elem].height();
				proposal[offset+num_regions*NUM_RVAR+5]=0;
				proposal[offset+num_regions*NUM_RVAR+6]=0;			
				
				proposal[elem*NUM_VAR+5]=num_regions;
				
				num_regions++;
				reg_change=1;
			}
			else
				continue;	
		}
		
		else if ((prop_select==REG_SPLIT) and (num_regions<MAX_ELEMENTS))
		{
			
	
			if (num_elements_reg[r]>1)
			{
				Box old_bb;
				old_bb.set(proposal[offset+r*NUM_RVAR+1], proposal[offset+r*NUM_RVAR+1]+proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+2], proposal[offset+r*NUM_RVAR+2]+proposal[offset+r*NUM_RVAR+4]);
				
				
				//split horizontally
				if (old_bb.width() > old_bb.height())
				{
					float mid_pt=(old_bb.l + old_bb.r)/2.0;
						
					proposal[offset+num_regions*NUM_RVAR+1]=mid_pt;
					proposal[offset+num_regions*NUM_RVAR+2]=proposal[offset+r*NUM_RVAR+2];
					proposal[offset+num_regions*NUM_RVAR+3]=proposal[offset+r*NUM_RVAR+3]/2.0;
					proposal[offset+num_regions*NUM_RVAR+4]=proposal[offset+r*NUM_RVAR+4];
					
					proposal[offset+r*NUM_RVAR+3]=proposal[offset+r*NUM_RVAR+3]/2.0;	
					
				}
				//split vertically
				else
				{
					float mid_pt=(old_bb.b + old_bb.t)/2.0;
		
					proposal[offset+num_regions*NUM_RVAR+1]=proposal[offset+r*NUM_RVAR+1];
					proposal[offset+num_regions*NUM_RVAR+2]=mid_pt;
					proposal[offset+num_regions*NUM_RVAR+3]=proposal[offset+r*NUM_RVAR+3];
					proposal[offset+num_regions*NUM_RVAR+4]=proposal[offset+r*NUM_RVAR+4]/2.0;
					
					proposal[offset+r*NUM_RVAR+4]=proposal[offset+r*NUM_RVAR+4]/2.0;	
				}
				
				proposal[offset+num_regions*NUM_RVAR]=proposal[offset+r*NUM_RVAR];
				proposal[offset+num_regions*NUM_RVAR+5]=proposal[offset+r*NUM_RVAR+5];
				proposal[offset+num_regions*NUM_RVAR+6]=proposal[offset+r*NUM_RVAR+6];
				
				
				flip_cnt=0;
				for (int i=0;i < ne;i++)
				{
					
					if ((int(proposal[i*NUM_VAR+5])==r) && (randu()<0.5))
					{
						proposal[i*NUM_VAR+5]=num_regions;	
						flip_cnt++;				
					}
				}
				//make sure we don't leave an empty region
				if (flip_cnt==num_elements_reg[r])
				{
					for (int i=0;i < ne;i++)
					{
						if (int(proposal[i*NUM_VAR+5])==num_regions)
						{
							proposal[i*NUM_VAR+5]=r;	
							break;	
						}				
					}
				}
				if (flip_cnt==0)
				{
					for (int i=0;i < ne;i++)
					{
						if (int(proposal[i*NUM_VAR+5])==r)
						{
							proposal[i*NUM_VAR+5]=num_regions;	
							break;	
						}				
					}
				}
				
				
				reg_change=1;
				num_regions++;
			}
			else
				continue;
		}
		else if (prop_select==REG_MERGE)
		{
			
			//assignElementRegion(d,proposal,curr_layout,elem,num_text_regions,num_graphic_regions);
			
			float can_merge=true;
			//for (int i=0;i < num_regions;i++)
			//	if ((i!=r) && (int(proposal[offset+r*NUM_RVAR])==int(proposal[offset+i*NUM_RVAR])))
			//		can_merge=true;
			
			if ((can_merge) && (num_regions>2))
			{
				//printf("can merge\n");
				
				int cnt=0;
				int r2=r;
				while ((r2==r) &&(cnt<100)) //|| (int(proposal[offset+r*NUM_RVAR])!=int(proposal[offset+r2*NUM_RVAR])))
				{
					r2=num_regions*randu();
					cnt++;
				}
				if (cnt==100)
				{
					printf("ERROR in selecting region %i, num regions %i\n",r,num_regions);
					
					return -1;					
				}
				
					
				if (r2<r)
				{
					int temp_r=r2;
					r2=r;
					r=temp_r;
				}
				
	
		        float loc_xdiff=-1*min((region_bb[r].r-region_bb[r2].l),(region_bb[r2].r-region_bb[r].l));
	        	float loc_ydiff=-1*min((region_bb[r].t-region_bb[r2].b),(region_bb[r2].t-region_bb[r].b));
				
				
				int out_of_bounds=0;
				if ((region_bb[r].l<0)||(region_bb[r2].l<0)||(region_bb[r].b<0)||(region_bb[r2].b<0)||(region_bb[r].r>d->width)||(region_bb[r2].r>d->width)||(region_bb[r].t>d->height)||(region_bb[r2].t>d->height))
	        		out_of_bounds=1;
	        		
	        	merge_r1=region_bb[r];
	        	merge_r2=region_bb[r2];
	        		
				//if ((max(loc_xdiff,loc_ydiff))<max(d->width,d->height)/5)
				//{
					
					//check for same type
				
					//if ((int(proposal[offset+r*NUM_RVAR])==2) && (randu()<0.05))
					//	display=true;
					
					//float new_width, new_height;
					
					//int x_align=int(proposal[offset+r*NUM_RVAR+5])%2;
					//int orientation=int(proposal[offset+r*NUM_RVAR+6]);
					
					proposal[offset+r*NUM_RVAR+1]=min(proposal[offset+r*NUM_RVAR+1],proposal[offset+r2*NUM_RVAR+1]);
					proposal[offset+r*NUM_RVAR+2]=min(proposal[offset+r*NUM_RVAR+2],proposal[offset+r2*NUM_RVAR+2]);
					
					
					proposal[offset+r*NUM_RVAR+3]=max(region_bb[r].r,region_bb[r2].r)-proposal[offset+r*NUM_RVAR+1];
					proposal[offset+r*NUM_RVAR+4]=max(region_bb[r].t,region_bb[r2].t)-proposal[offset+r*NUM_RVAR+2];
					
					
					if ((!out_of_bounds) && ((proposal[offset+r*NUM_RVAR+1]<0)||(proposal[offset+r*NUM_RVAR+2]<0)||(proposal[offset+r*NUM_RVAR+1]+proposal[offset+r*NUM_RVAR+3]>d->width)||(proposal[offset+r*NUM_RVAR+2]+proposal[offset+r*NUM_RVAR+4]>d->height)))
	        		{
						printf("ERROR. Region %i out of bounds after merge. region %.2f %.2f %.2f %.2f \n", r,prop_select,proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4]);
						printf("BBs were %.2f %.2f %.2f %.2f , %.2f %.2f %.2f %.2f\n",region_bb[r].l,region_bb[r].r,region_bb[r].b,region_bb[r].t,region_bb[r2].l,region_bb[r2].r,region_bb[r2].b,region_bb[r2].t);
						return -1;
	        		}
	        			
	        			
					
					
					if (display)
						printf("num regions %i try %i) %f %f %f %f %i) %f %f %f %f merged %f %f %f %f\n",num_regions, r,region_bb[r].l,region_bb[r].r,region_bb[r].b,region_bb[r].t,r2,region_bb[r2].l,region_bb[r2].r,region_bb[r2].b,region_bb[r2].t,  proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+1]+proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+2]+proposal[offset+r*NUM_RVAR+4]);
					
					
					
					if (randu()<0.5)
					{
						if (proposal[offset+r*NUM_RVAR+3]>proposal[offset+r*NUM_RVAR+4])
							proposal[offset+r*NUM_RVAR+6]=1;
						else
							proposal[offset+r*NUM_RVAR+6]=0;
					}
						
					//delete region r2
					//assign all elements from r2 to r
					//fill in r2 with the last region
					for (int i=0;i < ne;i++)
					{
						if ((int(proposal[i*NUM_VAR+5])==r2))
							proposal[i*NUM_VAR+5]=r;
						else if ((int(proposal[i*NUM_VAR+5])==num_regions-1)) 
							proposal[i*NUM_VAR+5]=r2;
					}
					
					
					//overwrite r2 with the last region, but only if they're different
					if (num_regions-1 != r2)
					{
						for (int i=0;i < NUM_RVAR;i++)
							proposal[offset+r2*NUM_RVAR+i]=proposal[offset+(num_regions-1)*NUM_RVAR+i];	
					}

					//blank the last region
					proposal[offset+(num_regions-1)*NUM_RVAR]=-1;		
					reg_change=-1;
					num_regions--;	

				//}
	
			}
			else
				continue;
			
		}
		else if(prop_select==REG_ORIENTATION)
		{
			
			//printf("was %i\n",int(proposal[offset+r*NUM_RVAR+6]));	
			
			if (int(proposal[offset+r*NUM_RVAR+6])==1)
			{
				proposal[offset+r*NUM_RVAR+6]=0;
		
			}
			else
			{
				proposal[offset+r*NUM_RVAR+6]=1;
				
		
				if (randu()<0.5)
				{
					for (int i=0;i < ne;i++)
					{	
						if (proposal[i*NUM_VAR+5]==r)
							proposal[i*NUM_VAR+2]=(region_bb[r].height()-2)/d->height;
						
					}
				}
				
				
			}
			
			//printf("set to %i\n",int(proposal[offset+r*NUM_RVAR+6]));	
		}
		else if(prop_select==REG_ALIGN)
		{
	
				
			int prop =int(proposal[offset+r*NUM_RVAR+5]);
			
			//flip x axis alignment
			if (randu()<0.5)
			{
				if (prop>2)
					proposal[offset+r*NUM_RVAR+5]=prop-3;
				else
					proposal[offset+r*NUM_RVAR+5]=prop+3;
					
				//if (debug)
				//	printf("1 %i\n", int(proposal[offset+r*NUM_RVAR+5]));
			}
			//flip y axis alignment
			else
			{
				if (prop>2)
					proposal[offset+r*NUM_RVAR+5]=int(3*randu())+3;
				else 
					proposal[offset+r*NUM_RVAR+5]=int(3*randu());
					
				//if (debug)
				//	printf("2 %i\n", int(proposal[offset+r*NUM_RVAR+5]));
			}
			
	
		}
		else if ((prop_select==REG_ELEMENT_HEIGHT))
		{
			float height_variance=0.05;
		
			if (d->type[elem]!=1)
				height_variance=height_variance*2;
		
			float new_height=0;
			int cnt=0;
			while ((cnt<100)&&(new_height<0.01))
			{
				float2 rands=randn();
				new_height=proposal[elem*NUM_VAR+2]+rands.y*height_variance;
				cnt++;
			}
			if (cnt==100)
			{
				printf("ERROR in setting new height\n");
				cnt++;
			}
	
			proposal[elem*NUM_VAR+2]=new_height;	
			
			height=(proposal[NUM_VAR*elem+2]*d->height);
			width=(height/aspect_ratio[elem]);
			xp=(proposal[NUM_VAR*elem]*d->width);
			yp=(proposal[NUM_VAR*elem+1]*d->height);
	
			bb[elem].set(round(xp+(width*d->bb_left[elem])), round(xp+(width*d->bb_right[elem])), round(yp + (height*d->bb_bottom[elem])), round(yp + (height*d->bb_top[elem])));
			
			
			cnt=0;
			for (int i=0;i < ne;i++)
				if (proposal[i*NUM_VAR+5]==proposal[elem*NUM_VAR+5])
					cnt++;
					
			if (cnt==1)
			{
				int r=proposal[elem*NUM_VAR+5];
				proposal[offset+r*NUM_RVAR+1]=bb[elem].l; 
				proposal[offset+r*NUM_RVAR+2]=bb[elem].b;
				proposal[offset+r*NUM_RVAR+3]=bb[elem].width();	
				proposal[offset+r*NUM_RVAR+4]=bb[elem].height();	
			}
				
		}
		else if (prop_select==REG_SWITCH_ALTERNATE)
		{
			
			int old_alt=proposal[elem*NUM_VAR+6];
			
			if (d->num_alt[elem]>0)
			{
				
				int alt=(d->num_alt[elem]*randu());
				
				float scale=d->alt_num_lines[elem*MAX_ALT+alt]/d->alt_num_lines[elem*MAX_ALT+old_alt];
				
				if ((d->alt_num_lines[elem*MAX_ALT+alt]>2)&&(d->alt_aspect_ratio[elem*MAX_ALT+alt]>0.5))
					continue;
				
				float scaled_height=(bb[elem].t-bb[elem].b)*scale;
				float scaled_width=scaled_height/d->alt_aspect_ratio[elem*MAX_ALT+alt];
				
				if ((scaled_width < 3) or (scaled_height < 3))
					continue;
					
				proposal[NUM_VAR*elem+2]=proposal[NUM_VAR*elem+2]*scale;
	
				
				aspect_ratio[elem]=d->alt_aspect_ratio[elem*MAX_ALT+alt];
				//num_lines[elem]=d->alt_num_lines[elem*MAX_ALT+alt];
	
	
				height=(proposal[NUM_VAR*elem+2]*d->height);
				width=(height/aspect_ratio[elem]);
				xp=(proposal[NUM_VAR*elem]*d->width);
				yp=(proposal[NUM_VAR*elem+1]*d->height);
		
				bb[elem].set(round(xp+(width*d->bb_left[elem])), round(xp+(width*d->bb_right[elem])), round(yp + (height*d->bb_bottom[elem])), round(yp + (height*d->bb_top[elem])));
				
				//printf("changed alt from %i to %i. num alt %i \n",int(proposal[elem*NUM_VAR+6]),alt,d->num_alt[elem]);
				proposal[elem*NUM_VAR+6]=alt;
			}
			else
				continue;
		
		}
		else if (prop_select==REG_ELEMENT_ORDER)
		{
			
			if (ne>1)
			{
				int other_elem=(ne)*randu();
					
				while (other_elem==elem)
				{
					other_elem=(ne)*randu();
				}
			
				float temp=proposal[elem*NUM_VAR+7];
				proposal[elem*NUM_VAR+7]=proposal[other_elem*NUM_VAR+7];
				proposal[other_elem*NUM_VAR+7]=temp;
			}	
			else
				continue;
		}
		
		
			
		for (int i=0;i<num_regions;i++)
			region_bb[i].set(proposal[offset+i*NUM_RVAR+1], proposal[offset+i*NUM_RVAR+1]+ proposal[offset+i*NUM_RVAR+3], proposal[offset+i*NUM_RVAR+2],proposal[offset+i*NUM_RVAR+2]+proposal[offset+i*NUM_RVAR+4]);
		
		

		
		bool intersect=false;
		for (int i=0;i<num_regions;i++)
		for (int j=i+1;j<num_regions;j++)
		{
			//(i!=j)&& &&(((region_bb[i].l>region_bb[j].l) && (region_bb[i].r<region_bb[j].r)) || ((region_bb[i].b>region_bb[j].b) && (region_bb[i].t<region_bb[j].t)))
			if ((anyBoxIntersection(region_bb[i],region_bb[j])) )
				intersect=true;
			
		}
		
			
		if ((not intersect) && (region_bb[r].l>0)&&(region_bb[r].b>0)&&(region_bb[r].r<d->width)&&(region_bb[r].t<d->height))
		{
			break;	
			//printf("ERROR: region out of bounds with prop select %i\n",prop_select);
			//printf(" region l/r: %.2f - %.2f, b/t: %.2f - %.2f \n",proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+1]+proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+2]+proposal[offset+r*NUM_RVAR+4]);		
		}
			
			
	}
	
	
	
 	//if (randu()<0.001)	
 	//	printf("prop select %i\n",prop_select);
			
	int modified_reg=r;
			
			
	//ASSERTION: the number of regions is correct
	int num_regions_check=0;
	for (int reg=0;reg<ne;reg++)
	{
		if (int(proposal[offset+reg*NUM_RVAR])!=-1)
			num_regions_check++;		
	}
	
	if (num_regions_check!=num_regions)
	{
	
		printf(" ERROR. num_regions_check %i, num_regions %i. m %i. Proposal %i. modified region %i region change %i. flipped %i of %i. num text %i num graphic %i\n",num_regions_check,num_regions,m,prop_select,modified_reg,reg_change,flip_cnt,num_elements_reg[modified_reg],num_text_regions, num_graphic_regions);
		
		return -1;
	}
	
			
			
			
	//put elements in region based on parameters
	for (int r=0;r<num_regions;r++)
	{
	
		//region_bb[r].set(proposal[offset+r*NUM_RVAR+1], proposal[offset+r*NUM_RVAR+2], proposal[offset+r*NUM_RVAR+3], proposal[offset+r*NUM_RVAR+4]);
		
		//if ((region_bb[r].height()<=0) || (region_bb[r].width()<=0))
		//	printf("t %i error in region %f %f %f %f\n",t, region_bb[r].l,region_bb[r].r,region_bb[r].b,region_bb[r].t);
		
		//if (debug)
		//	printf("region %i: %f %f %f %f\n",r, region_bb[r].l,region_bb[r].r,region_bb[r].b,region_bb[r].t);
		
		float reg_width=proposal[offset+r*NUM_RVAR+3];	
		float reg_height=proposal[offset+r*NUM_RVAR+4];	
		int x_align=int(proposal[offset+r*NUM_RVAR+5])%3;
		int y_align=int(proposal[offset+r*NUM_RVAR+5]>2);
		int orientation=proposal[offset+r*NUM_RVAR+6];	
		
		//orientation=1;
		
		
		
		float element_heights=0;
		float element_widths=0;
		int elem_cnt=0;
		
		for (int s=0;s < ne;s++)
		{
			int i=proposal[s*NUM_VAR+7];
			//int i=s;
			
			if (proposal[i*NUM_VAR+5]==r)
			{
				element_heights+=bb[i].height();
				element_widths+=bb[i].width();
				elem_cnt++;
			}
		}
		
		float scale=1;
		
		if (orientation==0)
		{
			scale=min(reg_height/element_heights,1.0);	
			//if (scale <0.99)
			//	printf("scale %f %i, elem height %f, reg height %f\n",scale,prop_select,element_heights,reg_height);
		
		}
		else
		{
			scale=min(reg_width/element_widths,1.0);
			
			//if (scale <0.99)
			//	printf("scale %f %i, elem width %f, reg width %f\n",scale,prop_select,element_widths,reg_width);
		
		}		
		
		
		if (elem_cnt==0)
			continue;		
					
		float spacing=0;
			
		if ((elem_cnt>1) && (orientation==0))
			spacing=(reg_height-element_heights*scale)/(elem_cnt-1);
		else if ((elem_cnt>1) && (orientation==1))
			spacing=(reg_width-element_widths*scale)/(elem_cnt-1);
		
		spacing = max(spacing,1.0);
		
		//if (spacing >400)
		//	printf("spacing %.3f, reg width/height %.3f %.3f, element heights %.3f, elem_cnt %i\n",spacing,reg_width,reg_height,element_heights,elem_cnt);
		
		
		float curr_height=0, curr_width=0;
		
		Box new_region;
		new_region.set(d->width,0,d->height,0);
			
		for (int s=0;s < ne;s++)
		{
			int i=proposal[s*NUM_VAR+7];
			//int i=s;

			
			if (proposal[i*NUM_VAR+5]==r)
			{
					
					
				proposal[i*NUM_VAR+2]=proposal[i*NUM_VAR+2]*scale;
				
				float new_width=bb[i].width()*scale;
				float new_height=bb[i].height()*scale;	
					
				float elem_scale=reg_width/new_width;
				
				if (new_width>reg_width)
				{
					
			
					new_width=new_width*elem_scale;
					new_height=new_height*elem_scale;
					proposal[i*NUM_VAR+2]=proposal[i*NUM_VAR+2]*elem_scale;
					new_width=reg_width;
					
					if (elem_scale>5)
					{
						
						printf("scale is large %.3f\n. reg width %.3f and new width %.3f",elem_scale,reg_width, new_width);
					}				
				}

			

				
				
				
				if (orientation==0)
				{
					proposal[i*NUM_VAR+1]=(proposal[offset+r*NUM_RVAR+2]+curr_height)/d->height;				
					
					curr_height+=proposal[i*NUM_VAR+2]*d->height+spacing;
	
					if (x_align==0)
						proposal[i*NUM_VAR]=proposal[offset+r*NUM_RVAR+1]/d->width;
					else if (x_align==1)
						proposal[i*NUM_VAR]=(proposal[offset+r*NUM_RVAR+1]+reg_width/2.0-(new_width)/2.0)/d->width;
					else if (x_align==2)
						proposal[i*NUM_VAR]=(proposal[offset+r*NUM_RVAR+1]+reg_width-new_width)/d->width;
							
				}
				else
				{
					proposal[i*NUM_VAR]=(proposal[offset+r*NUM_RVAR+1]+curr_width)/d->width;	
					
					curr_width+=new_width+spacing;
					
					if (y_align==0)
						proposal[i*NUM_VAR+1]=proposal[offset+r*NUM_RVAR+2]/d->height;
					else if (y_align==1)
						proposal[i*NUM_VAR+1]=(proposal[offset+r*NUM_RVAR+2]+reg_height/2.0-(new_height)/2.0)/d->height;
					else if (y_align==2)
						proposal[i*NUM_VAR+1]=(proposal[offset+r*NUM_RVAR+2]+reg_height-new_height)/d->height;			

				}
				
				
				if (((abs(proposal[NUM_VAR*i])>2) || (abs(proposal[NUM_VAR*i+1])>2)) && (m<MAX_PROPOSAL_TRIES))
				{
					int r= proposal[NUM_VAR*i+5];
					printf("ERROR: element %i, x/y: %.3f %.3f, with prop select %i (m %i) orientation %i, xy align %i %i, curr height %.3f width %.3f ,spacing %.3f, scale %.3f, new width/height %.3f %.3f  \n",i,proposal[NUM_VAR*i],proposal[NUM_VAR*i+1], prop_select,m,orientation,x_align,y_align,curr_height,curr_width,spacing,elem_scale,new_width,new_height);
				
					printf(" region l/r: %.2f - %.2f, b/t: %.2f - %.2f \n",proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+1]+proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+2]+proposal[offset+r*NUM_RVAR+4]);
					
					if (prop_select==10)
						printf(" merged region1 l/r: %.2f - %.2f, b/t: %.2f - %.2f, with r2 l/r: %.2f - %.2f, b/t: %.2f - %.2f\n",merge_r1.l,merge_r1.r,merge_r1.b,merge_r1.t,merge_r2.l,merge_r2.r,merge_r2.b,merge_r2.t );
				
				
					printf(" reg width/height  %.3f %.3f, element sum width/height %.3f %.3f \n",reg_width, reg_height,element_widths, element_heights);
					
					return -1;
				}
				//if ((d->fixed_regions) && (proposal[offset+r*NUM_RVAR]==1))
				//{
				//	printf("element %i with height %f, curr_height %f, spacing %f\n",i,proposal[i*NUM_VAR+2],curr_height,spacing);
				//}
				
					
				new_region.l=min(new_region.l,round(proposal[i*NUM_VAR]*d->width));
				new_region.r=max(new_region.r,round(proposal[i*NUM_VAR]*d->width+new_width));
				
				new_region.b=min(new_region.b,round(proposal[i*NUM_VAR+1]*d->height));
				new_region.t=max(new_region.t,round((proposal[i*NUM_VAR+1]+proposal[i*NUM_VAR+2])*d->height));		
				
				
				if (d->fixed_alignment[i]<0.001)
				{
					proposal[i*NUM_VAR+3]=x_align;
				}
					
				if ((display) && (proposal[offset+r*NUM_RVAR]==2))
					printf("setting element %i: %.3f %.3f %.3f, scale %.3f\n",i,proposal[i*NUM_VAR],proposal[i*NUM_VAR+1],proposal[i*NUM_VAR+2],scale);
				//}
			}
		}
		
		if (!(d->fixed_regions))
		{
			
			
			if (scale<1)
			{
				proposal[offset+r*NUM_RVAR+3]+=3;
				proposal[offset+r*NUM_RVAR+4]+=3;
			}
		
		
			proposal[offset+r*NUM_RVAR+1]=new_region.l;
			proposal[offset+r*NUM_RVAR+2]=new_region.b;
			proposal[offset+r*NUM_RVAR+3]=new_region.width();
			proposal[offset+r*NUM_RVAR+4]=new_region.height();
			
			if ((display) && (int(proposal[offset+r*NUM_RVAR])==2))
				printf("final regions %f %f %f %f. num regions %i\n",proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4],num_regions);
			
		}

	}
	

	//assert that the assigned regions all exist
	
	for (int i=0;i<ne;i++)
	{
		int r= proposal[NUM_VAR*i+5];
		
		if (int(proposal[offset+r*NUM_RVAR])==-1)
		{
			printf("ERROR. Assigned region doesn't exist. element %i region %i with prop select %i. region %.2f %.2f %.2f %.2f \n",i, r,prop_select,proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4]);
			return -1;
		}
	}


	for (int i=0;i<ne;i++)
	{
		int r= proposal[NUM_VAR*i+5];
		
		float height=(proposal[NUM_VAR*i+2]*d->height);
		float width=(height/aspect_ratio[i]);
		float xp=(proposal[NUM_VAR*i]*d->width);
		float yp=(proposal[NUM_VAR*i+1]*d->height);

		bb[i].set(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
		//bb[i].set(xp, xp+(width), yp, yp + (height));
		
		if ((bb[i].l>=bb[i].r)|| (bb[i].b>=bb[i].t))
		{
			printf("ERROR. output bounding box for element %i with prop_select %i l/r: %f %f b/t: %f %f\n",i,prop_select,bb[i].l,bb[i].r,bb[i].b,bb[i].t);
			
			printf("\t region l/b: %f %f w/h: %f %f\n",proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4]);
			return -1;
		}	
	}
	
	
	//assert that there are no empty regions
	for (int r=0;r<ne;r++)
	{
		
		if (int(proposal[offset+r*NUM_RVAR])!=-1)
		{
			bool empty_region=true;
			for (int i=0;i<ne;i++)
			{
				if (int(proposal[NUM_VAR*i+5])==r)
					empty_region=false;
				
			}
			
			if (empty_region)
			{
				printf("ERROR. No elements for region %i (type %i): %.2f %.2f %.2f %.2f  \n",r,int(proposal[offset+r*NUM_RVAR]), proposal[offset+r*NUM_RVAR+1],proposal[offset+r*NUM_RVAR+2],proposal[offset+r*NUM_RVAR+3],proposal[offset+r*NUM_RVAR+4]);
				
				printf(" Proposal %i. modified region %i region change %i. flipped %i of %i. num regions %i \n",prop_select,modified_reg,reg_change,flip_cnt,num_elements_reg[modified_reg],num_regions);
				
				
				return -1;
			}
		}
	}
		
	return prop_select;
}
*/


	/*
__device__ void assignElementRegion(Design *d, float *proposal, float *curr_layout, int elem, int num_text_regions, int num_graphic_regions)
{
		int region=0;
		if (d->type[elem]==1)
			region=((float)num_text_regions*randu());
		else
			region=((float)num_graphic_regions*randu());
		
		int cnt=0;
		
		for(int j=0;j< d->num_elements;j++)
		{
			if (curr_layout[d->num_elements*NUM_VAR+j*NUM_RVAR]==d->type[elem])
			{
				if (cnt==region)
				{
					proposal[elem*NUM_VAR+5]=j;
					break;
				}
				cnt++;						
			}
		}
}

*/