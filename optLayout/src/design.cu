#include "hip/hip_runtime.h"

#include "design.cuh"





__device__ bool anyBoxIntersection(Box b1, Box b2)
{
	if ((b1.l>=b2.r) or (b1.r<=b2.l) or (b1.t<=b2.b) or (b1.b>=b2.t))
		return false;

	return true;
}



__device__ Box getBoxIntersection(Box b1, Box b2)
{
	Box b;
	b.set(0,0,0,0);

	float xoverlap=min((b1.r-b2.l),(b2.r-b1.l));
	float yoverlap=min((b1.t-b2.b),(b2.t-b1.b));

	if (xoverlap>0)
	{
		b.l=max(b1.l,b2.l);
		b.r=min(b1.r,b2.r);
	}
	if (yoverlap>0)
	{
		b.b=max(b1.b,b2.b);
		b.t=min(b1.t,b2.t);
	}
	return b;
}


__device__ bool checkBoundingBoxOverlap(Box b1,Box *other_boxes, int num_boxes,int ignore_box)
{

	for (int i=0;i< num_boxes;i++)
	{
		if (i==ignore_box)
			continue;

		Box b2=other_boxes[i];

		if (anyBoxIntersection(b1,b2))
		{
			return true;
		}
	}
	return false;
}


char *getSubstring(char *string, char *start_tag, char *end_tag)
{

	char *str1=strstr(string,start_tag);
	char *str2=strstr(string,end_tag);

	if (!str1)
	{
		cout << "Didnt find tag " << start_tag<< endl;
		return 0;
	}
	if (!str2)
	{
		cout << "Didnt find tag " << end_tag<< endl;
		return 0;
	}

	int num_char=str2-(str1+strlen(start_tag));

	if (num_char<0)
	{
		cout << "num_char " << num_char<< endl;
		return 0;
	}
	char *substring=(char *) malloc(num_char);
	memset( substring, '\0', sizeof(char)*num_char );
	strncpy(substring,str1+strlen(start_tag),num_char );
	substring[num_char]='\0';
	return substring;

}

char *extractXMLElement(char *string, char *tag)
{

	int tag_len=strlen(tag);
	char *start_tag=(char *)malloc(tag_len+3);
	char *end_tag=(char *)malloc(tag_len+4);
	snprintf(start_tag,tag_len+3, "<%s>>",tag);
	snprintf(end_tag,tag_len+4,"</%s>>",tag);
	return getSubstring(string, start_tag, end_tag);


}


void printLayout(char *str,Design *d, float *layout,  float energy)
{


	int n=sprintf(str,"%i\n%s\n%i,%i\n%i\n",d->layout_counter,d->name,int(d->width),int(d->height),d->num_elements);
	str=str+n;


	for (int i=0;i<d->num_elements;i++)
	{
		int x = int(round(layout[i*NUM_VAR]*d->width));
		int y = int(round(layout[i*NUM_VAR+1]*d->height));
		int h = int(round(layout[i*NUM_VAR+2]*d->height));
		int a = int(round(layout[i*NUM_VAR+3]));
		float f = layout[i*NUM_VAR+4];
		int r = int(round(layout[i*NUM_VAR+5]));
		int alt = int(round(layout[i*NUM_VAR+6]));

		if ((x<-d->width) or (x>2*d->width) or (y<-d->height) or (y>2*d->height))
			printf("ERROR in layout %i %i %i %i\n", x,y,h,a);

		
		n=sprintf(str,"%i,%i,%i,%i,%.2f,%i,%i\n",x,y,h,a,f,r,alt);
		str=str+n;
	}
	
	n=sprintf(str,"E:%.3f\n",energy);
	str=str+n;

	//printf("sending: %s\n",str)	;
}


void writeLayoutToFile(Design *d, char *layout_str, char *filename)
{

	//printf("Writing layout out to file %s\n",filename);
    FILE *fp = NULL;
	fp = fopen(filename, "w");
	fprintf(fp,"%s",layout_str);
	fclose (fp);

}



float *parseLayout(Design *d,  char *str,int *num_regions, int *layout_counter)
{


	int width;
	int height;
	char name[100];
	int num_elements;
	sscanf(str,"%d\n",layout_counter);
	str=strchr(str,'\n')+1;
	sscanf(str,"%s\n",name);
	str=strchr(str,'\n')+1;
	
	printf("loading design %s\n",name);
	if (strcmp(name, "quit")==0)
	{
		printf("quitting\n");
		exit(-1);
	}
	
	

	sscanf(str,"%d,%d\n%d\n",&width,&height,&num_elements);
	str=strchr(str,'\n')+1;
	str=strchr(str,'\n')+1;
	//char background[100];
	//fscanf(fp,"%s\n",background);

	if ((num_elements!=d->num_elements) or (strcmp(name, d->name)!=0) or (width!=int(d->width)) or (height!=int(d->height)))
	{
		printf("Layout doesn't match design.\n");
		printf("Layout: %s, width: %d, height: %d, elem %d\n",name, width, height,num_elements);
		printf("Design: %s, width: %d, height: %d, elem %d\n", d->name,  int(d->width), int( d->height),d->num_elements);
		return 0;
	}
	float *new_layout=(float *)malloc(d->layout_size*sizeof(float));

	float x, y, h, align, fix,alt,region;

	for (int i=0;i<d->num_elements;i++)
	{
		sscanf(str,"%f,%f,%f,%f,%f,%f,%f\n",&x,&y,&h,&align,&fix,&region,&alt);
		str=strchr(str,'\n')+1;
		printf("Read in %.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n",x,y,h,align,fix,alt,region);
	

		new_layout[i*NUM_VAR]=x/d->width;
		new_layout[i*NUM_VAR+1]=y/d->height;
		new_layout[i*NUM_VAR+2]=h/d->height;
		new_layout[i*NUM_VAR+3]=align;
		new_layout[i*NUM_VAR+4]=fix;
		new_layout[i*NUM_VAR+5]=-1;
		new_layout[i*NUM_VAR+6]=alt;
		new_layout[i*NUM_VAR+7]=i;
		//printf("%i:%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n",i,new_layout[i*NUM_VAR],new_layout[i*NUM_VAR+1],new_layout[i*NUM_VAR+2],new_layout[i*NUM_VAR+3],new_layout[i*NUM_VAR+4],new_layout[i*NUM_VAR+6]);
	
	
	}
	
	for (int i=d->num_elements*NUM_VAR;i<d->layout_size;i++)
		new_layout[i]=-1;
	
	int num_constraints;
	sscanf(str,"%d constraints\n",&(num_constraints));
	str=strchr(str,'\n')+1;
	
	printf("num constraints %i\n",num_constraints);
	d->num_constraints=num_constraints;
	if (d->num_constraints>0)
	{
		//d->constraints=(int*) malloc(d->num_constraints*NUM_AVAR*sizeof(int));
		
		int elem_id,align_count,align_type,other_elem;
		for(int i=0;i< d->num_constraints;i++)
		{
			sscanf(str,"%d,%d,%d",&elem_id,&align_type,&align_count);
			
			str=strchr(str,',')+1;
			str=strchr(str,',')+1;
			
			if (align_count>0)
				str=strchr(str,',');
			
			int elem_idx=-1;
			for (int k=0;k<d->num_elements;k++)
			{
				if (d->id[k]==elem_id)
					elem_idx=k;
			}
			
			if (elem_idx==-1)
			{
				printf("ERROR, element index -1\n");
				return 0;
			}
			printf("alignment line. elem id: %i, idx %i, type: %i, count %i\n",elem_id,elem_idx,align_type,align_count);
			
			
			d->constraints[i*NUM_AVAR]=elem_idx;
			d->constraints[i*NUM_AVAR+1]=align_type;
			d->constraints[i*NUM_AVAR+2]=align_count;
			
			for (int j=0;j<align_count;j++)
			{
				sscanf(str,",%d",&other_elem);
				str=strchr(str,',')+1;

				
				elem_idx=-1;
				for (int k=0;k<d->num_elements;k++)
				{
					if (d->id[k]==other_elem)
						elem_idx=k;
				}
				
				if (elem_idx==-1)
				{
					printf("ERROR, element index -1\n");
					return 0;
				}			
				d->constraints[i*NUM_AVAR+3+j]=elem_idx;
				printf("\t other element: %i, idx %i\n",other_elem,elem_idx);
			}
			str=strchr(str,'\n')+1;
		}	
	}
	
	
	
	
	*num_regions=0;
	sscanf(str,"%d regions\n",num_regions);
	str=strchr(str,'\n')+1;
	
	printf("num regions %i\n",*num_regions);
	
	
	//float height,width,xp,yp;
	int cnt=0;
	for(int i=0;i< d->num_elements;i++)
	{				
		if (new_layout[i*NUM_VAR+4]>FIX_LAYOUT_THRESH)
		{
			new_layout[i*NUM_VAR+5]=-1;
			continue;
		}
		
		new_layout[i*NUM_VAR+5]=cnt;

		
		cnt++;
	}
		
	
	
	
	return new_layout;


}




float *readLayoutFromFile(Design *d,  char *filename,int *num_regions, int *layout_counter)
{

	printf("Reading layout from file %s\n",filename);
    FILE *fp = NULL;
	fp = fopen(filename, "r");

	if (fp<0)
	{
		printf("error reading layout file\n");
		return 0;
	}
	int width;
	int height;
	char name[100];
	int num_elements;
	fscanf(fp,"%d\n",layout_counter);
	fscanf(fp,"%s\n",name);
	
	printf("loading design %s\n",name);
	if (strcmp(name, "quit")==0)
	{
		printf("quitting\n");
		exit(-1);
	}
	

	fscanf(fp,"%d,%d\n%d\n",&width,&height,&num_elements);
	
	
	//char background[100];
	//fscanf(fp,"%s\n",background);

	if ((num_elements!=d->num_elements) or (strcmp(name, d->name)!=0) or (width!=int(d->width)) or (height!=int(d->height)))
	{
		printf("Layout doesn't match design.\n");
		printf("Layout: %s, width: %d, height: %d, elem %d\n",name, width, height,num_elements);
		printf("Design: %s, width: %d, height: %d, elem %d\n", d->name,  int(d->width), int( d->height),d->num_elements);
		return 0;
	}
	float *new_layout=(float *)malloc(d->layout_size*sizeof(float));

	float x, y, h, align, fix,alt,region;

	for (int i=0;i<d->num_elements;i++)
	{
		fscanf(fp,"%f,%f,%f,%f,%f,%f,%f\n",&x,&y,&h,&align,&fix,&region,&alt);
		
		printf("Read in %.2f,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n",x,y,h,align,fix,alt,region);
	

		new_layout[i*NUM_VAR]=x/d->width;
		new_layout[i*NUM_VAR+1]=y/d->height;
		new_layout[i*NUM_VAR+2]=h/d->height;
		new_layout[i*NUM_VAR+3]=align;
		new_layout[i*NUM_VAR+4]=fix;
		new_layout[i*NUM_VAR+5]=-1;
		new_layout[i*NUM_VAR+6]=alt;
		new_layout[i*NUM_VAR+7]=i;
		//printf("%i:%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n",i,new_layout[i*NUM_VAR],new_layout[i*NUM_VAR+1],new_layout[i*NUM_VAR+2],new_layout[i*NUM_VAR+3],new_layout[i*NUM_VAR+4],new_layout[i*NUM_VAR+6]);
	
	
	}
	
	for (int i=d->num_elements*NUM_VAR;i<d->layout_size;i++)
		new_layout[i]=-1;
	
	int num_constraints;
	fscanf(fp,"%d lines\n",&(num_constraints));
	
	printf("num align lines %i\n",num_constraints);
	d->num_constraints=num_constraints;
	if (d->num_constraints>0)
	{
		//d->constraints=(int*) malloc(d->num_constraints*NUM_AVAR*sizeof(int));
		
		int elem_id,align_count,align_type,other_elem;
		for(int i=0;i< d->num_constraints;i++)
		{
			fscanf(fp,"%d,%d,%d",&elem_id,&align_type,&align_count);
			printf("alignment line id: %i type: %i\n",elem_id,align_type);
			
			int elem_idx=-1;
			for (int k=0;k<d->num_elements;k++)
			{
				if (d->id[k]==elem_id)
					elem_idx=k;
			}
			
			if (elem_idx==-1)
			{
				printf("ERROR, element index -1\n");
				return 0;
			}
			
			d->constraints[i*NUM_AVAR]=elem_idx;
			d->constraints[i*NUM_AVAR+1]=align_type;
			d->constraints[i*NUM_AVAR+2]=align_count;
			
			for (int j=0;j<align_count;j++)
			{
				fscanf(fp,",%d",&other_elem);
				
				elem_idx=-1;
				for (int k=0;k<d->num_elements;k++)
				{
					if (d->id[k]==other_elem)
						elem_idx=k;
				}
				
				if (elem_idx==-1)
				{
					printf("ERROR, element index -1\n");
					return 0;
				}			
				d->constraints[i*NUM_AVAR+3+j]=elem_idx;
				printf("\t other element: %i\n",other_elem);
			}
			fscanf(fp,"\n");
		}	
	}
	
	
	*num_regions=0;
	fscanf(fp,"%d regions\n",num_regions);
	
	
	printf("num regions %i\n",*num_regions);
	
	/*
	int offset=d->num_elements*NUM_VAR;
		
	if (*num_regions>0)
	{

		float type, l,b,w,h;
		for(int i=0;i< *num_regions;i++)
		{
			fscanf(fp,"%f,%f,%f,%f,%f\n",&type,&l,&b,&w,&h);
				
			new_layout[offset+i*NUM_RVAR]=type;
			new_layout[offset+i*NUM_RVAR+1]=max(l,0.0);
			new_layout[offset+i*NUM_RVAR+2]=max(b,0.0);			
			new_layout[offset+i*NUM_RVAR+3]=w;
			new_layout[offset+i*NUM_RVAR+4]=h;	
			
	
			new_layout[offset+i*NUM_RVAR+5]=1.0;	
			
			if (w>1.5*h)
				new_layout[offset+i*NUM_RVAR+6]=1.0;
			else
				new_layout[offset+i*NUM_RVAR+6]=0.0;
			
		}
	}
	
	else
	{
		*/
	
	//float height,width,xp,yp;
	int cnt=0;
	for(int i=0;i< d->num_elements;i++)
	{				
		if (new_layout[i*NUM_VAR+4]>FIX_LAYOUT_THRESH)
		{
			new_layout[i*NUM_VAR+5]=-1;
			continue;
		}
		
		new_layout[i*NUM_VAR+5]=cnt;
		/*
		height=round(new_layout[NUM_VAR*i+2]*d->height);
		width=round(height/d->aspect_ratio[i]);
		xp=round(new_layout[NUM_VAR*i]*d->width);
		yp=round(new_layout[NUM_VAR*i+1]*d->height);
		
		
		new_layout[offset+cnt*NUM_RVAR]=d->type[i];
		new_layout[offset+cnt*NUM_RVAR+1]=round(xp+(width*d->bb_left[i])); 
		new_layout[offset+cnt*NUM_RVAR+2]=round(yp+(height*d->bb_bottom[i]));
		new_layout[offset+cnt*NUM_RVAR+3]=width;
		new_layout[offset+cnt*NUM_RVAR+4]=height;
		new_layout[offset+cnt*NUM_RVAR+5]=1.0;	
		new_layout[offset+cnt*NUM_RVAR+6]=0.0;
		*/
		
		cnt++;
	}
		
	//}
	
	
	
	fclose (fp);
	return new_layout;


}


Design *loadDesignFromFile(char *filename,bool interactive_mode)
{



	printf("Reading design from file %s\n",filename);
    FILE *fp = NULL;
	fp = fopen(filename, "r");

	if (fp<0)
	{
		printf("error reading design file\n");
		return 0;
	}
	
	Design *d = (Design*)malloc( sizeof(Design) );
	d->name=(char *)malloc(1000*sizeof(char));	
	
	//int width;
	//int height;
	int ne;
	int layout_counter;

	fscanf(fp,"%d\n",&layout_counter);
	fscanf(fp,"%s\n",d->name);
	fscanf(fp,"%f,%f\n%d\n",&(d->width),&(d->height),&ne);

	char background[500];
	fscanf(fp,"%s\n",background);

	cout << "Design Name " << d->name  << " Width "<< d->width << " Height "<< d->height << " Num elements "<< ne << endl;




	d->num_elements=ne;
	d->layout_counter=layout_counter;
	//cout << "Design Number of Elements " << d->num_elements << endl;


	d->id=(int *)malloc(d->num_elements*sizeof(int));
	d->importance=(int *)malloc(d->num_elements*sizeof(int));
	d->type=(int *)malloc(d->num_elements*sizeof(int));
	d->bb_left=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_right=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_bottom=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_top=(float *)malloc(d->num_elements*sizeof(float));

	d->tight_bb_left=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_right=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_bottom=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_top=(float *)malloc(d->num_elements*sizeof(float));

	d->num_lines=(int *)malloc(d->num_elements*sizeof(int));
	d->aspect_ratio=(float *)malloc(d->num_elements*sizeof(float));
	d->line_gap=(float *)malloc(d->num_elements*sizeof(float));
	d->group_id=(int *)malloc(d->num_elements*sizeof(int));
	d->alignment=(int *)malloc(d->num_elements*sizeof(int));
	d->fixed_alignment=(int *)malloc(d->num_elements*sizeof(int));
	
	d->num_constraints=0;
	d->constraints=(int*) malloc(MAX_ELEMENTS*3*NUM_AVAR*sizeof(int));
	
	//d->alt_alignment=(int **)malloc(d->num_elements*sizeof(int *));

	//d->layout_size=d->num_elements*NUM_VAR + MAX_ELEMENTS*NUM_RVAR;
	d->layout_size=d->num_elements*NUM_VAR;
	d->layout=(float *)malloc(d->layout_size*sizeof(float));
	d->init_layout=(float *)malloc(d->layout_size*sizeof(float));
	d->check_layout=(float *)malloc(d->layout_size*sizeof(float));
	d->check_layout_distances=(float *)malloc(MAX_ELEMENTS*sizeof(float));

	d->check_layout_exists=interactive_mode;
	d->fixed_regions=false;
	d->region_proposals=false;

	d->num_alt=(int *)malloc(d->num_elements*sizeof(int));
	d->alt_aspect_ratio=(float *)malloc(d->num_elements*MAX_ALT*sizeof(float));
	d->alt_num_lines=(int *)malloc(d->num_elements*MAX_ALT*sizeof(int));
	
	d->optional=(int *)malloc(d->num_elements*sizeof(int));
	
	if ((d->num_elements<0) || (d->num_elements>MAX_ELEMENTS))
	{
		free(d);
		
		printf("error reading design file\n");
		return 0;
	}

	for (int i=0 ; i < d->num_elements ; i++)
	{

		int x,y,h,align,alt;
		float fix;
		char fname[100];

		fscanf(fp,"%d,%d,%d,%d,%d,%f,%d,%d,%d,%d,%f,%d,%d,%f,%f,%f,%f,%f,%s\n",&(d->id[i]),&(d->type[i]),&(d->importance[i]),&(d->num_lines[i]),&(d->group_id[i]),&(d->aspect_ratio[i]),&x,&y,&h,&align,&fix,&alt,&(d->fixed_alignment[i]),&(d->optional[i]),&(d->bb_left[i]),&(d->bb_right[i]),&(d->bb_bottom[i]),&(d->bb_top[i]),fname);

		if ((d->importance[i]<0) || (d->importance[i]>10))
		{
			printf("design importance is fucked %i\n",d->importance[i]);
			return 0;
		}

		d->layout[i*NUM_VAR]=x/d->width;
		d->layout[i*NUM_VAR+1]=y/d->height;
		d->layout[i*NUM_VAR+2]=h/d->height;
		d->layout[i*NUM_VAR+3]=align;
		d->layout[i*NUM_VAR+4]=fix;
		d->layout[i*NUM_VAR+5]=-1;
		d->layout[i*NUM_VAR+6]=alt;
		d->layout[i*NUM_VAR+7]=i;
		
		d->alignment[i]=align;
		
		d->num_alt[i]=0;
		d->alt_num_lines[i*MAX_ALT]=d->num_lines[i];
		d->alt_aspect_ratio[i*MAX_ALT]=d->aspect_ratio[i];

		printf("Loaded element %d , id %d (%s), group id %i, imp %d, ar %f, fixed alignment %d. num lines %d.optional %d, alt %i,layout: %.2f,%.2f,%.2f,%.2f,%.2f  \n",i,d->id[i],fname,d->group_id[i], d->importance[i],d->aspect_ratio[i],d->fixed_alignment[i],d->num_lines[i],d->optional[i],alt,d->layout[i*NUM_VAR],d->layout[i*NUM_VAR+1],d->layout[i*NUM_VAR+2],d->layout[i*NUM_VAR+3],d->layout[i*NUM_VAR+4]);

		if (d->aspect_ratio[i]==0)
		{
			printf("aspect ratio is 0\n");
			return 0;
		}

	}
	
	
	/*	
	for (int i=d->num_elements*NUM_VAR;i<d->layout_size;i++)
		d->layout[i]=-1;


	int offset=d->num_elements*NUM_VAR;
	float height,width,xp,yp;
	for(int i=0;i< d->num_elements;i++)
	{				
		d->layout[i*NUM_VAR+5]=i;
		
		height=round(d->layout[NUM_VAR*i+2]*d->height);
		width=round(height/d->aspect_ratio[i]);
		xp=round(d->layout[NUM_VAR*i]*d->width);
		yp=round(d->layout[NUM_VAR*i+1]*d->height);
		
		d->layout[offset+i*NUM_RVAR]=d->type[i];
		d->layout[offset+i*NUM_RVAR+1]=round(xp+(width*d->bb_left[i])); 
		d->layout[offset+i*NUM_RVAR+2]=round(yp+(height*d->bb_bottom[i]));
		//d->layout[offset+i*NUM_RVAR+3]=round(yp+(height*d->bb_bottom[i]));
		d->layout[offset+i*NUM_RVAR+3]=width;	
		d->layout[offset+i*NUM_RVAR+4]=height;
		d->layout[offset+i*NUM_RVAR+5]=1.0;	
		d->layout[offset+i*NUM_RVAR+6]=0;
	}
	*/


	fscanf(fp,"%d overlap regions\n",&d->num_overlap_regions);

	d->overlap_region_elem=(int *)malloc(d->num_overlap_regions*sizeof(int));
	d->overlap_regions=(Box *)malloc(d->num_overlap_regions*sizeof(Box));

	int id;
	float x1,x2,y1,y2;
	for (int i=0 ; i <d->num_overlap_regions ; i++)
	{
		fscanf(fp,"%d,%f,%f,%f,%f\n",&id,&x1,&x2,&y1,&y2);

		if (id==0)
			d->overlap_region_elem[i]=-1;

		for (int j=0 ; j < d->num_elements ; j++)
		{
			if (id== d->id[j])
			{
				d->overlap_region_elem[i]=j;
			}
		}
		d->overlap_regions[i].l=x1;
		d->overlap_regions[i].r=x2;
		d->overlap_regions[i].b=y1;
		d->overlap_regions[i].t=y2;

		//printf ("%i: %f,%f,%f,%f\n",id,x1,x2,y1,y2);
	}

	//for (int i=0;i<d->num_overlap_regions;i++)
	//	printf("region %i, id %i\n", i,d->overlap_region_elem[i]);
	
	
	int num_alt_lines;
	fscanf(fp,"%d alternates\n",&num_alt_lines);
	
	for (int a=0; a <num_alt_lines; a++)
	{
		int elem_id=0;
		int num_alt=0;
		
		fscanf(fp,"%d,%d,",&elem_id,&num_alt);
		
		int idx=0;
		for(int i=0;i< d->num_elements;i++)
		{	
			if (elem_id==d->id[i])
				idx=i;
		}
		d->num_alt[idx]=num_alt;
		
		int num_lines,max_line_length;
		float aspect_ratio;
		for (int i=0; i <num_alt; i++)
		{
			fscanf(fp,"%d,%f,%d,",&num_lines,&aspect_ratio,&max_line_length);
			
			d->alt_num_lines[idx*MAX_ALT+i]=num_lines;
			d->alt_aspect_ratio[idx*MAX_ALT+i]=aspect_ratio;
			
			//printf("element %i has alternate with %i lines and ar %f\n",idx,num_lines,aspect_ratio);
			
			
			if (aspect_ratio==0)
			{
				printf("aspect ratio of alt %i is 0\n",i);
				return 0;
			}
		}
	
		fscanf(fp,"\n");
	
	}
	
	float curr_x=0,curr_y=0;
	for(int i=0;i< d->num_elements;i++)
	{		
		d->init_layout[i*NUM_VAR]=curr_x;
		

		if (d->type[i]==1)		
			d->init_layout[i*NUM_VAR+2]=((MIN_TEXT_SIZE*max(d->width,d->height)*d->num_lines[i])/10.0)/d->height + (5.0/d->height);
		else
			d->init_layout[i*NUM_VAR+2]=MIN_GRAPHIC_SIZE+(5.0/d->height);
		
		if (d->init_layout[i*NUM_VAR+2]+curr_y>1)
		{
			d->init_layout[i*NUM_VAR+1]=0;	
			
			curr_x+=0.34;
			if (curr_x>=1)
				curr_x=0;
				
			d->init_layout[i*NUM_VAR]=curr_x;
			curr_y=d->init_layout[i*NUM_VAR+2];	
		}
		else
		{
			d->init_layout[i*NUM_VAR+1]=curr_y;	
			curr_y+=d->init_layout[i*NUM_VAR+2];	
		}	
		
		for (int j=3;j<NUM_VAR;j++)
			d->init_layout[i*NUM_VAR+j]=d->layout[i*NUM_VAR+j];	
	}
	
	
	/*
	for(int i=0;i< d->num_elements;i++)
	{				
		d->init_layout[i*NUM_VAR+5]=i;
		
		height=round(d->init_layout[NUM_VAR*i+2]*d->height);
		width=round(height/d->aspect_ratio[i]);
		xp=round(d->init_layout[NUM_VAR*i]*d->width);
		yp=round(d->init_layout[NUM_VAR*i+1]*d->height);
		
		d->init_layout[offset+i*NUM_RVAR]=d->type[i];
		d->init_layout[offset+i*NUM_RVAR+1]=round(xp+(width*d->bb_left[i])); 
		d->init_layout[offset+i*NUM_RVAR+2]=round(yp+(height*d->bb_bottom[i]));
		d->init_layout[offset+i*NUM_RVAR+3]=width;	
		d->init_layout[offset+i*NUM_RVAR+4]=height;
		d->init_layout[offset+i*NUM_RVAR+5]=1.0;	
		d->init_layout[offset+i*NUM_RVAR+6]=0;
		d->init_layout[offset+i*NUM_RVAR+7]=i;
	}
	
	for(int i=d->num_elements;i < MAX_ELEMENTS;i++)
		d->init_layout[offset+i*NUM_RVAR]=-1;
	
		

	
	for (int j=0;j<d->num_elements;j++)
	{
		int check_r=d->init_layout[offset+j*NUM_RVAR];
		bool no_elements=true;
		if (check_r>-1)
		{
			for (int k=0;k<ne;k++)
			{
				if (d->init_layout[k*NUM_VAR+5]==j)
					no_elements=false;
			}
			
			if ((no_elements))
			{
				printf("ERROR in design init. region %i has no elements\n",j);
			}
		}

	}
	*/
		
	if (interactive_mode)	
	{
		memcpy(d->check_layout,d->layout, d->layout_size*sizeof(float));
		memcpy(d->init_layout,d->layout, d->layout_size*sizeof(float));
	}
	

	int max_num=1000;
	int skip=1;
	d->align_err=(float *)malloc(max_num*sizeof(float));
	for (int i=0;i< max_num;i++)
	{

		if (i <= skip)
			d->align_err[i]=0;
		else
		{
			float frac=(i-skip)/((float) max_num);
			d->align_err[i]=5*atan(frac/0.015);
		}
	}

	int max_atan_num=20000;
	d->atan_fixed=(float *)malloc(max_atan_num*sizeof(float));
	for (int i=0;i< max_atan_num;i++)
		d->atan_fixed[i]=atan(float(i)/200.0);



	cout << "Finished loading" << endl;


	return d;
}

/*
Design *loadDesignFromXML(char *filename)
{

	Design *d = (Design*)malloc( sizeof(Design) );

	//hacky. should find the size first
    int size=999999;

    FILE *fp = NULL;
	fp = fopen(filename, "r");

	if (!fp)
		return 0;

	char *str = (char *) malloc(size+1);
	int bytes_read = fread (str, sizeof (char), size, fp);
	fclose (fp);

	cout << "finished reading " << bytes_read << " bytes " << endl;


	d->name=extractXMLElement(str, "Name");
	d->width=float(atoi(extractXMLElement(str, "Width")));
	d->height=float(atoi(extractXMLElement(str, "Height")));

	cout << "Design Name " << d->name << endl;
	cout << "Design Width " << d->width << endl;
	cout << "Design Height " << d->height << endl;


	char *element_list=extractXMLElement(str, "ElementList");

	if (!element_list)
	{
		cout << "No Element List" << endl;
		return 0;
	}

	char *em_list=element_list;


	int em_cnt=0;
	while (true)
	{
		char *element_str=getSubstring(em_list, "<Element>","</Element>");
		if (!element_str)
			break;
		em_list=strstr(em_list,"</Element>")+9;
		em_cnt++;
		free(element_str);
	}

	d->num_elements=em_cnt;
	cout << "Design Number of Elements " << d->num_elements << endl;



	d->id=(int *)malloc(d->num_elements*sizeof(int));
	d->importance=(int *)malloc(d->num_elements*sizeof(int));
	d->type=(int *)malloc(d->num_elements*sizeof(int));
	d->bb_left=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_right=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_bottom=(float *)malloc(d->num_elements*sizeof(float));
	d->bb_top=(float *)malloc(d->num_elements*sizeof(float));

	d->tight_bb_left=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_right=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_bottom=(float *)malloc(d->num_elements*sizeof(float));
	d->tight_bb_top=(float *)malloc(d->num_elements*sizeof(float));

	d->num_lines=(int *)malloc(d->num_elements*sizeof(int));
	d->aspect_ratio=(float *)malloc(d->num_elements*sizeof(float));
	d->line_gap=(float *)malloc(d->num_elements*sizeof(float));
	d->group_id=(int *)malloc(d->num_elements*sizeof(int));
	d->alignment=(int *)malloc(d->num_elements*sizeof(int));
	d->num_alt=(int *)malloc(d->num_elements*sizeof(int));

	//d->alt_alignment=(int **)malloc(d->num_elements*sizeof(int *));



	d->layout_size=d->num_elements*NUM_VAR;
	d->layout=(float *)malloc(d->layout_size*sizeof(float));
	d->init_layout=(float *)malloc(d->layout_size*sizeof(float));
	d->check_layout=(float *)malloc(d->layout_size*sizeof(float));

	d->check_layout_exists=false;
	d->fixed_regions=false;
	d->region_proposals=true;

	em_list=element_list;
	for (int i=0 ; i < d->num_elements ; i++)
	{
		char *element_str=extractXMLElement(em_list, "Element");

		if (!element_str)
			break;


		em_list=strstr(em_list,"</Element>")+9;

		d->id[i]=i+1;
		d->importance[i]=atoi(extractXMLElement(element_str, "Importance"));
		d->type[i]=atoi(extractXMLElement(element_str, "Type"));


		d->group_id[i]=atoi(extractXMLElement(element_str, "GroupID"));
		d->aspect_ratio[i]=atof(extractXMLElement(element_str, "AspectRatio"));
		d->line_gap[i]=atof(extractXMLElement(element_str, "LineGap"));
		d->num_lines[i]=atoi(extractXMLElement(element_str, "NumberOfLines"));
		//d->width[i]=atoi(extractXMLElement(element_str, "Width"));
		//d->height[i]=atoi(extractXMLElement(element_str, "Height"));
		d->alignment[i]=atoi(extractXMLElement(element_str, "Alignment"));
		d->num_alt[i]=atoi(extractXMLElement(element_str, "NumberAlternate"));


		d->bb_left[i]=atof(extractXMLElement(element_str, "BoundingBoxLeft"))-0.05;
		d->bb_right[i]=atof(extractXMLElement(element_str, "BoundingBoxRight"))+0.05;
		d->bb_top[i]=atof(extractXMLElement(element_str, "BoundingBoxTop"))+0.05;
		d->bb_bottom[i]=atof(extractXMLElement(element_str, "BoundingBoxBottom"))-0.05;

		//d->bb_left[i]=max(atof(extractXMLElement(element_str, "BoundingBoxLeft")),0.0);
		//d->bb_right[i]=min(atof(extractXMLElement(element_str, "BoundingBoxRight")),1.0);
		//d->bb_top[i]=min(atof(extractXMLElement(element_str, "BoundingBoxTop")),1.0);
		//d->bb_bottom[i]=max(atof(extractXMLElement(element_str, "BoundingBoxBottom")),0.0);

		//printf("bb of element %i: %.2f %.2f %.2f %.2f\n",i,d->bb_left[i],d->bb_right[i],d->bb_bottom[i],d->bb_top[i]);

		d->tight_bb_left[i]=atof(extractXMLElement(element_str, "TightBoundingBoxLeft"));
		d->tight_bb_right[i]=atof(extractXMLElement(element_str, "TightBoundingBoxRight"));
		d->tight_bb_top[i]=atof(extractXMLElement(element_str, "TightBoundingBoxTop"));
		d->tight_bb_bottom[i]=atof(extractXMLElement(element_str, "TightBoundingBoxBottom"));

		d->layout[i*NUM_VAR]=atof(extractXMLElement(element_str, "X"))/d->width;
		d->layout[i*NUM_VAR+1]=atof(extractXMLElement(element_str, "Y"))/d->height;
		d->layout[i*NUM_VAR+2]=atof(extractXMLElement(element_str, "Height"))/d->height;
		d->layout[i*NUM_VAR+3]=atof(extractXMLElement(element_str, "Alternate"));
		d->layout[i*NUM_VAR+4]=0;

		//d->alt_alignment[i]=(int *)malloc(d->num_alt[i]*sizeof(int));

		cout << "Loaded element: " <<extractXMLElement(element_str, "FileName") << endl;

		free(element_str);
	}

	d->num_overlap_regions=0;

	int max_num=1000;
	int skip=1;
	d->align_err=(float *)malloc(max_num*sizeof(float));
	for (int i=0;i< max_num;i++)
	{

		if (i <= skip)
			d->align_err[i]=0;
		else
		{
			float frac=(i-skip)/((float) max_num);
			d->align_err[i]=5*atan(frac/0.015);
		}
	}

	int max_atan_num=20000;
	d->atan_fixed=(float *)malloc(max_atan_num*sizeof(float));
	for (int i=0;i< max_atan_num;i++)
		d->atan_fixed[i]=atan(float(i)/200.0);


	free(str);

	return d;
}
*/

int freeDesign(Design *d)
{

	cout << "Free design " << endl;
	free(d->id);
	free(d->type);
	free(d->importance);
	free(d->group_id);
	free(d->num_lines);
	free(d->name);
	free(d->alignment);
	free(d->fixed_alignment);
	free(d->line_gap);
	free(d->bb_left);
	free(d->bb_right);
	free(d->bb_top);
	free(d->bb_bottom);
	free(d->tight_bb_left);
	free(d->tight_bb_right);
	free(d->tight_bb_top);
	free(d->tight_bb_bottom);
	free(d->align_err);
	free(d->atan_fixed);
	free(d->layout);
	free(d->init_layout);
	free(d->check_layout);
	free(d->check_layout_distances);
	free(d->overlap_region_elem);
	free(d->overlap_regions);
	free(d->num_alt);
	free(d->alt_num_lines);
	free(d->alt_aspect_ratio);
	free(d->optional);
	free(d->constraints);
	
	free(d);
	return 0;
}



int freeDeviceDesign(Design *d)
{

	Design *h=(Design *)malloc(sizeof(Design));
	ASSERT(hipSuccess == hipMemcpy(h, d, sizeof(Design), hipMemcpyDeviceToHost),"cuda copy to device fail",-1);
	ASSERT(hipSuccess == hipFree(h->id),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->type),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->importance),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->group_id),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->num_lines),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->name),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->alignment),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->line_gap),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->bb_left),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->bb_right),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->bb_top),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->bb_bottom),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->tight_bb_left),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->tight_bb_right),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->tight_bb_top),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->tight_bb_bottom),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->align_err),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->atan_fixed),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->layout),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->init_layout),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->check_layout),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->check_layout_distances),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->overlap_region_elem),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->overlap_regions),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->num_alt),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->alt_num_lines),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->alt_aspect_ratio),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->optional),       "Device deallocation failed", -1);
	ASSERT(hipSuccess == hipFree(h->constraints),       "Device deallocation failed", -1);
	//for (int i=0;i<d->num_elements;i++)
	//	ASSERT(hipSuccess == hipFree(d->alt_alignment[i]),       "Device deallocation failed", -1);

	//ASSERT(hipSuccess == hipFree(d->alt_alignment),       "Device deallocation failed", -1);



	free(h);
	return 0;
}

int copyDesignToDevice(Design *device, Design *host)
{

	//cout << "Starting design copy" << endl;

	Design *host_copy=(Design *)malloc(sizeof(Design));
	memcpy(host_copy, host,sizeof(Design));



	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->name), strlen(host->name)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->name), host->name, strlen(host->name), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->id),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->id), host->id, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->group_id),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->group_id), host->group_id, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->type),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->type), host->type, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->importance),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->importance), host->importance, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->num_lines),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->num_lines), host->num_lines, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->alignment),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->alignment), host->alignment, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->optional),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->optional), host->optional, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->num_alt),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->num_alt), host->num_alt, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->fixed_alignment),host->num_elements*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->fixed_alignment), host->fixed_alignment, host->num_elements*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->alt_num_lines),host->num_elements*MAX_ALT*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->alt_num_lines), host->alt_num_lines, host->num_elements*MAX_ALT*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->alt_aspect_ratio),host->num_elements*MAX_ALT*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->alt_aspect_ratio), host->alt_aspect_ratio, host->num_elements*MAX_ALT*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->aspect_ratio),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->aspect_ratio), host->aspect_ratio, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->line_gap),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->line_gap), host->line_gap, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->bb_left),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->bb_left), host->bb_left, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->bb_right),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->bb_right), host->bb_right, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->bb_top),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->bb_top), host->bb_top, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->bb_bottom),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->bb_bottom), host->bb_bottom, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->tight_bb_left),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->tight_bb_left), host->tight_bb_left, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->tight_bb_right),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->tight_bb_right), host->tight_bb_right, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->tight_bb_top),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->tight_bb_top), host->tight_bb_top, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->tight_bb_bottom),host->num_elements*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->tight_bb_bottom), host->tight_bb_bottom, host->num_elements*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->layout),host->layout_size*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->layout), host->layout, host->layout_size*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->init_layout),host->layout_size*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->init_layout), host->init_layout, host->layout_size*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->check_layout),host->layout_size*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->check_layout), host->check_layout, host->layout_size*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->check_layout_distances),MAX_ELEMENTS*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->check_layout_distances), host->check_layout_distances, MAX_ELEMENTS*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);


	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->align_err),1000*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->align_err), host->align_err, 1000*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->atan_fixed),20000*sizeof(float)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->atan_fixed), host->atan_fixed, 20000*sizeof(float), hipMemcpyHostToDevice),"cuda copy to device fail",-1);


	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->overlap_region_elem),host->num_overlap_regions*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->overlap_region_elem), host->overlap_region_elem, host->num_overlap_regions*sizeof(int), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc( (void **)(&host_copy->overlap_regions),host->num_overlap_regions*sizeof(Box)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->overlap_regions), host->overlap_regions, host->num_overlap_regions*sizeof(Box), hipMemcpyHostToDevice),"cuda copy to device fail",-1);

	ASSERT(hipSuccess == hipMalloc((void **) (&host_copy->constraints),MAX_ELEMENTS*3*NUM_AVAR*sizeof(int)),"cuda malloc fail",-1);
	ASSERT(hipSuccess == hipMemcpy((host_copy->constraints), host->constraints, MAX_ELEMENTS*3*NUM_AVAR*sizeof(int), hipMemcpyHostToDevice),"cuda copy constraints to device fail",-1);


	ASSERT(hipSuccess == hipMemcpy(device, host_copy, sizeof(Design), hipMemcpyHostToDevice),"cuda copy host_copy to device fail",-1);

	free(host_copy);
	cout << "finished design copy" << endl;

	return 0;
}
