#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project
* example application, doesn't use cutil library.
*/

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <time.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <signal.h>

#include <sys/socket.h>
#include <netinet/in.h>
#include <netdb.h>


#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "design.cuh"
#include "model.cuh"
#include "saoptimizer.cuh"
#include "settings.cuh"

#include "svd.c"


using namespace std;

#ifdef _WIN32
#define STRCASECMP  _stricmp
#define STRNCASECMP _strnicmp
#else
#define STRCASECMP  strcasecmp
#define STRNCASECMP strncasecmp
#endif

//char *home_dir="./";

char default_param_file[1024];
char param_file[1024];
char param_type[1024];
char param_change_file[1024];
char design_file[1024];
char check_layout_file[1024];
int run_id=1;
bool interactive_run=true;

struct addrinfo *host_info;
char hostname[100];

char *host_name = "127.0.0.1";
	
char layout_file[1024],layout_tmp_file[1024],layout_features_file[1024],layout_features_file2[1024],check_layout_features_file[1024],run_host_file[1024],host_run_file[1024],host_pid_file[1024];;
	
Design *h_d, *d;

float *h_evals;
float *h_opt_layouts;
float *h_params_grads;
float *h_temperatures;
int *h_temp_ids;
float *init_params;
float *h_params;
float *layouts,*evals,*params,*params_grads, *opt_layouts,*proposals, *temperatures;
float *opt_steps_all;
float *opt_steps;
int *temp_ids;


#define IGNORE_VAL -9999

hipError_t err;

#define CHECKCALL(x)\
		err=x;\
		if (err!=hipSuccess){\
			printf("ERROR: %s\n", hipGetErrorString(err));\
			exit(-1);\
		}


void sig_handler (int sig);

//extern "C" void float *loadParametersFromFile(char *filename);
int optimizeLayout(Design *d,int num_kernels,int num_params,int num_outer, int num_inner, float *params,float *temperatures,int *temp_ids,float *layouts,float *params_grads,float *opt_layouts, float *evals,float *proposals,Design *h_d,float *h_temperatures,float *h_evals,int *h_temp_ids,float *h_params_grads,float *h_opt_layouts,bool silent,int debug_mode,float *h_opt_steps);
int nio(Design *d,Design *h_d,int num_kernels,int num_params,int num_outer,int num_inner,float *params,float *temperatures,int *temp_ids,float *layouts,float *params_grads,float *opt_layouts, float *evals,float *proposals,float *h_temperatures,float *h_evals,int *h_temp_ids,float *h_params_grads,float *h_opt_layouts, int debug_mode,int nio_layout_num,char *design_name);
float evaluateLayoutHost(Design *d,Design *h_d,int num_params, float *params, float *params_grad,  float *h_layout,float *h_params,  float *h_params_grad, int num_prev_layout,float *previous_layout, bool debug);
void finiteDiffGrad(Design *d,Design *h_d,int num_params, float *params, float *params_grad,  float *h_layout,float *h_params,  float *h_params_grad,float *h_params_grad_fd);
bool updateCheckLayout(Design *d,Design *h_d, int num_params, int num_kernels,float *params, float *params_grad,float *layouts, float *h_params,char *check_layout_features_file);
bool updateParameterFile(float **h_params, float *params, int num_params);
void sendLayoutToServer(Design *d, float *layout, float energy);
float constrainedGradientDescent(Design *h_d,Design *d,int num_params, float *params,float *atan_params, float *params_grads, float *h_params,  float *h_init_layout,int num_previous_layout, float *previous_layout);
int getConstraints(Design *d, float *layout, float ***C_out,float **b_out,float **loc_out,float ***H_out,  bool debug);
void getSVD(int num_constraints, int num_var, float **C, float ***Q_out, float ***W_out, float **a_out);
int  getConstraintDirections(Design *h_d, float *h_init_layout, float **D_out, bool debug);
void writeOptimizationSteps(int run_id, int num_outer,int num_inner,int curr_run, int num_run,float *opt_steps_all);
void freeMemory();
void cleanup();
void writeHostFiles();
void writeOutFeatures(char *features_file,float opt_eval,float *h_params,float *h_params_grad);
bool loadParameterSample(float **h_params, float *params, int num_params);

int main(int argc, char **argv)
{

	const int kb = 1024;
    const int mb = kb * kb;
	int devCount;
    hipGetDeviceCount(&devCount);
    cout << "CUDA Devices: "  << endl;
	 hipError_t cuda_status;

    for(int i = 0; i < devCount; ++i)
    {
      	hipSetDevice (i);
      	hipDeviceReset ();
      	
      	if ( hipSuccess != cuda_status ){
      		printf("Error: hipDeviceReset fails, %s \n", hipGetErrorString(cuda_status) );
      	}
    	
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        

        
        cout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        cout << "  Block registers: " << props.regsPerBlock << endl << endl;

        cout << "  Warp size:         " << props.warpSize << endl;
        cout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        cout << "  Multiprocessors:         " << props.multiProcessorCount << endl;
        cout << "  Concurrent kernels:         " << props.concurrentKernels << endl;
        cout << endl;
        
       
     	size_t free_byte ;
        size_t total_byte ;
        cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ){
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
        
        
    }
	    
	    
	 signal(SIGTERM, sig_handler);
	 //signal(SIGINT, sig_handler);
	// hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	//CHECKCALL(hipSetDevice(0));
	//cout << " post list" << endl;

    //optimization parameters
	int num_outer=1000000;
	int num_inner=25;
    int num_kernels=128;
    int num_run=200;
    char *design_name="artshow";
    bool nio_mode=false;
    int nio_layout_num=-1;
    bool interactive_mode=false;
    sprintf(param_type,"near");
    
    int debug_mode=0;


    int num_params=3*NUM_FEATURES;


    
    for (int i=1;i < argc;i++)
    {
    	if (strcmp(argv[i],"-o")==0)
    		num_outer=atoi(argv[i+1]);
    	else if (strcmp(argv[i],"-e")==0)
    		num_inner=atoi(argv[i+1]);
    	else if (strcmp(argv[i],"-i")==0)
    		interactive_mode=true;
    	else if (strcmp(argv[i],"-b")==0)
    		debug_mode=atoi(argv[i+1]);    		
    	else if (strcmp(argv[i],"-k")==0)
    		num_kernels=atoi(argv[i+1]);
    	else if (strcmp(argv[i],"-r")==0)
    		run_id=atoi(argv[i+1]);
    	else if (strcmp(argv[i],"-d")==0)
    		design_name=argv[i+1];
    	else if (strcmp(argv[i],"-t")==0)
    		sprintf(param_type,argv[i+1]);
    	else if (strcmp(argv[i],"-n")==0)
    		nio_mode=true;
    	else if (strcmp(argv[i],"-l")==0)
    	{
    		nio_layout_num=atoi(argv[i+1]);
    	}
    	else
    	{
    		printf("unrecognized parameter %s\n",argv[i]);
    		//i--;
    	}
    }
    
    	

    if (interactive_mode)
    {
    	num_outer=1000000;
    	num_inner=25;
    	nio_mode=false;
    	num_run=1;
    }

	if ((nio_mode) || (strcmp(param_type,"nio")==0))
	{
    	num_outer=1000;
    	num_inner=25;
    	interactive_mode=false;
    	nio_mode=true;
	
		sprintf(param_type,"nio_init");
		
	}

    printf("Starting optimization with run ID %d\nDesign %s\nKernels:%d, outer iterations %d, inner iterations %d, debug mode %i\nNIO %i, layout num %d\n\n", run_id, design_name, num_kernels, num_outer, num_inner,debug_mode, nio_mode, nio_layout_num);

    char init_design_file[1024];
    char init_fname[100];
    sprintf(init_design_file,"%s/data/default/%s_150_100_default.data",home_dir,design_name);
    sprintf(init_fname,"%s/data/default/%s_150_100_default.data",home_dir,design_name);

	sprintf(check_layout_file,"%sdata/runs/r%d_check_layout.data",home_dir,run_id);
	sprintf(design_file,"%sdata/runs/r%d_design.data",home_dir,run_id);

    sprintf(param_file,"%sdata/%s_parameters.data",home_dir,param_type);
    sprintf(default_param_file,"%sdata/default_parameters.data",home_dir);
    sprintf(param_change_file,"%sdata/runs/r%d_parameter_change.data",home_dir,run_id);


	struct addrinfo hints;
	memset(&hints, 0, sizeof hints);
	hints.ai_family = AF_UNSPEC;
	hints.ai_socktype = SOCK_STREAM;
	

	getaddrinfo(host_name, "8080", &hints, &host_info);

	gethostname(hostname,100);
	
	
	sprintf(layout_file, "%sdata/runs/r%d_opt_layout.data",home_dir,run_id);
	sprintf(layout_tmp_file, "%sdata/runs/r%d_opt_layout.data.tmp",home_dir,run_id);
	sprintf(layout_features_file, "%sdata/runs/r%d_opt_layout_features.txt",home_dir,run_id);
	sprintf(layout_features_file2, "%sdata/runs/r%d_opt_layout_features2.txt",home_dir,run_id);
	sprintf(check_layout_features_file, "%sdata/runs/r%d_user_layout_features.txt",home_dir,run_id);
	sprintf(run_host_file, "%sdata/runs/r%d_host.data",home_dir,run_id);
	sprintf(host_run_file, "%sdata/runs/%s_runid.data",home_dir,hostname);
	sprintf(host_pid_file, "%sdata/runs/pid_%s_%d.data",home_dir,hostname,run_id);
	
	writeHostFiles();




    if (nio_layout_num>=0)
    	sprintf(init_design_file,"%s/data/nio/%s_%d.data",home_dir,design_name,nio_layout_num);
    else
    {
    	strcpy(init_design_file,design_file);
    	strcpy(init_fname,design_file);
    }


    h_d= loadDesignFromFile(init_design_file,interactive_mode);
    if (!h_d)
    {
    	cout << "Problem loading XML file " << init_design_file << endl;
    	return 0;
    }
    
    /*
    Design *h_init_d = loadDesignFromFile(init_fname);
	if (!h_init_d)
	{
		cout << "Problem loading XML file " << init_fname << endl;
		return 0;
	}

	memcpy(h_d->init_layout, h_init_d->layout, h_d->layout_size*sizeof(float));
	freeDesign(h_init_d);
	*/


	



	//cout << " loading parameters from " << param_file << endl;
    init_params= loadParametersFromFile(default_param_file,param_file,num_params);
    h_params=(float *)malloc(num_params*sizeof(float));
	//cout << " num_params "  << num_params <<endl;

    //allocate host memory
    h_evals=(float *)malloc(num_kernels*sizeof(float));
    h_opt_layouts= (float *)malloc(num_kernels*h_d->layout_size*sizeof(float));
    h_params_grads= (float *)malloc(num_kernels*num_params*sizeof(float));
    h_temperatures=(float *)malloc(num_kernels*sizeof(float));
    h_temp_ids=(int *)malloc(num_kernels*sizeof(int));

	//cout << " allocate device memory "<< h_d->layout_size <<endl;
 

    CHECKCALL(hipMalloc(&layouts,num_kernels*h_d->layout_size*sizeof(float)));
    CHECKCALL(hipMalloc(&opt_layouts,num_kernels*h_d->layout_size*sizeof(float)));
    CHECKCALL(hipMalloc(&proposals,num_kernels*h_d->layout_size*sizeof(float)));
    CHECKCALL(hipMalloc(&params,num_params*sizeof(float)));
    CHECKCALL(hipMalloc(&params_grads,num_kernels*num_params*sizeof(float)));
    CHECKCALL(hipMalloc(&evals,num_kernels*sizeof(float)));
    CHECKCALL(hipMalloc(&temperatures,num_kernels*sizeof(float)));
    CHECKCALL(hipMalloc(&temp_ids,num_kernels*sizeof(int)));
    CHECKCALL(hipMalloc(&d, sizeof(Design)));
    
    //cout << " copyDesignToDevice "  << endl;
    copyDesignToDevice(d,h_d);

    CHECKCALL(hipMemcpy(params, init_params,num_params*sizeof(float), hipMemcpyHostToDevice));

    for (int i=0;i< num_kernels;i++)
    {
    	CHECKCALL(hipMemcpy(&layouts[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
    	CHECKCALL(hipMemcpy(&opt_layouts[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
    	CHECKCALL(hipMemcpy(&proposals[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
    }


    if (nio_mode)
    {
    	nio(d,h_d, num_kernels, num_params,num_outer,num_inner,params,temperatures,temp_ids,layouts,params_grads,opt_layouts, evals,proposals,h_temperatures,h_evals,h_temp_ids,h_params_grads,h_opt_layouts, debug_mode,nio_layout_num,design_name);
    	CHECKCALL(hipMemcpy(h_params, params,num_params*sizeof(float), hipMemcpyDeviceToHost));
    	return 0;
    	
    }


    int best_id=0;
    float eval_run[1000];

    clock_t start=clock();
    float eval_mean=0;
    
	opt_steps_all=(float *)malloc(num_run*num_outer*sizeof(float));
	opt_steps=(float *)malloc(num_outer*sizeof(float));
	
	
    for (int r=0;r< num_run;r++)
    {
    	

        for (int i=0;i< num_kernels;i++)
        {
        	CHECKCALL(hipMemcpy(&layouts[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
        	CHECKCALL(hipMemcpy(&opt_layouts[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
        	CHECKCALL(hipMemcpy(&proposals[i*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
        }

    	best_id=optimizeLayout(d, num_kernels, num_params,num_outer,num_inner,params,temperatures,temp_ids,layouts,params_grads,opt_layouts, evals,proposals,h_d,h_temperatures,h_evals,h_temp_ids,h_params_grads,h_opt_layouts,true,debug_mode,opt_steps); //(num_run>1)and(r>0)
    	eval_run[r]=h_evals[best_id];
    	eval_mean+=eval_run[r];
    	
    	memcpy(&(opt_steps_all[r*num_outer]),opt_steps,num_outer*sizeof(float));
    	
    	printf("run %i of %i. eval %.3f. mean eval %.3f, mean inner time %f\n",r,num_run,eval_run[r],eval_mean/(r+1),1000*(((double)(clock() - start)/(r+1)) / CLOCKS_PER_SEC)/num_outer);
    	
    	writeOptimizationSteps(run_id,num_outer,num_inner,r+1,num_run,opt_steps_all);
    }
    CHECKCALL(hipMemcpy(h_params, params,num_params*sizeof(float), hipMemcpyDeviceToHost));



    clock_t stop=clock();
    float t1=((double)(stop - start) / CLOCKS_PER_SEC);

	eval_mean=eval_mean/float(num_run);

	float eval_var=0;
	for (int r=0;r< num_run;r++)
		eval_var+=(eval_run[r]-eval_mean)*(eval_run[r]-eval_mean);
	eval_var=eval_var/float(num_run);

	printf("Mean time for %i runs: %.3f. Mean eval %f, var %f \n",num_run,t1/num_run,eval_mean,eval_var);


    //float opt_eval=h_evals[best_id];
    //float *output_layout;
    //output_layout=&h_opt_layouts[best_id*h_d->layout_size];


	//float *opt_params_grad=&h_params_grads[best_id*num_params];

	/*
	float h_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grads, output_layout,h_params, h_params_grads,true);

	//printf("\nLayout Energy: \n");
	for (int i=0;i<NUM_FEATURES;i++)
		printf("%i\t%5.1f \t %3.2f\t %3.2f \t  nl: %3.2f\t %3.2f \t %s \n", i,h_params[i],h_params_grads[i],h_params[i]*h_params_grads[i],h_params[i+NUM_FEATURES],h_params_grads[i+NUM_FEATURES],feat_names[i]);
	cout << "Design " << h_d->name << " eval: " << opt_eval<<" h_eval: " << h_eval<<endl;

	h_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_d->layout,h_params, h_params_grads,true);
	printf("\nOriginal Layout Layout Energy: \n");
	for (int i=0;i<NUM_FEATURES;i++)
		printf("%i\t%5.1f \t %.2f\t %4.2f\t  nl: %4.2f\t %4.2f \t %s \n", i,h_params[i],h_params_grads[i],h_params[i]*h_params_grads[i],h_params[i+NUM_FEATURES],h_params_grads[i+NUM_FEATURES],feat_names[i]);
	cout << "Original Layout h_eval: " << h_eval<<endl;
	

	//render output layout
	char layout_file[ 1024 ];
	sprintf(layout_file,"%s/data/opt/%s_%d.data",home_dir,design_name,nio_layout_num);
	writeLayoutToFile(h_d, output_layout, layout_file);

	*/

	//char commandline[ 1024 ];
	//snprintf( commandline, sizeof (commandline), "python /Users/donovan/Documents/work/aptana/GraphicDesign/src/renderDesign.py %s", layout_file);
	//printf(commandline);
	//system( commandline );




    //cleanup


    printf("Finished Optimization");

    return 0;

}


void freeMemory()
{
	
	CHECKCALL(hipGetLastError());
	CHECKCALL(hipDeviceSynchronize());	
	
	printf("Freeing device memory\n");
	
	freeDeviceDesign(d);
	CHECKCALL(hipFree(d));
	CHECKCALL(hipFree(evals));
	CHECKCALL(hipFree(layouts));
	CHECKCALL(hipFree(params));
	CHECKCALL(hipFree(params_grads));
	CHECKCALL(hipFree(opt_layouts));
	CHECKCALL(hipFree(temperatures));
	CHECKCALL(hipFree(temp_ids));

	printf("Freeing host memory\n");

	freeDesign(h_d);
    free(h_evals);
    free(h_params_grads);
    free(h_opt_layouts);
    free(h_temperatures);
    free(h_temp_ids);
    free(h_params);
    free(init_params);
    free(opt_steps_all);
    free(opt_steps);
}




void writeOptimizationSteps(int run_id, int num_outer,int num_inner,int curr_run, int num_run,float *opt_steps_all)
{
	char output_file[1024];
    sprintf(output_file,"%s/data/steps-r%i_%i_%i_%i.data",home_dir,run_id,num_outer,num_inner,num_run);
    
	FILE *fp=fopen(output_file,"w");

	if (fp>0)
	{		
	    
	    for (int i=0;i<num_outer;i++)
	    {
	    	float opt_sum=0;
	    	for (int j=0;j<curr_run;j++)
	    		opt_sum+=opt_steps_all[j*num_outer+i];
			
			fprintf(fp,"%.3f\n",opt_sum/curr_run);	    
	    }
		fclose(fp);
	}
	else
		printf("Error opening file %s\n",output_file);
	
}



void setParams(int num_params,float *h_params,float *h_params_temp,float *h_nio_gradient,float lambda)
{
	for (int j=0;j<num_params;j++)
	{
		float nio_grad=lambda*(h_nio_gradient[j]);
		if ( (j>=NUM_FEATURES*2))//(j==NUM_FEATURES+TEXT_SIZE_FEAT) or
		{
			h_params_temp[j]=h_params[j]-min(max(nio_grad,-1.0),1.0);
		}
		else
		{
			float log_param=log(h_params[j]);
			h_params_temp[j]=exp(log_param-min(max(nio_grad,-1.0),1.0));
		}

	}
}



void writeOutFeatures(char *features_file,float opt_eval,float *h_params,float *h_params_grad)
{

	FILE *fp=fopen(features_file,"w");

	if (fp>0)
	{
		
		float test_eval=-500;
		for (int k=0;k<NUM_FEATURES;k++)
			test_eval+=h_params[k]*h_params_grad[k];
		
		fprintf(fp,"Layout Energy: %f %f\n",opt_eval,test_eval*0.25);
		for (int k=0;k<NUM_FEATURES;k++)
			fprintf(fp,"%i\t%5.1f \t %4.2f\t %4.1f \tnl:%3.1f %s \n", k,h_params[k],h_params_grad[k],h_params[k]*h_params_grad[k],h_params[k+NUM_FEATURES],feat_names[k]);

		fprintf(fp,"Eval: %f %f\n",opt_eval,test_eval*0.25);
		
		if (abs(test_eval*0.25-opt_eval)>0.1)
			printf("Error. Opt eval %.3f doesnt match the double check %.3f\n",opt_eval,test_eval);

		fclose(fp);	

	}
}

int nio(Design *d,Design *h_d,int num_kernels,int num_params,int num_outer,int num_inner,float *params,float *temperatures,int *temp_ids,float *layouts,float *params_grads,float *opt_layouts, float *evals,float *proposals,float *h_temperatures,float *h_evals,int *h_temp_ids,float *h_params_grads,float *h_opt_layouts, int debug_mode,int nio_layout_num,char *design_name)
{
	printf("Starting NIO\n");

	float f_init;
	//float *f_init=(float *)malloc(sizeof(float));

	float *h_param_grad_init= (float *)malloc(num_params*sizeof(float));
	float *h_param_grad_temp= (float *)malloc(num_params*sizeof(float));
	float *h_params_grad_opt= (float *)malloc(num_params*sizeof(float));
	float *h_param_grad_fd= (float *)malloc(num_params*sizeof(float));
	float *h_params= (float *)malloc(num_params*sizeof(float));
	float *h_params_temp= (float *)malloc(num_params*sizeof(float));
	CHECKCALL(hipMemcpy(h_params, params,  num_params*sizeof(float), hipMemcpyDeviceToHost));

	float *h_nio_gradient= (float *)malloc(num_params*sizeof(float));


    for (int j=0;j< num_kernels;j++)
    	CHECKCALL(hipMemcpy(&layouts[j*h_d->layout_size], h_d->init_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    int best_id=0;

    int nio_iter=200;

    float *h_layout_list= (float *)malloc((nio_iter+1)*h_d->layout_size*sizeof(float));
    memcpy(&h_layout_list[0],h_d->layout,h_d->layout_size*sizeof(float));


	float *opt_steps=(float *)malloc(num_outer*sizeof(float));



	 char nio_param_file[1024];
  	 sprintf(nio_param_file,"%s/data/nio/%s_%d_params.data",home_dir,design_name,nio_layout_num);
	 char nio_param_file2[1024];
  	 sprintf(nio_param_file2,"%s/data/nio_r%d.data",home_dir,run_id);

	int run_outer=200;

	//main NIO loop
	for (int i=0;i <nio_iter;i++)
	{

	    clock_t start=clock();
	    clock_t stop;
	    
	   //printf("starting finite diff\n");
	   // finiteDiffGrad(d,h_d, num_params,params, params_grads, h_d->layout,h_params, h_param_grad_init,h_param_grad_fd);
		//printf("ending finite diff\n");

	 	f_init=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_d->layout,h_params, h_param_grad_init,0,0, false);
	 	
		

	    //float *set_layout= &h_opt_layouts[best_id*h_d->layout_size];
	    float *set_layout=h_d->init_layout;

	    /*
	    if (i%5!=0)
	    {
	    	//run_outer=num_outer/4;

			float f_check_min=99999;
			for (int j=0;j<=i;j++)
			{

				float f_check=evaluateLayoutHost(d,h_d, num_params,params, params_grads, &h_layout_list[j*h_d->layout_size],h_params, h_param_grad_temp,false);

				if (f_check < f_check_min)
				{
					f_check_min=f_check;
					set_layout=&h_layout_list[j*h_d->layout_size];
				}
				//printf("fcheck %i %.3f\n",j,f_check);
			}
			// printf("f_check_min: %.3f f_init %.3f", f_check_min,f_init);
	    }
	    */




	    for (int k=0;k< num_kernels;k++)
			CHECKCALL(hipMemcpy(&layouts[k*h_d->layout_size], set_layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));


		best_id= optimizeLayout(d, num_kernels, num_params,run_outer,num_inner,params,temperatures,temp_ids,layouts,params_grads,opt_layouts, evals,proposals,h_d,h_temperatures,h_evals,h_temp_ids,h_params_grads,h_opt_layouts,true,debug_mode,opt_steps);

		stop=clock();
		float t1=((double)(stop - start) / CLOCKS_PER_SEC);

		float f_opt=h_evals[best_id];
		float *h_layout_opt=&h_opt_layouts[best_id*h_d->layout_size];
		

		memcpy(&h_layout_list[(i+1)*h_d->layout_size],h_layout_opt,h_d->layout_size*sizeof(float));
		
		float f_check=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_layout_opt,h_params, h_params_grad_opt,0,0, false);
		if (f_check!=f_opt)
			printf("ERROR. fcheck %.3f doesnt match fopt %.3f\n",f_check,f_opt);
		


		float lambda=0.25;
		
		printf("writing out init\n");
		writeOutFeatures(check_layout_features_file,f_init,h_params,h_param_grad_init);
		printf("writing out opt\n");
		writeOutFeatures(layout_features_file,f_opt,h_params,h_params_grad_opt);

		sendLayoutToServer(h_d, h_layout_opt,f_opt);
		
		
		if (f_opt<f_init)
		//if (true)
		{
		

		

			float weight_sum=0;
			for (int j=0;j<num_params;j++)
				weight_sum+=h_params[j];


			for (int j=0;j<num_params;j++)
			{
				h_nio_gradient[j]=h_param_grad_init[j]-h_params_grad_opt[j];


				if (j>=NUM_FEATURES)
				{
					h_nio_gradient[j]=0.1*(h_nio_gradient[j]);
					
					//if the feature weight is 0, the  has to be 0.
					if (h_params[j%NUM_FEATURES]==0)
						h_nio_gradient[j]=0;
				}
				
			}

			bool searching=true;
			while (searching)
			{

				setParams(num_params,h_params,h_params_temp,h_nio_gradient,lambda);



				float eval1Orig=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_d->layout,h_params_temp, h_param_grad_temp,0,0,false);
				float eval1Opt=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_layout_opt,h_params_temp, h_param_grad_temp,0,0,false);


				setParams(num_params,h_params,h_params_temp,h_nio_gradient,0.5*lambda);


				float eval2Orig=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_d->layout,h_params_temp, h_param_grad_temp,0,0,false);
				float eval2Opt=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_layout_opt,h_params_temp, h_param_grad_temp,0,0,false);

                float diff1=eval1Orig-eval1Opt;// + (paramReg*paramDiff1).sum()
                float diff2=eval2Orig-eval2Opt;// + (paramReg*paramDiff2).sum()

                //printf("lsearch %1.3f 1: %3.1f - %3.1f = %3.3f, 2: %3.1f - %3.1f = %3.3f\n",lambda, eval1Orig, eval1Opt, eval1Orig-eval1Opt, eval2Orig, eval2Opt, eval2Orig-eval2Opt);


                if ((diff1>diff2) and (diff2>0) and (lambda>0.01))
					lambda=lambda*0.5;
				else
					searching=false;
			}

			/*
			printf("NIO Gradient Check Breakdown: \n");
			for (int j=0;j<NUM_FEATURES;j++)
			{
				if (!strstr(feat_names[j], "Element"))
					printf("%2.0i: %5.1f \t (%5.2f / %5.2f) \t nl1: %5.2f\t (%5.2f / %5.2f) \t %s \n", j,h_params[j],h_param_grad_init[j],h_param_grad_fd[j],h_params[j+1*NUM_FEATURES],h_param_grad_init[j+1*NUM_FEATURES],h_param_grad_fd[j+1*NUM_FEATURES],feat_names[j]);
			}
			*/

			//h_params[j+NUM_FEATURES],h_param_grad_init[j+NUM_FEATURES],h_params_grad_opt[j+NUM_FEATURES],h_nio_gradient[j+NUM_FEATURES],
			printf("NIO Layout Energy: \n");
			for (int j=0;j<NUM_FEATURES;j++)
			{
				if (!strstr(feat_names[j], "Element"))
					printf("%2.0i: %5.1f \t (%3.2f / %3.2f) %3.2f \t nl1: %3.2f\t (%3.2f / %3.2f)  %3.2f \t %s \n", j,h_params[j],h_param_grad_init[j],h_params_grad_opt[j],h_nio_gradient[j],h_params[j+1*NUM_FEATURES],h_param_grad_init[j+1*NUM_FEATURES],h_params_grad_opt[j+1*NUM_FEATURES],h_nio_gradient[j+1*NUM_FEATURES],feat_names[j]);

			}

			setParams(num_params,h_params,h_params,h_nio_gradient,lambda);
			//for (int j=0;j<num_params;j++)
			//{
			//	float log_param=log(h_params[j]);
			//	float nio_grad=lambda*(h_nio_gradient[j]);
			//	h_params[j]=exp(log_param-min(max(nio_grad,-1.0),1.0));
			//}

			saveParametersToFile(nio_param_file,h_params,num_params);
			saveParametersToFile(nio_param_file2,h_params,num_params);

			CHECKCALL(hipMemcpy(params, h_params,num_params*sizeof(float), hipMemcpyHostToDevice));

		}
		else
			run_outer=min(run_outer+20,1000);

		printf("NIO Iteration %i g %.4f, f_init %.2f, f_opt %.2f, lambda %.2f, time %.2f\n",i,f_init-f_opt,f_init,f_opt,lambda,t1);

	}

	free(h_layout_list);
	free(h_nio_gradient);
	free(h_param_grad_init);
	free(h_param_grad_temp);
	free(h_param_grad_fd);
	free(h_params);


	return 0;

}





void sendLayoutToServer(Design *d, float *layout, float energy)
{
	
	char layout_str[1000];
	memset(layout_str,0,1000);
	printLayout(layout_str,d,layout,energy);
	
	

	int sockfd = socket(host_info->ai_family, host_info->ai_socktype, host_info->ai_protocol);
	
	if (sockfd<0)
	{
		printf("can't create socket\n");
		return;
	}
	
	if (connect(sockfd, host_info->ai_addr, host_info->ai_addrlen)<0)
	{
		printf("can't create connection\n");
		return;
	}
	
	
	char send_string[1000];
	sprintf(send_string, "%d\n%s", run_id,layout_str);
	
	char sendline[1500];
	sprintf(sendline, 
	     "GET /design/computeSendLayout HTTP/1.0\r\n" 
	     "Host: %s\r\n"    
	     "Content-type: application/x-www-form-urlencoded\r\n"
	     "Content-length: %d\r\n\r\n"
	     "%s\r\n", host_name,(unsigned int)strlen(send_string), send_string);
	     
	
	int max_line=5000;
	char recvline[max_line];
	memset(recvline,0,max_line);
	size_t n;
	
	if (write(sockfd, sendline, strlen(sendline))>= 0) 
	{
	    while ((n = read(sockfd, recvline, max_line)) > 0) 
	    {
	        recvline[n] = '\0';
	    }          
	}
	
	//todo: check the re
	
	close(sockfd);
	
}

bool updateCheckLayout(Design *d,Design *h_d, int num_params, int num_kernels,float *params, float *params_grad,float *layouts, float *h_params, char *check_layout_features_file)
{


	bool using_files=false;
	bool is_new_layout=false;
	static clock_t check_layout_access_time;
	static struct stat checkLayoutStat;
	
	char layout[1000];
	int max_line=5000;
	char recvline[max_line];
	char *layout_ptr;
		
	if (using_files)
	{

		stat(check_layout_file,&checkLayoutStat);
		check_layout_access_time=0;
		
		static clock_t first_access_time=checkLayoutStat.st_mtime;
		if (first_access_time>=checkLayoutStat.st_mtime)
			return false;
			
		is_new_layout=(checkLayoutStat.st_mtime-check_layout_access_time!=0);
		
	}
	else
	{

		int sockfd = socket(host_info->ai_family, host_info->ai_socktype, host_info->ai_protocol);
		
		if (sockfd<0)
		{
			printf("can't create socket\n");
			return false;
		}
		
		if (connect(sockfd, host_info->ai_addr, host_info->ai_addrlen)<0)
		{
			printf("can't create connection\n");
			return false;
		}
		
	
		
		char run_string[10];
		sprintf(run_string, "%d", run_id);
		
		char sendline[500];
		sprintf(sendline, 
		     "GET /design/computeGetLayout HTTP/1.0\r\n" 
		     "Host: %s\r\n"    
		     "Content-type: application/x-www-form-urlencoded\r\n"
		     "Content-length: %d\r\n\r\n"
		     "%s\r\n", host_name,(unsigned int)strlen(run_string), run_string);
		     
		

		memset(recvline,0,max_line);
		size_t n;
		
		if (write(sockfd, sendline, strlen(sendline))>= 0) 
		{
		    while ((n = read(sockfd, recvline, max_line)) > 0) 
		    {
		        recvline[n] = '\0';
		    }          
		}
		close(sockfd);
		//printf("received: %s\n", recvline);
		
		int ret_code=0;
		float http_version;
		int idx=sscanf(recvline, "HTTP/%f %d OK",&http_version,&ret_code);
		
		layout_ptr=recvline;
		if (ret_code==200)
		{

			//printf("received: %s\n", layout_ptr);
			layout_ptr=strstr(layout_ptr, "Content-Type:");
			
			if (layout_ptr==0)
			{
				printf("Err: %s\n", recvline);
				return false;
			}
			layout_ptr=strstr(layout_ptr, "\n");
			
			if (layout_ptr==0)
			{
				printf("Err: %s\n", recvline);
				return false;
			}
			layout_ptr+=3;			
		}
				
				
				
		int layout_number;

		int ret=sscanf(layout_ptr,"%d\ndesign%s",&layout_number,layout);
		//printf("layout number is %d, ret %d\n",layout_number,ret);
		
		if (ret!=2)
		{
			//printf("bad line %i%s\n",recvline);
			return false;
		}
		else
		{
			is_new_layout=layout_number>h_d->layout_counter;
			//is_new_layout=false;
		}
		
		
	}
	

	if (is_new_layout)
	{
		printf("Check layout has changed, loading...\n");

		int num_regions, layout_counter;
		
		float * check_layout;
		/*
		if (using_files)
		{
			printf("Check layout file has changed, loading new file\n");
			
			check_layout=readLayoutFromFile(h_d,check_layout_file,&num_regions,&layout_counter);
	
			if (!check_layout)
			{
				printf("problem loading layout file. try next iteration\n");
				return false;
			}
			check_layout_access_time=checkLayoutStat.st_mtime;
		}
		else
		{*/
		
		//printf("parsing layout\n");
		check_layout=parseLayout(h_d,layout_ptr,&num_regions,&layout_counter);

		if (!check_layout)
		{
			printf("problem getting new layout from server. try next iteration\n");
			return false;
		}
			
			
			
			


	
		//float *directions;
	 	//int num_dir=getConstraintDirections(h_d, check_layout, &directions,false);
		/*
		
		//already has a layout, modify the parameters
		if (h_d->check_layout_exists)
		{
			
			//printf("Modifying parameters\n");
			


			/*
			int large_change=0;
			for (int i=0;i< ne;i++)
			{
				if ((abs(h_d->check_layout[i*NUM_VAR]-check_layout[i*NUM_VAR])>(100.0/h_d->width)) or (abs(h_d->check_layout[i*NUM_VAR+1]-check_layout[i*NUM_VAR+1])>(100.0/h_d->height)))
					large_change++;
			}
			
			if ((large_change==1))
			{
				
				
				float *h_layouts = (float *)malloc((h_d->layout_size)*num_kernels*sizeof(float));
				CHECKCALL(hipMemcpy(h_layouts, layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyDeviceToHost));
				
				for (int k=0;k<num_kernels;k++)
				{
					float *old_layout=&(h_layouts[k*((h_d)->layout_size)]);
					for(int i=0;i<ne;i++)
					{
						if (h_d->type[i]==1)		
							old_layout[i*NUM_VAR+2]=((MIN_TEXT_SIZE*max(h_d->width,h_d->height)*h_d->num_lines[i])/10.0)/h_d->height + (5.0/h_d->height);
						else
							old_layout[i*NUM_VAR+2]=MIN_GRAPHIC_SIZE+(5.0/h_d->height);
					}
				}
				
				CHECKCALL(hipMemcpy(layouts,h_layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyHostToDevice));
				free(h_layouts);
			}
	
			
				
			float *h_layouts = (float *)malloc((h_d->layout_size)*num_kernels*sizeof(float));
			CHECKCALL(hipMemcpy(h_layouts, layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyDeviceToHost));
			
			memcpy(h_layouts, check_layout, h_d->layout_size*sizeof(float));
			
			CHECKCALL(hipMemcpy(layouts,h_layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyHostToDevice));
			free(h_layouts);		
			
			
		}
		*/
	
		printf("align lines %i\n",h_d->num_constraints);
		if (h_d->num_constraints>0)
		{
			for (int i =0;i<4;i++)
				printf ("%i ",h_d->constraints[i]);
			printf("\n");
		}


		printf("Finished loading new layout %i. num regions %i\n",layout_counter,num_regions);
			
		int ne=h_d->num_elements;
	
		for(int i=0;i< ne;i++)
			h_d->check_layout_distances[i]=-1;

		memcpy(h_d->check_layout, check_layout, h_d->layout_size*sizeof(float));
		h_d->check_layout_exists=true;
		h_d->layout_counter=layout_counter;

		h_d->region_proposals=true;	
		h_d->fixed_regions=false;
		
		

		float *h_layouts = (float *)malloc((h_d->layout_size)*num_kernels*sizeof(float));
		CHECKCALL(hipMemcpy(h_layouts, layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyDeviceToHost));
		
		for (int k=0;k<num_kernels;k++)
		{
			float *old_layout=&(h_layouts[k*((h_d)->layout_size)]);
			
			
			for (int i=0;i<ne;i++)
			{
				int update=2;
				if (strcmp(param_type,"autoupdate")!=0)
					update=10;
				
				if ((check_layout[i*NUM_VAR+4]>FIX_LAYOUT_THRESH) || ((k%update==1)) || (num_regions==-2))
					for (int j=0;j<NUM_VAR;j++)
						old_layout[i*NUM_VAR+j]=check_layout[i*NUM_VAR+j];
						
				old_layout[i*NUM_VAR+4]=check_layout[i*NUM_VAR+4];
				
			}				
		}
		CHECKCALL(hipMemcpy(layouts,h_layouts,num_kernels*(h_d->layout_size)*sizeof(float), hipMemcpyHostToDevice));
		free(h_layouts);



		float *h_params_grad_temp=(float *)malloc(num_params*sizeof(float));
		float check_layout_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grad, check_layout,h_params, h_params_grad_temp,0,0,false);
		
		h_d->refine=true;
		if ((h_params_grad_temp[TEXT_OVERLAP_FEAT]>0.01) || (h_params_grad_temp[GRAPHIC_OVERLAP_FEAT]>0.01)  || (h_params_grad_temp[GRAPHIC_TEXT_OVERLAP_FEAT]>0.01) )
			h_d->refine=false;
		
			
		freeDeviceDesign(d);
		copyDesignToDevice(d,h_d);
		
		
		
		printf("align lines %i\n",h_d->num_constraints);
		if (h_d->num_constraints>0)
		{
			for (int i =0;i<4;i++)
				printf ("%i ",h_d->constraints[i]);
			printf("\n");
		}
		
	
		printf("evaluating new layout\n");
		float check_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grad, check_layout,h_params, h_params_grad_temp,0,0,true);

		FILE *fp=fopen(check_layout_features_file,"w");

		float test_eval=-500;
		if (fp>0)
		{
			fprintf(fp,"Layout Energy: %f\n",check_eval);
			for (int i=0;i<NUM_FEATURES;i++)
			{
				fprintf(fp,"%i\t%5.1f \t %4.2f\t %4.1f \tnl:%3.1f %s \n", i,h_params[i],h_params_grad_temp[i],h_params[i]*h_params_grad_temp[i],h_params[i+NUM_FEATURES],feat_names[i]);
				test_eval+=h_params[i]*h_params_grad_temp[i];
			}
			fprintf(fp,"Eval: %f %f\n",check_eval,test_eval*0.25);
			fclose(fp);
		}
		
		free(h_params_grad_temp);
	
	
		printf("Finished updating check layout\n");
		
	
	
	
		
		return true;


	}
	return false;
}



/*
void sampleStyleParameter(float **h_params, float *params, int num_params)
{
	
		float *h_params_new= loadParametersFromFile(default_param_file, param_file,num_params);
		free(*h_params);
		*h_params=h_params_new;
		CHECKCALL(hipMemcpy(params,*h_params, num_params*sizeof(float), hipMemcpyHostToDevice));
}
*/


bool updateParameterFile(float **h_params, float *params, int num_params)
{


	static struct stat paramFileStat;
	stat(param_file,&paramFileStat);
	static clock_t param_access_time=paramFileStat.st_mtime;

	//printf("Checking file %s %d\n",param_file,param_access_time);

	if (paramFileStat.st_mtime-param_access_time!=0)
	{
		printf("Parameter file has changed, loading new file\n");
		param_access_time=paramFileStat.st_mtime;
		float *h_params_new= loadParametersFromFile(default_param_file, param_file,num_params);
		free(*h_params);
		*h_params=h_params_new;
		CHECKCALL(hipMemcpy(params,*h_params, num_params*sizeof(float), hipMemcpyHostToDevice));
		return true;
	}


	static struct stat paramChangeFileStat;
	stat(param_change_file,&paramChangeFileStat);
	static clock_t param_change_access_time=paramChangeFileStat.st_mtime;

	//printf("Checking file %s %d\n",param_file,param_access_time);

	if (paramChangeFileStat.st_mtime-param_change_access_time!=0)
	{
		printf("Parameter change file has changed, loading new file\n");
		param_change_access_time=paramChangeFileStat.st_mtime;

		//load parameters
		float *h_params_new= loadParametersFromFile(default_param_file, param_file,num_params);

		addParameterOffsets(param_change_file,num_params,h_params_new);

		free(*h_params);
		*h_params=h_params_new;
		CHECKCALL(hipMemcpy(params,*h_params, num_params*sizeof(float), hipMemcpyHostToDevice));
		return true;
	}


	return false;
}

bool loadParameterSample(float **h_params, float *params, int num_params)
{

	int r = (rand() / (double)RAND_MAX)*999;

	char param_file[1024];	
	sprintf(param_file,"%sdata/style_samples/gen_style%i.data",home_dir,r);	
	
	float *h_params_new= loadParametersFromFile(default_param_file, param_file,num_params);

	free(*h_params);
	*h_params=h_params_new;
	CHECKCALL(hipMemcpy(params,*h_params, num_params*sizeof(float), hipMemcpyHostToDevice));
	return true;


}


bool updateDesign(Design *d,Design **h_d, float **opt_layouts,float **h_opt_layouts,int num_kernels )
{
	static struct stat designFileStat;
	stat(design_file,&designFileStat);
	static clock_t design_access_time=designFileStat.st_mtime;

	//printf("Checking file %s %d\n",design_file,design_access_time);

	if (designFileStat.st_mtime-design_access_time!=0)
	{



		/*
		for (int k=0;k<3;k++)
		{
			printf("layout kernel %i\n",k);
			float *new_layout=&((*h_opt_layouts)[k*((*h_d)->layout_size)]);

			for(int j=0;j<(*h_d)->num_elements;j++)
			{
				printf("setting %i %i, %f,%f,%f,%f,%f \n", k,j,new_layout[j*NUM_VAR], new_layout[j*NUM_VAR+1], new_layout[j*NUM_VAR+2], new_layout[j*NUM_VAR+3], new_layout[j*NUM_VAR+4]);
			}
		}
		*/

		Design *h_d_new= loadDesignFromFile(design_file,true);
		if (!h_d_new)
		{
			printf("problem loading design file. try next iteration");
			return false;
		}
		
		
		printf("Design file has changed, loading new file\n");
		design_access_time=designFileStat.st_mtime;

		//if (h_d_new->num_elements!=(*h_d)->num_elements)
		//{

		float *h_opt_layouts_new = (float *)malloc((h_d_new->layout_size)*num_kernels*sizeof(float));
		memset(h_opt_layouts_new,0,(h_d_new->layout_size)*num_kernels*sizeof(float));

		//printf("num_kernels %i\n",num_kernels);
		//printf("old layout size %i\n",((*h_d)->layout_size));
		//printf("new layout size %i\n",(h_d_new->layout_size));

		
		for (int k=0;k<num_kernels;k++)
		{
			//printf("Copying for kernel %i\n",k);
			float *old_layout=&((*h_opt_layouts)[k*((*h_d)->layout_size)]);
			float *new_layout=&(h_opt_layouts_new[k*(h_d_new->layout_size)]);

			memcpy(new_layout,h_d_new->layout, h_d_new->layout_size*sizeof(float));


			//int offset_old=(*h_d)->num_elements*NUM_VAR;
			//int offset_new=h_d_new->num_elements*NUM_VAR;

			for(int j=0;j<h_d_new->num_elements;j++)
			{
				for(int i=0;i<(*h_d)->num_elements;i++)
				{
					if ((*h_d)->id[i]== h_d_new->id[j])
					{
						for(int n=0;n<NUM_VAR;n++)
							if ((n!=3) && (n!=6))
								new_layout[j*NUM_VAR+n]=old_layout[i*NUM_VAR+n];
					}
				}
			}
			
			/*
			if ((*h_d)->num_elements==h_d_new->num_elements)
			{
				for(int j=0;j<(*h_d)->num_elements;j++)
				{													
					for(int n=0;n<NUM_RVAR;n++)						
						new_layout[offset_new+j*NUM_RVAR+n]=old_layout[offset_old+j*NUM_RVAR+n];
				}	
			}
			else
			{
				for(int j=0;j<h_d_new->num_elements;j++)
				{
					new_layout[j*NUM_VAR+5]=j;
					//new_layout[offset_new+j*NUM_RVAR]
				}
			}
			*/
		}

		cout << "finished setting layout "  <<endl;

		free(*h_opt_layouts);
		*h_opt_layouts=h_opt_layouts_new;


		//}


		*h_d=h_d_new;
		freeDeviceDesign(d);
		copyDesignToDevice(d,h_d_new);

		return true;
	}
	return false;
}


/*

int optimizeLayout(Design *d,int num_kernels,int num_params,int num_outer,int num_inner, float *params,float *temperatures,int *temp_ids,float *layouts,float *params_grads,float *opt_layouts, float *evals,float *proposals,Design *h_d,float *h_temperatures,float *h_evals,int *h_temp_ids,float *h_params_grads,float *h_opt_layouts,bool silent,bool output_layouts,int debug_mode)
{


	float end_temp=0.6;
	float start_temp=0.01;

	//float temp_range=0.6;
	//float range_step=temp_range/num_outer;
	printf("end temp %f\n",end_temp);


    for (int i=0;i< num_kernels;i++)
    {
    	h_temp_ids[i]=i;
    	h_temperatures[i]=10.0*((end_temp-start_temp)* (float(num_kernels-i)/float(num_kernels)))+start_temp;
    }


    CHECKCALL(hipMemcpy(temperatures, h_temperatures,num_kernels*sizeof(float), hipMemcpyHostToDevice));
    CHECKCALL(hipMemcpy(temp_ids, h_temp_ids,num_kernels*sizeof(int), hipMemcpyHostToDevice));


    //for (int i=0;i< num_kernels;i++)
    //	CHECKCALL(hipMemcpy(&layouts[i*h_d->layout_size], h_d->layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    dim3 cudaGridSize(16, 1);
    dim3 cudaBlockSize(num_kernels/16,1);
    //dim3 cudaGridSize(1, 1);
    //dim3 cudaBlockSize(1,1);


    float *h_params= (float *)malloc(num_params*sizeof(float));
    CHECKCALL(hipMemcpy(h_params, params,num_params*sizeof(float), hipMemcpyDeviceToHost));

	float *atan_params;
	CHECKCALL(hipMalloc(&atan_params,num_params*sizeof(float)));
	float *h_atan_params= (float *)malloc(num_params*sizeof(float));

	for (int j=0;j<num_params;j++)
		h_atan_params[j]=atan(h_params[j]);
	CHECKCALL(hipMemcpy(atan_params, h_atan_params,num_params*sizeof(float), hipMemcpyHostToDevice));



	float *previous_layout;
	CHECKCALL(hipMalloc(&previous_layout,h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMemset(previous_layout, 0,h_d->layout_size*sizeof(float)));


	float *random_seed;
	CHECKCALL(hipMalloc(&random_seed,num_kernels*4*sizeof(float)));
	CHECKCALL(hipMemset(random_seed, 0,num_kernels*4*sizeof(float)));

	float *h_random_seed= (float *)malloc(num_kernels*4*sizeof(float));

	//static struct stat paramFileStat;
	//if(stat(param_file,&paramFileStat) < 0)
	//	return -1;
	//static clock_t param_access_time=paramFileStat.st_atime;


	char layout_file[1024],layout_features_file[1024],check_layout_features_file[1024];
	sprintf(layout_file, "%sdata/runs/r%d_opt_layout.data",home_dir,run_id);
	sprintf(layout_features_file, "%sdata/runs/r%d_opt_layout_features.txt",home_dir,run_id);
	sprintf(check_layout_features_file, "%sdata/runs/r%d_user_layout_features.txt",home_dir,run_id);

	int best_id=0;
	float opt_eval=999999;
	float curr_opt_eval=opt_eval;
	bool new_params=true;
	//float *h_params_new;

    clock_t start=clock();
    clock_t stop;
    float pt_time_sum=0,pt_time_cnt=0;

    printf("Starting optimization with %i iterations\n",num_outer);
    

    for (int i=0;i <num_outer;i++)
    {


    	if (i%3==1)
    	{


    		if (updateParameterFile(&h_params, params, num_params) or updateCheckLayoutFile(d, h_d,num_params,num_kernels,params,params_grads,layouts,  h_params,check_layout_features_file))
    		{
    			opt_eval=999999;
    			new_params=true;
    			
    			
    			//todo: move this to updateParameterFile
				for (int j=0;j<num_params;j++)
					h_atan_params[j]=atan(h_params[j]);
				CHECKCALL(hipMemcpy(atan_params, h_atan_params,num_params*sizeof(float), hipMemcpyHostToDevice));
    			
    			
    		}

    		if (updateDesign(d, &h_d,&opt_layouts,&h_opt_layouts,num_kernels))
    		{
    			opt_eval=999999;
    			new_params=true;



    			pt_time_sum=0;
    			pt_time_cnt=1;
    			printf("Layout Size %i\n", h_d->layout_size);


    			CHECKCALL(hipFree(opt_layouts));
    			CHECKCALL(hipMalloc(&opt_layouts,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(opt_layouts, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    			CHECKCALL(hipFree(layouts));
    			CHECKCALL(hipMalloc(&layouts,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(layouts, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    			CHECKCALL(hipFree(proposals));
    			CHECKCALL(hipMalloc(&proposals,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(proposals, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

				CHECKCALL(hipFree(previous_layout));
				CHECKCALL(hipMalloc(&previous_layout,h_d->layout_size*sizeof(float)));
				CHECKCALL(hipMemcpy(previous_layout, h_d->layout, h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
    		}
    	}
    	
    	for (int rs=0;rs<num_kernels*4;rs++)
    		h_random_seed[rs]=(float)rand()/(float)RAND_MAX;

		CHECKCALL(hipMemcpy(random_seed, h_random_seed,num_kernels*4*sizeof(float), hipMemcpyHostToDevice));


    	clock_t start_pt=clock();
    	parallelTempering<<<cudaGridSize, cudaBlockSize>>>(d,debug_mode,i, -1,-1, num_inner,num_params,params, atan_params,temperatures,temp_ids ,layouts,proposals, params_grads, opt_layouts,evals,num_previous_layout, previous_layout,random_seed);


    	CHECKCALL(hipGetLastError());
    	CHECKCALL(hipDeviceSynchronize());

    	clock_t stop_pt=clock();
    	pt_time_sum+=stop_pt-start_pt;
    	pt_time_cnt++;

    	CHECKCALL(hipMemcpy(h_evals, evals,  num_kernels*sizeof(float), hipMemcpyDeviceToHost));


		
    	//switch layouts based on
    	for (int j=0;j< num_kernels-1;j++)
    	{
    		int r1=h_temp_ids[j];
    		int r2=h_temp_ids[j+1];

    		float temp1=h_temperatures[r1];
    		float temp2=h_temperatures[r2];

    		float energy1=h_evals[r1];
    		float energy2=h_evals[r2];
    		
    		if ((energy1==9999) || (energy2==9999))
    		{
    			printf("quitting\n");
    			return -1;
    			
    		}
    			

    		float prop=exp((1.0/temp1-1.0/temp2)*(energy1-energy2));
    		float r=rand()/ double(RAND_MAX);			

    		if (r<min(1.0,prop))
			{
				int temp_id=h_temp_ids[j];
				h_temp_ids[j]=h_temp_ids[j+1];
				h_temp_ids[j+1]=temp_id;

				float temp_temp=h_temperatures[r1];
				h_temperatures[r1]=h_temperatures[r2];
				h_temperatures[r2]=temp_temp;

			}
    	}
    	
    	

    	CHECKCALL(hipMemcpy(temperatures, h_temperatures,num_kernels*sizeof(float), hipMemcpyHostToDevice));

        stop=clock();
        
        float energy_sum=0;


    	best_id=0;
    	curr_opt_eval=h_evals[0]+1;
    	for (int j=0;j < num_kernels;j++)
    	{
    		energy_sum+=h_evals[j];
    		if (h_evals[j]<curr_opt_eval)
    		{
    			curr_opt_eval=h_evals[j];
    			best_id=j;
    		}
    	}
    	
    	

    	
    	if ((output_layouts) and (i%3==1) or (i%10==1))
    	{
    		CHECKCALL(hipMemcpy(h_opt_layouts, opt_layouts, num_kernels*h_d->layout_size*sizeof(float), hipMemcpyDeviceToHost));
			float *output_layout=&h_opt_layouts[(best_id)*h_d->layout_size];

			if ((curr_opt_eval<opt_eval) or (new_params) or (i%10==1))
			{
				if (curr_opt_eval<opt_eval)
					opt_eval=curr_opt_eval;
				
				writeLayoutToFile(h_d, output_layout,layout_file );

				new_params=false;
				CHECKCALL(hipMemcpy(h_params_grads, params_grads, num_kernels*num_params*sizeof(float), hipMemcpyDeviceToHost));

				CHECKCALL(hipFree(previous_layout));
				CHECKCALL(hipMalloc(&previous_layout,h_d->layout_size*sizeof(float)));
				CHECKCALL(hipMemcpy(previous_layout, output_layout, h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

				float *h_params_grad_opt=&h_params_grads[best_id*num_params];
				FILE *fp=fopen(layout_features_file,"w");


				if (fp>0)
				{
				
			
					//if (curr_opt_eval<opt_eval) 
					//float *h_params_grad_temp=(float *)malloc(num_params*sizeof(float));
					//float check_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grads, output_layout,h_params, h_params_grad_temp,true);
					
					float test_eval=-500;
					fprintf(fp,"Layout Energy: %f\n",opt_eval);
					for (int k=0;k<NUM_FEATURES;k++)
					{
						fprintf(fp,"%i\t%5.1f \t %4.2f\t %4.2f \t nl: %4.2f %s \n", k,h_params[k],h_params_grad_opt[k],h_params[k]*h_params_grad_opt[k],h_params[k+NUM_FEATURES],feat_names[k]);
						test_eval+=h_params[k]*h_params_grad_opt[k];
					}


					fprintf(fp,"Eval: %f %f\n",opt_eval,test_eval*0.25);
					fclose(fp);	

					//free(h_params_grad_temp);
				}



			}

    	}

        if ((output_layouts) and (i%20==1) )
        {
        	float overall_time=((double)(stop - start) / CLOCKS_PER_SEC);
        	float pt_time=((double)(stop_pt - start_pt) / CLOCKS_PER_SEC);

        	printf("PT Iteration %i, steps %i, opt %f, overall time %f, pt time %f (mean %f), mean energy %f debug_mode %i,region proposals %i\n", i,num_inner,opt_eval,overall_time, pt_time, (pt_time_sum/pt_time_cnt)/ CLOCKS_PER_SEC,energy_sum/float(num_kernels),debug_mode,h_d->region_proposals);
        }

        start=stop;
    }

    free(h_params);
    free(h_atan_params);
    free(h_random_seed);
    CHECKCALL(hipFree(atan_params));
    CHECKCALL(hipFree(random_seed));
    //CHECKCALL(hipFree(check_layout));

	CHECKCALL(hipMemcpy(h_evals, evals,  num_kernels*sizeof(float), hipMemcpyDeviceToHost));
	CHECKCALL(hipMemcpy(h_params_grads, params_grads, num_kernels*num_params*sizeof(float), hipMemcpyDeviceToHost));
	CHECKCALL(hipMemcpy(h_opt_layouts, opt_layouts, num_kernels*h_d->layout_size*sizeof(float), hipMemcpyDeviceToHost));


	best_id=0;
	opt_eval=h_evals[0]+1;
	for (int j=0;j < num_kernels;j++)
	{
		if (h_evals[j]<opt_eval)
		{
			opt_eval=h_evals[j];
			best_id=j;
		}
	}
	return best_id;

}

*/



int optimizeLayout(Design *d,int num_kernels,int num_params,int num_outer,int num_inner, float *params,float *temperatures,int *temp_ids,float *layouts,float *params_grads,float *opt_layouts, float *evals,float *proposals,Design *h_d,float *h_temperatures,float *h_evals,int *h_temp_ids,float *h_params_grads,float *h_opt_layouts,bool silent,int debug_mode,float *h_opt_steps)
{


	float end_temp=0.6;
	float start_temp=0.01;

	//float temp_range=0.6;
	//float range_step=temp_range/num_outer;
	//printf("end temp %f\n",end_temp);

	
	int num_ladders=2;
	int ladder_size=num_kernels/num_ladders;

    for (int i=0;i< num_ladders;i++)
    for (int j=0;j< ladder_size;j++)
    	h_temperatures[i*ladder_size+j]=10.0*((end_temp-start_temp)* (float(ladder_size-j)/float(ladder_size)))+start_temp;
   
  	/*
  	int grid_size=sqrt(num_kernels);
    
    for (int i=0;i< grid_size;i++)
    for (int j=0;j< grid_size;j++)
    	h_temperatures[i*grid_size+j]=10.0*((end_temp-start_temp)* (float(grid_size-j)/float(grid_size)))+start_temp;
    */
    
    //for (int i=0;i< num_kernels;i++)
    //{
    //	h_temperatures[i]=((end_temp-start_temp)* (float(i)/float(num_kernels)))+start_temp;
    //}


    CHECKCALL(hipMemcpy(temperatures, h_temperatures,num_kernels*sizeof(float), hipMemcpyHostToDevice));
    //CHECKCALL(hipMemcpy(temp_ids, h_temp_ids,num_kernels*sizeof(int), hipMemcpyHostToDevice));


    //for (int i=0;i< num_kernels;i++)
    //	CHECKCALL(hipMemcpy(&layouts[i*h_d->layout_size], h_d->layout,h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    dim3 cudaGridSize(16, 1);
    dim3 cudaBlockSize((num_kernels/16)*EVAL_SPLIT_NUM,1);
    //dim3 cudaGridSize(1, 1);
    //dim3 cudaBlockSize(1,1);


    float *h_params= (float *)malloc(num_params*sizeof(float));
    CHECKCALL(hipMemcpy(h_params, params,num_params*sizeof(float), hipMemcpyDeviceToHost));

	float *atan_params;
	CHECKCALL(hipMalloc(&atan_params,num_params*sizeof(float)));
	float *h_atan_params= (float *)malloc(num_params*sizeof(float));

	for (int j=0;j<num_params;j++)
		h_atan_params[j]=atan(h_params[j]);
	CHECKCALL(hipMemcpy(atan_params, h_atan_params,num_params*sizeof(float), hipMemcpyHostToDevice));


	
	int *barrier, *h_barrier;
	//int *barrier, *h_barrier=(int *)malloc(num_kernels*sizeof(int));
	//for (int j=0;j<num_kernels;j++)
	//	h_barrier[j]=EVAL_SPLIT_NUM;
	//CHECKCALL(hipMalloc(&barrier,num_kernels*sizeof(int)));
	//CHECKCALL(hipMemcpy(barrier, h_barrier,num_kernels*sizeof(int), hipMemcpyHostToDevice));
	
	float *eval_sum;
	CHECKCALL(hipMalloc(&eval_sum,num_kernels*sizeof(float)));
	CHECKCALL(hipMemset(eval_sum, 0,num_kernels*sizeof(float)));	

	float *previous_layout;
	CHECKCALL(hipMalloc(&previous_layout,num_outer*h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMemset(previous_layout, 0,num_outer*h_d->layout_size*sizeof(float)));

	float *h_previous_layout= (float *)malloc(num_outer*h_d->layout_size*sizeof(float));

	float *h_best_layout= (float *)malloc(h_d->layout_size*sizeof(float));

	//static struct stat paramFileStat;
	//if(stat(param_file,&paramFileStat) < 0)
	//	return -1;
	//static clock_t param_access_time=paramFileStat.st_atime;

	float *random_seed;
	CHECKCALL(hipMalloc(&random_seed,num_kernels*4*sizeof(float)));
	CHECKCALL(hipMemset(random_seed, 0,num_kernels*4*sizeof(float)));

	float *h_random_seed= (float *)malloc(num_kernels*4*sizeof(float));
	
	

	
	int best_id=0;
	float opt_eval=999999;
	float curr_opt_eval=opt_eval;
	//bool new_params=true;
	//float *h_params_new;
	clock_t last_time=clock();
    clock_t start=clock();
    clock_t stop;
    float pt_time_sum=0;
    float it_time_sum=0;

	cout << "Starting optimization with "<< num_outer<<" iterations " << endl;
    
    
    int num_previous_layout=0;
    
    int last_add=-100;
    
    bool gallery=(strcmp(param_type,"gallery")==0);	
    bool nio=(strcmp(param_type,"nio_init")==0);
	
    	
    float refine_design= ((!gallery) && (!nio));
    	
    float gd_imp_sum=0,gd_time_sum=0;
    int gd_cnt=0,gd_imp_fail=0;
    
    float *h_params_grad_temp=(float *)malloc(num_params*sizeof(float));
    
    int last_update=-1;
    clock_t last_update_time=clock();

    for (int r=0;r <num_outer;r++)
    {
		//printf("Iteration %i\n",r);
		
		//printf("iteration %i\n",r);
    	
		
    	//if (r%2==1)
    	//{

			//updateParameterFile(&h_params, params, num_params) or 
    		if ((refine_design)&& updateCheckLayout(d, h_d,num_params,num_kernels,params,params_grads,layouts,  h_params,check_layout_features_file))
    		{
    			
    			last_update=r;
    			
    			opt_eval=999999;
    			//new_params=true;
    			last_update_time=clock();

			    			
    		}

    		if (updateDesign(d, &h_d,&opt_layouts,&h_opt_layouts,num_kernels))
    		{
    			opt_eval=999999;
    			//new_params=true;


    			pt_time_sum=0;
    			printf("Layout Size %i\n", h_d->layout_size);


    			CHECKCALL(hipFree(opt_layouts));
    			CHECKCALL(hipMalloc(&opt_layouts,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(opt_layouts, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    			CHECKCALL(hipFree(layouts));
    			CHECKCALL(hipMalloc(&layouts,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(layouts, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    			CHECKCALL(hipFree(proposals));
    			CHECKCALL(hipMalloc(&proposals,num_kernels*h_d->layout_size*sizeof(float)));
    			CHECKCALL(hipMemcpy(proposals, h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

				CHECKCALL(hipFree(previous_layout));
				CHECKCALL(hipMalloc(&previous_layout,num_outer*h_d->layout_size*sizeof(float)));
				CHECKCALL(hipMemcpy(previous_layout, h_d->layout, h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));

    		}
    	//}

		clock_t start_pt=clock();
		
    	parallelTempering<<<cudaGridSize, cudaBlockSize>>>(d,debug_mode,r, -1,-1, num_inner,num_params,params, atan_params,temperatures,temp_ids ,layouts,proposals, params_grads, opt_layouts,evals,num_previous_layout, previous_layout,random_seed,barrier,eval_sum);


    	CHECKCALL(hipGetLastError());
    	CHECKCALL(hipDeviceSynchronize());

    	clock_t stop_pt=clock();
    	pt_time_sum+=stop_pt-start_pt;

    	CHECKCALL(hipMemcpy(h_evals, evals,  num_kernels*sizeof(float), hipMemcpyDeviceToHost));
		CHECKCALL(hipMemcpy(h_opt_layouts, opt_layouts, num_kernels*h_d->layout_size*sizeof(float), hipMemcpyDeviceToHost));


		//for (int i=0;i< num_kernels;i++)
		//	for (int j=0;j<h_d->num_elements;j++)
		//		if ((h_opt_layouts[i*h_d->layout_size + j*NUM_VAR+2]==0) && (h_opt_layouts[i*h_d->layout_size+ j*NUM_VAR+3]==0))
		//			printf("ERROR-1 in kernel %i element %i. init element is 0\n",i,j);
    	
    	
    	//do refinement updates first
		for (int i=0;i< num_ladders;i++)
		{
	    	for (int j=0;j< ladder_size-1;j++)
	    	{
	    		int curr=i*ladder_size+j;
	    		float temp1=h_temperatures[curr];
	    		float temp2=h_temperatures[curr+1];
	
	    		float energy1=h_evals[curr];
	    		float energy2=h_evals[curr+1];
	
	    		float prop=exp((1.0/temp1-1.0/temp2)*(energy1-energy2));
	    		float r=rand()/ double(RAND_MAX);			
						
				
				//switch state
	    		if (r<min(1.0,prop))
				{
					
					float temp_eval=h_evals[curr];
					h_evals[curr]=h_evals[curr+1];
					h_evals[curr+1]=temp_eval;
					
					for (int k=0;k<h_d->layout_size;k++)
					{
						float temp_layout=h_opt_layouts[curr*h_d->layout_size+k];
						h_opt_layouts[curr*h_d->layout_size+k]=h_opt_layouts[(curr+1)*h_d->layout_size+k];
						h_opt_layouts[(curr+1)*h_d->layout_size+k]=temp_layout;
					}
					
					
					//just promoted a new state to the top rung, move it to the next ladder.
					if ((i<num_ladders-1)&& (j==ladder_size-2))
					{
						//printf("reseting refinement optimization %.2f %.2f. temp %.2f %.2f. energy %.2f %.2f \n",r,prop,temp1, temp2, energy1,energy2);
						
						for (int m=0;m< ladder_size;m++) 
						{
							h_evals[(i+1)*ladder_size+m]=h_evals[curr+1];	
							
							for (int k=0;k<h_d->layout_size;k++)
								h_opt_layouts[((i+1)*ladder_size+m)*h_d->layout_size+k]=h_opt_layouts[(curr+1)*h_d->layout_size+k];
								
						}
						
					}
				
				}
			}
    	}


    	CHECKCALL(hipMemcpy(evals, h_evals,num_kernels*sizeof(float), hipMemcpyHostToDevice));
    	//CHECKCALL(hipMemcpy(opt_layouts,h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
		CHECKCALL(hipMemcpy(layouts,h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
		
		
		
        
        
        float energy_sum=0;


    	best_id=0;
    	curr_opt_eval=h_evals[0]+1;
    	for (int j=0;j < num_kernels;j++)
    	{
    		energy_sum+=h_evals[j];
    		if (h_evals[j]<curr_opt_eval)
    		{
    			curr_opt_eval=h_evals[j];
    			best_id=j;
    			
    			
    		}    		
    	}
      	
      	h_opt_steps[r]=opt_eval;
    	
    	
    	
    	
    	float *output_layout=&h_opt_layouts[(best_id)*h_d->layout_size];
    	
    	
    	bool write_layout=(r-last_update>0);
    	
		
		if ((gallery) && (r>120) && (h_opt_steps[r-50]-h_opt_steps[r]<1) && (r-last_add>120))
		{
		
			last_add=r;
			

			
			//check distance to previous layouts
			
			float min_dist=999;
			
			for (int i=0;i< num_previous_layout;i++)
			{
			
				float *prev_layout =&(h_previous_layout[i*h_d->layout_size]);
				float curr_dist=0;
				
				for (int j=0;j<h_d->num_elements;j++)
				{
					curr_dist+=abs(prev_layout[j*NUM_VAR]-output_layout[j*NUM_VAR]);
					curr_dist+=abs(prev_layout[j*NUM_VAR+1]-output_layout[j*NUM_VAR+1]);
					curr_dist+=2*abs(prev_layout[j*NUM_VAR+2]-output_layout[j*NUM_VAR+2]);
				}
					

				printf("dist to prev layout %i: %.3f\n",i,curr_dist/h_d->num_elements); 
				min_dist=min(curr_dist/h_d->num_elements,min_dist);
				
			}
			
			
			if (min_dist>0.1)
			{
				memcpy(&(h_previous_layout[num_previous_layout*h_d->layout_size]),output_layout,h_d->layout_size*sizeof(float));
				CHECKCALL(hipMemcpy(previous_layout, h_previous_layout, num_outer*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
				num_previous_layout++;
				
				//opt_eval=999999;
				printf("added previous layout, iter %i, count %i, curr_opt_eval %f\n",r,num_previous_layout,curr_opt_eval);
			
				CHECKCALL(hipMemcpy(h_best_layout,output_layout ,  h_d->layout_size*sizeof(float), hipMemcpyHostToHost));
	
				
				
				sendLayoutToServer(h_d, h_best_layout,curr_opt_eval);
				opt_eval=curr_opt_eval;
			
			}
			
		
			loadParameterSample(&h_params, params, num_params);
			
    			
			//todo: move this to updateParameterFile
			for (int j=0;j<num_params;j++)
				h_atan_params[j]=atan(h_params[j]);
			CHECKCALL(hipMemcpy(atan_params, h_atan_params,num_params*sizeof(float), hipMemcpyHostToDevice));
			
			
			write_layout=false;
			
		}
		
		if (write_layout)
		{	


			if ((curr_opt_eval<opt_eval))
			{
				
			
				CHECKCALL(hipMemcpy(h_best_layout,output_layout ,  h_d->layout_size*sizeof(float), hipMemcpyHostToHost));




				if (refine_design)
					sendLayoutToServer(h_d, h_best_layout,curr_opt_eval);
				
				opt_eval=curr_opt_eval;
				
			}
			else if ((r%10==1) && (refine_design))
				sendLayoutToServer(h_d, h_best_layout,opt_eval);
			
				/*
				float new_eval=curr_opt_eval;
				clock_t start_gd=clock();
				if (debug_mode==2)
					new_eval=constrainedGradientDescent(h_d,d,num_params, params,atan_params, params_grads, h_params,output_layout,num_previous_layout, previous_layout);
				clock_t end_gt=clock();
				
				
				gd_time_sum+=(end_gt-start_gd);
				gd_imp_sum+=abs(new_eval-curr_opt_eval);
				gd_cnt++;		
				
				
				if (new_eval==curr_opt_eval)
					gd_imp_fail++;
			
				if (new_eval<curr_opt_eval)
				{
					for (int m=0;m< ladder_size;m++) 
					{
						h_evals[ladder_size+m]=new_eval;	
						
						for (int k=0;k<h_d->layout_size;k++)
							h_opt_layouts[(ladder_size+m)*h_d->layout_size+k]=output_layout[k];
							
					}
					
	    			CHECKCALL(hipMemcpy(evals, h_evals,num_kernels*sizeof(float), hipMemcpyHostToDevice));
					CHECKCALL(hipMemcpy(layouts,h_opt_layouts,  num_kernels*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
					
				}
				else if (abs(new_eval-curr_opt_eval)>0.01)
					printf("ERROR. curr_opt_eval %f < new_eval %f\n",curr_opt_eval,new_eval);

					*/
					
			

			
	
			
    	}
    	
    	stop=clock();
		it_time_sum+=stop - start;

        if (r%40==1) 
        {
        	
        	
        	float time_since_last_update= (stop-last_update_time)  / CLOCKS_PER_SEC;
        	
        	
        	
        	if (time_since_last_update>300)
        	{
        		printf("inactive. quitting...");
        		cleanup();
        		exit(1);
        	}
        	
        	
        	
        	float overall_time=1000*(it_time_sum/r) / CLOCKS_PER_SEC;
        	float pt_time=1000*(pt_time_sum/r)/ CLOCKS_PER_SEC;
        	float gd_time=1000*(gd_time_sum/gd_cnt)/ CLOCKS_PER_SEC;
        		

			clock_t curr_time=clock();
			last_time=curr_time;
		
        	printf("PT Iteration %i, steps %i, opt %.2f, overall time mean %.2f, pt time mean %.2f, GD imp %.2f time %.2f fail %.2f\n", r,num_inner,opt_eval,overall_time, pt_time,gd_imp_sum/gd_cnt,gd_time,float(gd_imp_fail)/gd_cnt);
        	
        	writeHostFiles();
		
			//for (int j=0;j<h_d->num_elements;j++)
			//	printf("opt layout. elem %i %.1f %.1f %.1f %.2f %.2f\n", j,  h_best_layout[j*NUM_VAR]*h_d->width,h_best_layout[j*NUM_VAR+1]*h_d->height,h_best_layout[j*NUM_VAR+2]*h_d->height,h_best_layout[j*NUM_VAR+3],h_best_layout[j*NUM_VAR+4]);
			
			float check_eval=evaluateLayoutHost(d,h_d, num_params,params, params_grads, h_best_layout,h_params, h_params_grad_temp,num_previous_layout,previous_layout, false);
			
			if (nio)
				writeOutFeatures(layout_features_file2,check_eval,h_params,h_params_grad_temp);
			else
				writeOutFeatures(layout_features_file,check_eval,h_params,h_params_grad_temp);

		
			
        }

        start=stop;
    }

	free(h_params_grad_temp);
    free(h_params);
    free(h_atan_params);
    
    //CHECKCALL(hipFree(barrier));
    //CHECKCALL(hipFree(atan_params));
   // free(h_row_ids);
    //CHECKCALL(hipFree(check_layout));

	CHECKCALL(hipMemcpy(h_evals, evals,  num_kernels*sizeof(float), hipMemcpyDeviceToHost));
	CHECKCALL(hipMemcpy(h_params_grads, params_grads, num_kernels*num_params*sizeof(float), hipMemcpyDeviceToHost));
	CHECKCALL(hipMemcpy(h_opt_layouts, opt_layouts, num_kernels*h_d->layout_size*sizeof(float), hipMemcpyDeviceToHost));



	best_id=0;
	curr_opt_eval=h_evals[0]+1;
	for (int j=0;j < num_kernels;j++)
	{
		if (h_evals[j]<opt_eval)
		{
			curr_opt_eval=h_evals[j];
			best_id=j;
		}
	}
	
	
	h_evals[best_id]=opt_eval;
	CHECKCALL(hipMemcpy(&h_opt_layouts[best_id*h_d->layout_size], h_best_layout,  h_d->layout_size*sizeof(float), hipMemcpyHostToHost));
	free(h_best_layout);
	
	
	return best_id;

}



void writeHostFiles()
{
	int pid=getpid();
	
	FILE *fp=fopen(run_host_file,"w");
	if (fp>0)
		fprintf(fp,"%s %d\n",hostname,pid);
	fclose(fp);
	
	fp=fopen(host_run_file,"w");
	if (fp>0)
		fprintf(fp,"%d  %d\n",run_id,pid);
	fclose(fp);
	
	fp=fopen(host_pid_file,"w");
	if (fp>0)
		fprintf(fp,"%d\n",pid);
	fclose(fp);
}



int getConstraints(Design *d, float *layout, float ***C_out,  float ***H_out, bool debug)
{
	
	if (debug)
		printf("getting position constraints\n");

	int ne=d->num_elements;
	
	Box elem_bb[MAX_ELEMENTS];
	float aspect_ratio[MAX_ELEMENTS];
	
	float height, width, xp, yp;
	
	for (int i=0;i< d->num_elements;i++)
	{
		
		int alt=int(layout[NUM_VAR*i+6]);
		if (alt>-1)
		{
			if ((d->num_alt[i]>0))
				aspect_ratio[i]=d->alt_aspect_ratio[i*MAX_ALT+alt];
			else
				aspect_ratio[i]=d->aspect_ratio[i];
				
		
			height=round(layout[NUM_VAR*i+2]*d->height);
			width=round(height/aspect_ratio[i]);
			xp=round(layout[NUM_VAR*i]*d->width);
			yp=round(layout[NUM_VAR*i+1]*d->height);
	
			elem_bb[i].set_h(round(xp+(width*d->bb_left[i])), round(xp+(width*d->bb_right[i])), round(yp + (height*d->bb_bottom[i])), round(yp + (height*d->bb_top[i])));
			//printf("%i: %f %f %f %f (w/h) %f %f\n",i, layout[NUM_VAR*i],layout[NUM_VAR*i+1],layout[NUM_VAR*i+2],layout[NUM_VAR*i+3],width,height);
			
			if (debug)
				printf("%i: l/r: %3.3f - %3.3f b/t: %3.3f - %3.3f\n",i,elem_bb[i].l,elem_bb[i].r,elem_bb[i].b,elem_bb[i].t);
		}
	}
	
	
	int aligned[6][MAX_ELEMENTS][MAX_ELEMENTS];
	float align_dist[6][MAX_ELEMENTS][MAX_ELEMENTS];
	float locations[MAX_ELEMENTS];


	float scale=max(d->width, d->height);

	int align_cnt=0;

	for (int k=0;k<6;k++)
	{
		for (int i=0;i<ne;i++)
		{
			if (k==0)
				locations[i]=elem_bb[i].l/scale;
			else if (k==1)
				locations[i]=((elem_bb[i].l+elem_bb[i].r)/2.0)/scale;
			else if (k==2)
				locations[i]=elem_bb[i].r/scale;
			else if (k==3)
				locations[i]=elem_bb[i].b/scale;
			else if (k==4)
				locations[i]=((elem_bb[i].t+elem_bb[i].b)/2.0)/scale;
			else
				locations[i]=elem_bb[i].t/scale;
		}

		for (int i=0;i<ne;i++)
		{
			//aligned[k][i][i]=0;
	
			for (int j=i+1;j<ne;j++)
			{
				
				aligned[k][i][j]=0;
				aligned[k][j][i]=0;
				
				float loc_diff=min(abs(locations[i]-locations[j]),0.99);
			
				align_dist[k][i][j]=loc_diff;
				//align_dist[k][j][i]=loc_diff;

			}
		}
	}
	

	for (int i=0;i<ne;i++)
	{
		for (int k=0;k<6;k++)
			aligned[k][i][i]=0;

		for (int j=i+1;j<ne;j++)
		{

			
			float d0=align_dist[0][i][j];
			float d1=align_dist[1][i][j];
			float d2=align_dist[2][i][j];
			float d3=align_dist[3][i][j];
			float d4=align_dist[4][i][j];
			float d5=align_dist[5][i][j];



			if (min(min(d0 ,d1),d2)<ALIGN_THRESH)
			{
				if ((d0< d1) and (d0<d2))
				{
					aligned[0][i][j]=1;
					//aligned[0][j][i]=1;
				}
				else if ((d1< d0) and (d1<d2))
				{
					aligned[1][i][j]=1;
					//aligned[1][j][i]=1;
				}
				else
				{
					aligned[2][i][j]=1;
					//aligned[2][j][i]=1;
				}
				align_cnt++;
			}

			if (min(min(d3 ,d4),d5)<ALIGN_THRESH)
			{
				if ((d3< d4) and (d3<d5))
				{
					aligned[3][i][j]=1;
					//aligned[3][j][i]=1;
				}
				else if ((d4< d3) and (d3<d5))
				{
					aligned[4][i][j]=1;
					//aligned[4][j][i]=1;
				}
				else
				{
					aligned[5][i][j]=1;
					//aligned[5][j][i]=1;
				}
				align_cnt++;
			}
		}
	}

	
	
	
	int num_constraints=align_cnt;
	
	float **C=(float **)malloc(num_constraints*sizeof(float *));
	
	for (int i=0;i <num_constraints;i++)
	{
		C[i]=(float *)malloc(ne*2*sizeof(float));
		//memset(C[i],0,ne*2*sizeof(float));
		for (int j=0;j < 2*ne;j++)
			C[i][j]=0;
		
	}

	
	
	//float *loc=(float *)malloc(ne*2*sizeof(float));
	//memset(loc,0,ne*2*sizeof(float));
	//float *b=(float *)malloc(num_constraints*sizeof(float));
	//memset(b,0,num_constraints*sizeof(float));
	
	
	int cnt=0;
	
	for (int k=0;k<6;k++)
	{
		int offset = (int(k>=3))*ne;
		for (int i=0;i<ne;i++)
		for (int j=i+1;j<ne;j++)
		{		
				if (aligned[k][i][j])
				{
					if (debug)
						printf("constraint between element %i and %i, type %i, dist %f\n",i,j,k,align_dist[k][i][j]);
					C[cnt][offset+i]=1;
					C[cnt][offset+j]=-1;
					//b[cnt] = -1*align_dist[k][i][j];
					cnt++;
				}
		}
	}
	
	/*
	for (int i=0;i<ne;i++)
	{
		loc[i]=elem_bb[i].l/scale;
		loc[i+ne]=elem_bb[i].b/scale;
	}
	*/
	
	if (debug)
		printf("getting height constraints\n");
	
	
	float **H=(float **)malloc(ne*sizeof(float *));
	
	//int num_height_constraint=0;
	
	for (int i=0;i <ne;i++)
	{
		H[i]=(float *)malloc(ne*3*sizeof(float));
		//memset(C[i],0,ne*2*sizeof(float));
		for (int j=0;j < 3*ne;j++)
			H[i][j]=0;
		
		//check x-alignment
		int x_align=0;
		int y_align=3;
		

		
		for (int j=0;j<ne;j++)
		{
			int m=min(i,j);
			int n=max(i,j);
			
			if (aligned[2][m][n])
				x_align=2;
			else if (aligned[1][m][n])
				x_align=1;
				
			if (aligned[5][m][n])
				y_align=5;
			else if (aligned[4][m][n])
				y_align=4;			
		}
		
		if (debug)
			printf("height constraints %i, align %i %i\n",i,x_align,y_align);
			
		//set height 
		H[i][i+2*ne]=1;
		
		//x-center aligned, so have to shift left half the new width
		if (x_align==1)
		{
			H[i][i]=-0.5*(1.0/aspect_ratio[i]);
			if (debug)
				printf("\t center aligned, aspect ratio %f, shift %.2f\n",aspect_ratio[i],-0.5*(1.0/aspect_ratio[i]));
		}
		//right aligned, so have to shift left the new width
		else if(x_align==2)
			H[i][i]=-(1.0/aspect_ratio[i]);
		
		
		//y-center aligned, so have to shift down half the new width
		if (y_align==4)
			H[i][i+ne]=-0.5;
		//top aligned, so have to shift down the new height
		else if(y_align==5)
			H[i][i+ne]=-1;
		
	}
	
	
	
	//free(loc);

	//*b_out=b;
	//*loc_out=loc;
	*C_out=C;
	*H_out=H;
	
	return num_constraints;
	
}

       
void getSVD(int num_constraints, int num_var, float **C, float ***Q_out, float ***W_out, float **a_out)
{
	

	float **Q=(float **)malloc(num_var*sizeof(float *));
	float **W=(float **)malloc(num_var*sizeof(float *));
	
	for (int i=0;i <num_var;i++)
	{
		Q[i]=(float *)malloc(num_var*sizeof(float));
		memset(Q[i],0,num_var*sizeof(float));
		W[i]=(float *)malloc(num_var*sizeof(float));
		memset(W[i],0,num_var*sizeof(float));	
	}
	
	
	float *a=(float *)malloc(num_var*sizeof(float));
	memset(a,0,num_var*sizeof(float));	
	
	
	for (int i=0;i<num_var;i++)
	for (int j=0;j<num_var;j++)
	{
		float prod=0;
		
		for (int k=0;k<num_constraints;k++)
			prod+=C[k][i]*C[k][j];
		
		Q[i][j]=prod;
		
	}
	
	
	dsvd(Q, num_var, num_var, a,W);
	
	for (int i=0;i<num_var;i++)
	{
		//printf("SV %i, %.3f\n",i,a[i]);
		
		for (int j=0;j<num_var;j++)
		{
			//printf("\t %.3f\n",W[j][i]);
			if 	(a[i]<0.001)
			{
				if (abs(W[j][i])>0.01)
					W[j][i]=1;
				else
					W[j][i]=0;
			}
		}
		
	}
	
	/*
    Q=np.dot(C.T, C)
    W,A,V=la.svd(Q)  
    A2=np.diag(1/np.sqrt(A+0.0001))
    D=np.dot(W,A2)
	*/



	*Q_out=Q;
	*W_out=W;
	*a_out=a;
	
}      
       
       
int  getConstraintDirections(Design *h_d, float *h_init_layout, float **D_out,bool debug)
{
	
	int nv=h_d->num_elements*3;
	int ne=h_d->num_elements;
	
	float **C,**H;
	//float *b, *loc;
	int nc= getConstraints(h_d, h_init_layout, &C, &H,debug);
	
	
	float **Q,**W,*a;
	getSVD(nc,h_d->num_elements*2,C, &Q, &W, &a);
	
	
	
	int nd=0;

	for (int i=0;i <ne*2;i++)
	{
		if (a[i]<0.01)
			nd++;
		
	}

	nd+=ne;
	
	if (debug)
		printf("num dir %i, num var %i,num constraints %i, num elements %i\n",nd,nv,nc,ne);
	


	float *D=(float *)malloc(nv*nd*sizeof(float));
	memset(D,0,nv*nd*sizeof(float));

	
	//set position directions
	int d_cnt=0;
	for (int i=0;i <ne*2;i++)
	{
		if (a[i]<0.01)
		{	
			if (debug)
				printf("constraint direction %i %i\n",i,nd);
			
			for (int j=0;j<ne*2;j++)
			{
				if ((W[j][i]!=0) && (debug))
					printf("\t x/y:%i, elem:%i, c:%.3f\n",j/(ne),j%(ne),W[j][i]);
				D[d_cnt*nv+j]=W[j][i];
			}
			d_cnt+=1;
		}
	}
	
	//memcpy(&(D[((d_cnt)*nv)]), H, ne*3*sizeof(float));
	
	
	//heights
	for (int i=0;i <ne;i++)
	{	
		if (debug)
			printf("height direction %i\n",i);
			
		int idx=(i+d_cnt)*nv ;
		for (int j=0;j <3*ne;j++) 
		{
			if  ((H[i][j]!=0) && (debug))
				printf("\t %.3f\n",H[i][j]);
			D[idx+ j]=H[i][j];
		}
	}
	
	
	//free memory
	for (int i=0;i <nc;i++)
		free(C[i]);
		
	for (int i=0;i <ne;i++)
		free(H[i]);
	
	if (nc>0)
	{
		free(C);
		free(H);
		//free(b);
		//free(loc);
	}
	
	
	for (int i=0;i <ne*2;i++)
	{
		free(Q[i]);
		free(W[i]);
	}
	free(Q);
	free(W);
	free(a);
	
	
	*D_out=D;
	
	return nd;
}      
       

float constrainedGradientDescent(Design *h_d,Design *d,int num_params, float *params,float *atan_params, float *params_grads, float *h_params,  float *h_init_layout,int num_previous_layout, float *previous_layout)
{
	
	
	int num_var=h_d->num_elements*3 ;
	
	static float dir_time=0;
	static float gd_time=0;
	static float fd_time=0;
	static int cnt_time=0;
	//float fd_time1=0;
	int fd_cnt=0;
	
	clock_t start=clock();
	
	
	float *h_directions;
	int num_dir= getConstraintDirections(h_d, h_init_layout, &h_directions,false);
	
	clock_t end_dir=clock();
	
	cnt_time++;
	dir_time+=end_dir-start;
	
	
	float *h_dir_map=(float *)malloc(num_dir*h_d->layout_size*sizeof(float));
	memset(h_dir_map,0,num_dir*h_d->layout_size*sizeof(float));
	
	for (int i=0;i< num_dir;i++)
	{
		float *h_dir=&(h_directions[i*num_var]);
		float *h_map=&(h_dir_map[i*h_d->layout_size]);
		
		for(int j=0;j<num_var;j++)
		{
			if (abs(h_dir[j])>0.01)
			{
				
				int elem=(j%h_d->num_elements);
				int elem_var=(j/h_d->num_elements);
				int idx=elem*NUM_VAR+elem_var;
				//printf("elem %i, var %i, idx %i, %f\n",elem,elem_var,idx,h_dir[j]);
				
				if (h_init_layout[elem*NUM_VAR+4]<FIX_LAYOUT_THRESH)
					h_map[idx]=h_dir[j];	
								
			}
		}
	}
		
	float *dir_map;
	CHECKCALL(hipMalloc(&dir_map,num_dir*h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMemcpy(dir_map, h_dir_map, num_dir*h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
	
	

	float *h_gradient=(float *)malloc(num_dir*sizeof(float));
	memset(h_gradient,0,num_dir*sizeof(float) );
	
	
	float *h_line_search=(float *)malloc(num_dir*NUM_LINE_STEPS*sizeof(float));
	memset(h_line_search,0,num_dir*NUM_LINE_STEPS*sizeof(float) );
	
	
	float *h_layout=(float *)malloc(h_d->layout_size*sizeof(float));
	memcpy(h_layout,h_init_layout,h_d->layout_size*sizeof(float));
	
	float *h_layout_next=(float *)malloc(h_d->layout_size*sizeof(float));
	memcpy(h_layout_next,h_init_layout,h_d->layout_size*sizeof(float));
	
	float *h_params_grad=(float *)malloc(num_params*sizeof(float));
	
	
	int max_iter=100;
	
	double init_fx=evaluateLayoutHost(d,h_d,num_params, params, params_grads,  h_layout,h_params,  h_params_grad,0,0,false);
	//printf("Init fx %f\n",init_fx);
	
	double fx=init_fx;
	double fx_next=fx+1;

	
    dim3 cudaGridSize(1, 1);
    dim3 cudaBlockSize(num_dir,1);
	
	
    //dim3 cudaGridSize(1, 1);
   // dim3 cudaBlockSize(num_var,1);
	
	float *layouts,*layout,*gradient, *directions,*line_search;
	CHECKCALL(hipMalloc(&layouts,num_var*h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMalloc(&layout,h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMalloc(&directions,num_var*num_dir*sizeof(float)));
	CHECKCALL(hipMalloc(&gradient,num_var*sizeof(float)));
	CHECKCALL(hipMalloc(&line_search,num_dir*NUM_LINE_STEPS*sizeof(float)));
	
	CHECKCALL(hipMemcpy(gradient, h_gradient, num_dir*sizeof(float), hipMemcpyHostToDevice));
	CHECKCALL(hipMemcpy(directions, h_directions, num_var*num_dir*sizeof(float), hipMemcpyHostToDevice));
	
	bool refresh_grad=true;
	
	int i=0;
	int last_update=0;
	for (i=0;i< max_iter;i++)
	for (int dir=0;dir<num_dir;dir++)
	{
		
		
		if (refresh_grad)
		{
			CHECKCALL(hipMemcpy(layout, h_layout,  h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
			
			clock_t fd_start=clock();
			
			
			finiteDiffLayoutGrad<<<cudaGridSize, cudaBlockSize>>>(d,num_dir,directions,dir_map,num_params,params, atan_params, params_grads, layout,layouts, gradient,line_search,num_previous_layout, previous_layout);
	
	    	CHECKCALL(hipGetLastError());
	    	CHECKCALL(hipDeviceSynchronize());
	
			CHECKCALL(hipMemcpy(h_gradient, gradient,num_dir*sizeof(float), hipMemcpyDeviceToHost));
			CHECKCALL(hipMemcpy(h_line_search, line_search,num_dir*NUM_LINE_STEPS*sizeof(float), hipMemcpyDeviceToHost));
			
			fd_time+=clock()-fd_start;
			fd_cnt++;
			
			//printf("i %i, dir %i, fd_cnt %i\n",i,dir,fd_cnt);
			
			refresh_grad=false;
		}
		
		
		float *h_dir=&(h_dir_map[dir*h_d->layout_size]);
		if ((!finite(h_gradient[dir])))
		{
			printf("ERROR in gradient %f, dir %i\n",h_gradient[dir],dir);
			continue;			
		}
		
		//memcpy(h_layout_next,h_layout,h_d->layout_size*sizeof(float));
		
		
		float *h_lsearch=&(h_line_search[dir*NUM_LINE_STEPS]);
		
		
		double delta=FD_DELTA;
		if (h_gradient[dir]>0)
			delta=-1*delta;
			
		//printf("start, eval %f\n",h_lsearch[0]);
		
		float min_eval=h_lsearch[0];
		float min_delta=0;
			
		for (int j=1;j<NUM_LINE_STEPS;j++)
		{
			if (min_eval>h_lsearch[j])
			{
				min_eval=h_lsearch[j];
				min_delta=delta;
				
			}
			else
				break;
			
			delta=delta*2;
		}
		
		for(int j=0;j<h_d->layout_size;j++)
			h_layout_next[j]=h_layout[j]+min_delta*h_dir[j];	

		
		fx_next=min_eval;
		//fx_next=evaluateLayoutHost(d,h_d,num_params, params, params_grads,  h_layout_next,h_params,  h_params_grad,false);
		//if (fx_next!=min_eval)
		//	printf("ERROR dir %i, fx next %f, min_eval %f\n",dir,fx_next,min_eval);
			
		if (fx_next==INVALID_ELEMENT_ERROR)
			printf("INVALID_ELEMENT_ERROR called from outer constraint step\n");
		
		

		
		//printf("dir %i, grad %.3f,stepsize %.3f, fx %.3f,last fx %.3f,diff %.3f\n",dir,h_gradient[dir],h_gradient[dir]*grad_par,fx,last_fx,fx-last_fx);
		
		/*
		if (!finite(fx_next)) 
		{
			fx_next=evaluateLayoutHost(d,h_d,num_params, params, params_grads,  h_layout_next,h_params,  h_params_grad,true);
				
			printf("ERROR. Not finite.  %f\n",fx_next);
			
			for (int k=0;k< h_d->num_elements;k++)
				printf("%f %f %f\n",h_layout_next[k*NUM_VAR],h_layout_next[k*NUM_VAR+1],h_layout_next[k*NUM_VAR+2]);
			
			
			for (int k=0;k<NUM_FEATURES;k++)
				printf("%i\t%5.1f \t %4.2f\t %4.2f \t nl: %4.2f %s \n", k,h_params[k],h_params_grad[k],h_params[k]*h_params_grad[k],h_params[k+NUM_FEATURES],feat_names[k]);
		
		}
		*/
		
		//try to make an update 
		if ((finite(fx_next)) && (fx_next<fx-0.05))
		{
			memcpy(h_layout,h_layout_next,h_d->layout_size*sizeof(float));
			last_update=i;
			//printf("GD iteration %i,fx %.3f, dir %i, grad %.3f, fx diff %.3f\n",i,fx,dir,h_gradient[dir],last_fx-fx);	
			refresh_grad=true;
			
			fx=fx_next;
		}
	
		
		//printf("GD iteration %i, fx %.3f\n",i, fx);		
		
		if (i-last_update>1)
		{
			
			
			free(h_layout_next);
			free(h_gradient);
			
			CHECKCALL(hipFree(line_search));
			CHECKCALL(hipFree(directions));
			CHECKCALL(hipFree(layouts));
			CHECKCALL(hipFree(layout));
			CHECKCALL(hipFree(gradient));
				
			clock_t end=clock();
			gd_time+=(end-end_dir);
			
			
			
			printf("iteration %i, last_update %i,fd cnt %i mean time %.3f, dir time %.3f, gd time %.3f\n",i,last_update,fd_cnt, 1000*(fd_time/cnt_time)/CLOCKS_PER_SEC,1000*(dir_time/cnt_time)/CLOCKS_PER_SEC , 1000*(gd_time/cnt_time)/CLOCKS_PER_SEC); 
			
			
			if (fx<init_fx)
			{
				//printf("GD from %.4f to %.4f. imp %.3f, in %i iterations \n",init_fx,fx,fx-init_fx,i-1);
				memcpy(h_init_layout ,h_layout,h_d->layout_size*sizeof(float));
				free(h_layout);
				return fx;
			}
			
			return init_fx;
		}
	}
	
	printf("Returning init fx %f\n",init_fx);
	//CHECKCALL(hipFree(layouts));
	//return init_fx;
	return init_fx;

}




/*
void finiteDiffLayoutGrad(Design *d,Design *h_d,int num_params, float *params, float *params_grads, float *h_params,  float *h_params_grad, float *h_layout, float *h_layout_grad)
{

	int num_var=h_d->num_elements*NUM_VAR;
	
	memset(h_layout_grad,0,sizeof(float) * h_d->layout_size);

	float *h_layout_copy=(float *)malloc(sizeof(float) * h_d->layout_size);
	memcpy(h_layout_copy,h_layout,sizeof(float) * h_d->layout_size);

	double delta=0.001;

	for (int i=0;i<num_var;i++)
	{
		
		if (i%NUM_VAR>2)
			continue;

		for(int j=0;j<num_var;j++)
			h_layout_copy[j]=h_layout[j];

		h_layout_copy[i]=h_layout[i]+delta;

		double y2=evaluateLayoutHost(d,h_d,num_params, params, params_grads,  h_layout_copy,h_params,  h_params_grad,false);


		h_layout_copy[i]=h_layout[i]-delta;

		double y1=evaluateLayoutHost(d,h_d,num_params, params, params_grads,  h_layout_copy,h_params,  h_params_grad,false);



		h_layout_grad[i]=(float)((y2-y1)/(2.0*delta));

		//printf("%i y1: %.3f, y2: %.3f, grad: %.3f\n",i,y1,y2,h_params_grad_fd[i]);
	}

	free(h_layout_copy);
}
*/




void finiteDiffGrad(Design *d,Design *h_d,int num_params, float *params, float *params_grad,  float *h_layout,float *h_params,  float *h_params_grad,float *h_params_grad_fd)
{

	float *h_params_copy=(float *)malloc(sizeof(float) * num_params);

	double delta=0.001;

	for (int i=0;i<num_params;i++)
	{

		for(int j=0;j<num_params;j++)
			h_params_copy[j]=h_params[j];

		h_params_copy[i]=h_params_copy[i]+delta;

		double y2=evaluateLayoutHost(d,h_d,num_params, params, params_grad,  h_layout,h_params_copy,  h_params_grad,0,0,false);

		for(int j=0;j<num_params;j++)
			h_params_copy[j]=h_params[j];

		h_params_copy[i]=h_params_copy[i]-delta;

		double y1=evaluateLayoutHost(d,h_d,num_params, params, params_grad,  h_layout,h_params_copy,  h_params_grad,0,0,false);

		//if (i>=NUM_FEATURES)
		//	delta2=exp(delta);

		h_params_grad_fd[i]=(float)((y2-y1)/(2.0*delta));

		//printf("%i y1: %.3f, y2: %.3f, grad: %.3f\n",i,y1,y2,h_params_grad_fd[i]);
	}

	free(h_params_copy);

}


float evaluateLayoutHost(Design *d,Design *h_d,int num_params, float *params, float *params_grad,  float *h_layout,float *h_params,  float *h_params_grad,int num_prev_layout,float *previous_layout, bool debug)
{
	float *eval, h_eval;
	CHECKCALL(hipMalloc(&eval,sizeof(float)));
	

	float *layout;
	CHECKCALL(hipMalloc(&layout,h_d->layout_size*sizeof(float)));
	CHECKCALL(hipMemcpy(layout, h_layout,  h_d->layout_size*sizeof(float), hipMemcpyHostToDevice));
	CHECKCALL(hipMemcpy(params, h_params,  num_params*sizeof(float), hipMemcpyHostToDevice));


	float *atan_params;
	CHECKCALL(hipMalloc(&atan_params,num_params*sizeof(float)));
	float *h_atan_params= (float *)malloc(num_params*sizeof(float));

	for (int j=0;j<num_params;j++)
		h_atan_params[j]=atan(h_params[j]);
	CHECKCALL(hipMemcpy(atan_params, h_atan_params,num_params*sizeof(float), hipMemcpyHostToDevice));


	if (debug)
		evaluateLayoutKernelDebug<<<1, 1>>>(d,layout,params, atan_params,params_grad,num_prev_layout,previous_layout, eval);
	else
		evaluateLayoutKernel<<<1, 1>>>(d,layout,params, atan_params,params_grad,num_prev_layout,previous_layout, eval);

	CHECKCALL(hipGetLastError());
	CHECKCALL(hipDeviceSynchronize());

	CHECKCALL(hipMemcpy(&h_eval, eval,  sizeof(float), hipMemcpyDeviceToHost));
	CHECKCALL(hipMemcpy(h_params_grad, params_grad,  num_params*sizeof(float), hipMemcpyDeviceToHost));


	/*
	float test_eval1=-500;
	for (int i=0;i<NUM_FEATURES;i++)
		test_eval1+=h_params[i]*h_params_grad[i];
	
	if ((!isnan(test_eval1)) && (abs(test_eval1-h_eval)>0.1))
		printf("error %f %f\n",test_eval1,h_eval);
	*/


	free(h_atan_params);
	CHECKCALL(hipFree(eval));
	CHECKCALL(hipFree(layout));
	CHECKCALL(hipFree(atan_params));

	return h_eval;
}




/* Catches signal interrupts from Ctrl+c.
   If 1 signal is detected the simulation finishes the current frame and
   exits in a clean state. If Ctrl+c is pressed again it terminates the
   application without completing writes to files or calculations but
   deallocates all memory anyway. */
void sig_handler (int sig)
{
  if ((sig == SIGTERM) || (sig==SIGINT))
    {     

      // write a function to free dynamycally allocated memory
      //free_mem ();
  
      cleanup();
    
   	  exit (9);
    }
}

void cleanup()
{
  
  freeMemory();

  int devCount;
  hipGetDeviceCount (&devCount);

  for (int i = 0; i < devCount; ++i)
    {
    std::cout << "hipDeviceReset\n";
      hipSetDevice (i);
      hipDeviceReset ();
    }


}







/*
size_t free_byte ;
size_t total_byte ;
hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
if ( hipSuccess != cuda_status ){
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(1);
}

double free_db = (double)free_byte ;
double total_db = (double)total_byte ;
double used_db = total_db - free_db ;

printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
    used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
*/



    /*

		
		//do refinement updates first
		for (int i=0;i< num_kernels-1;i++)
    	{
    		int curr=i;
    		float temp1=h_temperatures[curr];
    		float temp2=h_temperatures[curr+1];

    		float energy1=h_evals[curr];
    		float energy2=h_evals[curr+1];

    		float prop=exp((1.0/temp1-1.0/temp2)*(energy1-energy2));
    		float r=rand()/ double(RAND_MAX);			
			
			
			//switch state
    		if (r<min(1.0,prop))
			{
				
				float temp_eval=h_evals[curr];
				h_evals[curr]=h_evals[curr+1];
				h_evals[curr+1]=temp_eval;
				
				for (int k=0;k<h_d->layout_size;k++)
				{
					float temp_layout=h_opt_layouts[curr*h_d->layout_size+k];
					h_opt_layouts[curr*h_d->layout_size+k]=h_opt_layouts[(curr+1)*h_d->layout_size+k];
					h_opt_layouts[(curr+1)*h_d->layout_size+k]=temp_layout;
				}
			}
			
    	}
    	
	 	for (int i=0;i< num_kernels;i++)
				for (int j=0;j<h_d->num_elements;j++)
					if ((h_opt_layouts[i*h_d->layout_size + j*NUM_VAR+2]==0) && (h_opt_layouts[i*h_d->layout_size+ j*NUM_VAR+3]==0))
						printf("ERROR-2 in kernel %i element %i. init element is 0\n",i,j);
	    
    		
    	
    	//do refinement updates first
		for (int i=0;i< grid_size;i++)
    	for (int j=0;j< grid_size-1;j++)
    	{
    		int curr=i*grid_size+j;
    		float temp1=h_temperatures[curr];
    		float temp2=h_temperatures[curr+1];

    		float energy1=h_evals[curr];
    		float energy2=h_evals[curr+1];

    		float prop=exp((1.0/temp1-1.0/temp2)*(energy1-energy2));
    		float r=rand()/ double(RAND_MAX);			
			
			
			//switch state
    		if (r<min(1.0,prop))
			{
				
				float temp_eval=h_evals[curr];
				h_evals[curr]=h_evals[curr+1];
				h_evals[curr+1]=temp_eval;
				
				for (int k=0;k<h_d->layout_size;k++)
				{
					float temp_layout=h_opt_layouts[curr*h_d->layout_size+k];
					h_opt_layouts[curr*h_d->layout_size+k]=h_opt_layouts[(curr+1)*h_d->layout_size+k];
					h_opt_layouts[(curr+1)*h_d->layout_size+k]=temp_layout;
				}
			}
    	}
    	
    	
 		for (int i=0;i< num_kernels;i++)
			for (int j=0;j<h_d->num_elements;j++)
				if ((h_opt_layouts[i*h_d->layout_size + j*NUM_VAR+2]==0) && (h_opt_layouts[i*h_d->layout_size+ j*NUM_VAR+3]==0))
					printf("ERROR-2 in kernel %i element %i. init element is 0\n",i,j);

    	//switch layouts based on the diagonal
    	for (int i=0;i< grid_size-1;i++)
    	{
    		int curr=i*grid_size+i;
    		int next=(i+1)*grid_size+i+1;
    		
    		float temp1=h_temperatures[curr];
    		float temp2=h_temperatures[curr+1];

    		float energy1=h_evals[curr];
    		float energy2=h_evals[curr+1];

    		float prop=exp((1.0/temp1-1.0/temp2)*(energy1-energy2));
    		float r=rand()/ double(RAND_MAX);	

    		if (r<min(1.0,prop))
			{
				float temp_eval=h_evals[curr];
				h_evals[curr]=h_evals[next];
				h_evals[next]=temp_eval;

				//overwrite entire refinement optimizer with new layout
				
				
				for (int k=0;k<h_d->layout_size;k++)
				{
					float temp_layout=h_opt_layouts[curr*h_d->layout_size+k];
					h_opt_layouts[curr*h_d->layout_size+k]=h_opt_layouts[next*h_d->layout_size+k];
					h_opt_layouts[next*h_d->layout_size+k]=temp_layout;
				}
				
				for (int j=0;j< grid_size;j++) 
				{
					
					h_evals[i*grid_size+j]=h_evals[curr];
					h_evals[(i+1)*grid_size+j]=h_evals[next];		
					
					for (int k=0;k<h_d->layout_size;k++)
					{
						
						h_opt_layouts[(i*grid_size+j)*h_d->layout_size+k]=h_opt_layouts[(curr)*h_d->layout_size+k];
						h_opt_layouts[((i+1)*grid_size+j)*h_d->layout_size+k]=h_opt_layouts[(next)*h_d->layout_size+k];
						
					}
				}
			}
    	}
    	
   		for (int i=0;i< num_kernels;i++)
			for (int j=0;j<h_d->num_elements;j++)
				if ((h_opt_layouts[i*h_d->layout_size + j*NUM_VAR+2]==0) && (h_opt_layouts[i*h_d->layout_size+ j*NUM_VAR+3]==0))
					printf("ERROR-3 in kernel %i element %i. init element is 0\n",i,j);
    	* */
   
   

